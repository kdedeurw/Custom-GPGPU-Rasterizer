#include "hip/hip_runtime.h"
#include "PCH.h"
#include "CUDARenderer.cuh"
#include <vector>

#include "DEFINES.h"

//Project CUDA includes
#include "GPUTextureSampler.cuh"

//Project includes
#include "WindowHelper.h"
#include "SceneManager.h"
#include "SceneGraph.h"
#include "Mesh.h"
#include "Camera.h"
#include "Vertex.h"
#include "PrimitiveTopology.h"
#include "BoundingBox.h"
#include "GPUTextures.h"
#include "RGBRaw.h"
#include "Light.h"
#include "GPUTextures.h"
#include "CullingMode.h"
#include "CUDABenchMarker.h"

#pragma region STRUCT DECLARATIONS

struct RenderData
{
	FPoint3 camPos;
	FMatrix4 viewProjectionMatrix;
	FMatrix4 worldMatrix;
};

union RenderDataRaw
{
	//compiler BUG: attempting to reference a deleted function?
	float* data;
	RenderData renderData;
};

struct RasterTriangle
{
	FPoint4 v0;
	FPoint4 v1;
	FPoint4 v2;
};

struct TriangleIdx
{
	unsigned int idx0;
	unsigned int idx1;
	unsigned int idx2;
	//bool isCulled; //padded anyway
};

struct PixelShade // size == 38
{
	unsigned int colour;
	float zInterpolated;
	float wInterpolated;
	FVector2 uv;
	FVector3 n;
	FVector3 tan;
	FVector3 vd;
	GPUTexturesCompact textures;
};

struct CUDAMesh
{
	FPoint4* pPositions;
	FVector2* pUVs;
	FVector3* pNormals;
	FVector3* pTangents;
	RGBColor* pColours;
};

#pragma endregion

#pragma region GLOBAL VARIABLES

constexpr int NumTextures = 4;

//CONST DEVICE MEMORY - Does NOT have to be allocated or freed
GPU_CONST_MEMORY static float dev_RenderData_const[sizeof(RenderData) / sizeof(float)]{};
GPU_CONST_MEMORY float dev_CameraPos_const[sizeof(FPoint3) / sizeof(float)]{};
GPU_CONST_MEMORY float dev_WVPMatrix_const[sizeof(FMatrix4) / sizeof(float)]{};
GPU_CONST_MEMORY float dev_WorldMatrix_const[sizeof(FMatrix4) / sizeof(float)]{};
GPU_CONST_MEMORY float dev_RotationMatrix_const[sizeof(FMatrix3) / sizeof(float)]{};
//NOTE: cannot contain anything else besides primitive variables (int, float, etc.)

//DEVICE MEMORY - Does have to be allocated and freed
static unsigned int* dev_FrameBuffer{};
static int* dev_DepthBuffer{}; //defined as INTEGER type for atomicCAS to work properly
static int* dev_Mutex{};
static PixelShade* dev_PixelShadeBuffer{}; //(== fragmentbuffer)
//static std::vector<FPoint4*> dev_IPositions{};
//static std::vector<float*> dev_IVertexDatas{};
//static std::vector<FPoint4*> dev_OPositions{};
//static std::vector<float*> dev_OVertexDatas{};
static std::vector<IVertex_Point4*> dev_IVertexBuffer{};
static std::vector<unsigned int*> dev_IndexBuffer{};
static std::vector<OVertex*> dev_OVertexBuffer{};
static std::vector<TriangleIdx*> dev_Triangles{};
static std::vector<unsigned int*> dev_TextureData{};
/*
//DEPRECATED
//Texture references have to be statically declared in global memory and bound to CUDA texture memory
//They cannot be referenced in functions, nor used in arrays
typedef texture<unsigned int, hipTextureType2D, hipReadModeElementType> CUDA32bTexture2D;
static CUDA32bTexture2D dev_DiffuseTextureReference{};
static CUDA32bTexture2D dev_NormalTextureReference{};
static CUDA32bTexture2D dev_SpecularTextureReference{};
static CUDA32bTexture2D dev_GlossinessTextureReference{};
*/

#pragma endregion

//--------------------------

CPU_CALLABLE
CUDARenderer::CUDARenderer(const WindowHelper& windowHelper)
	: m_WindowHelper{ windowHelper }
	, m_TotalNumTriangles{}
	, m_TotalVisibleNumTriangles{}
	, m_TimerMs{}
	, m_h_pFrameBuffer{}
	, m_MeshIdentifiers{}
	, m_TextureObjects{}
{
	InitCUDADeviceBuffers();
	CheckErrorCuda(hipEventCreate(&m_StartEvent));
	CheckErrorCuda(hipEventCreate(&m_StopEvent));
}

CPU_CALLABLE
CUDARenderer::~CUDARenderer()
{
	CheckErrorCuda(DeviceSynchroniseCuda());
	CheckErrorCuda(hipEventDestroy(m_StartEvent));
	CheckErrorCuda(hipEventDestroy(m_StopEvent));
	FreeCUDADeviceBuffers();
}

#pragma region MISC HELPER FUNCTIONS

CPU_CALLABLE static
std::string ToKbs(size_t bytes)
{
	const size_t toKbs = 1024;
	std::string output{ std::to_string(bytes / toKbs) + "Kb" };
	return output;
}

CPU_CALLABLE static
std::string ToMbs(size_t bytes)
{
	const size_t toMBs = 1024 * 1024;
	std::string output{ std::to_string(bytes / toMBs) + "Mb" };
	return output;
}

CPU_CALLABLE static
std::string ToGbs(size_t bytes)
{
	const size_t toGBs = 1024 * 1024 * 1024;
	std::string output{ std::to_string(bytes / toGBs) + "Gb" };
	return output;
}

BOTH_CALLABLE static
float GetMinElement(float val0, float val1, float val2)
{
	float min = val0;
	if (val1 < min)
		min = val1;
	if (val2 < min)
		min = val2;
	return min;
}

BOTH_CALLABLE static
float GetMaxElement(float val0, float val1, float val2)
{
	float max = val0;
	if (val1 > max)
		max = val1;
	if (val2 > max)
		max = val2;
	return max;
}

#pragma endregion

#pragma region CPU HELPER FUNCTIONS

CPU_CALLABLE
void CUDARenderer::DisplayGPUSpecs(int deviceId)
{
	std::string yn{};

	std::cout << "\n---General---\n";
	hipDeviceProp_t prop;
	CheckErrorCuda(hipGetDeviceProperties(&prop, deviceId));
	std::cout << "Device detected: " << prop.name << '\n';
	std::cout << "Compute Capability: " << prop.major << '.' << prop.minor << '\n';
	std::cout << "Compute Mode: ";
	switch (prop.computeMode)
	{
	case hipComputeModeDefault:
		std::cout << "Default\n";
		break;
	case hipComputeModeExclusive:
		std::cout << "Exclusive\n";
		break;
	case hipComputeModeProhibited:
		std::cout << "Prohibited\n";
		break;
	case hipComputeModeExclusiveProcess:
		std::cout << "ExclusiveProcess\n";
		break;
	default:
		std::cout << "Undefined\n";
		break;
	}
	if (prop.isMultiGpuBoard)
	{
		std::cout << "Multi GPU setup: Yes\n";
		std::cout << "Multi GPU boardgroup ID: " << prop.multiGpuBoardGroupID << '\n';
	}
	else
	{
		std::cout << "Multi GPU setup: No\n";
	}
	std::cout << "Async Engine (DMA) count: " << prop.asyncEngineCount << '\n';
	yn = prop.deviceOverlap ? "Yes\n" : "No\n";
	std::cout << "Can concurrently copy memory between host and device while executing kernel: " << yn;
	switch (prop.asyncEngineCount)
	{
	case 0:
		std::cout << "Device cannot concurrently copy memory between host and device while executing a kernel\n";
		break;
	case 1:
		std::cout << "Device can concurrently copy memory between host and device while executing a kernel\n";
		break;
	case 2:
		std::cout << "Device can concurrently copy memory between host and device in both directions and execute a kernel at the same time\n";
		break;
	default:
		break;
	}
	yn = prop.concurrentKernels ? "Yes\n" : "No\n";
	std::cout << "Device supports executing multiple kernels within the same context simultaneously: " << yn;
	yn = prop.integrated ? "Yes\n" : "No\n";
	std::cout << "Integrated Graphics: " << yn;

	std::cout << "\n---Memory---\n";
	std::cout << "Total amount of Global Memory: " << ToMbs(prop.totalGlobalMem) << '\n';
	std::cout << "Total amount of Const Memory: " << ToKbs(prop.totalConstMem) << '\n';
	//size_t free{}, total{};
	//hipMemGetInfo(&free, &total);
	//std::cout << "Total amount of VRAM: " << total << '\n';
	//std::cout << "Free amount of VRAM: " << free << '\n';
	std::cout << "Shared Memory per Multiprocessor: " << ToKbs(prop.sharedMemPerMultiprocessor) << '\n';
	std::cout << "Shared Memory per Block: " << ToKbs(prop.sharedMemPerBlock) << '\n';

	std::cout << "Shared Memory Reserved by CUDA driver per Block: " << prop.reservedSharedMemPerBlock << " bytes\n";

	yn = prop.unifiedAddressing ? "Yes\n" : "No\n";
	std::cout << "Unified Addressing supported: " << yn;
	yn = prop.managedMemory ? "Yes\n" : "No\n";
	std::cout << "Managed Memory supported: " << yn;
	yn = prop.pageableMemoryAccess ? "Yes\n" : "No\n";
	std::cout << "Device can coherently access Pageable Memory (non-pinned memory): " << yn;
	yn = prop.pageableMemoryAccessUsesHostPageTables ? "Yes\n" : "No\n";
	std::cout << "Device can access pageable memory via host's page tables: " << yn;
	yn = prop.canMapHostMemory ? "Yes\n" : "No\n";
	std::cout << "Can Map host memory: " << yn;

	std::cout << "\n---Memory - Caching---\n";
	yn = prop.globalL1CacheSupported ? "Yes\n" : "No\n";
	std::cout << "Global L1 Cache Supported: " << yn;
	yn = prop.localL1CacheSupported ? "Yes\n" : "No\n";
	std::cout << "Local L1 Cache Supported: " << yn;
	std::cout << "L2 Cache Size: " << ToKbs(prop.l2CacheSize) << '\n';;
	std::cout << "Persisting L2 Cache Max Size: " << prop.persistingL2CacheMaxSize << " bytes\n";

	std::cout << "\n---Other---\n";
	std::cout << "ClockRate: " << prop.clockRate / 1000 << "Khz\n";
	std::cout << "Memory ClockRate: " << prop.memoryClockRate / 1000 << "Khz\n";
	std::cout << "Memory Pitch: " << prop.memPitch << " bytes\n";
	std::cout << "Maximum number of 32-bit registers per Multiprocessor: " << prop.regsPerMultiprocessor << "\n";
	std::cout << "Maximum number of 32-bit registers per Block: " << prop.regsPerBlock << "\n";

	std::cout << "\n---Thread specifications---\n";
	std::cout << "Max threads per Multiprocessor: " << prop.maxThreadsPerMultiProcessor << '\n';
	std::cout << "Max threads per Block: " << prop.maxThreadsPerBlock << '\n';
	std::cout << "Max threads Dimensions: X: " << prop.maxThreadsDim[0] << ", Y: " << prop.maxThreadsDim[1] << ", Z: " << prop.maxThreadsDim[2] <<'\n';
	std::cout << "Warp Size (in threads): " << prop.warpSize << '\n';
	std::cout << '\n';
}

CPU_CALLABLE
void CUDARenderer::InitCUDADeviceBuffers()
{
	DisplayGPUSpecs(0);

	size_t size{};
	const unsigned int width = m_WindowHelper.Width;
	const unsigned int height = m_WindowHelper.Height;

	//CUDAHOSTALLOC FLAGS
	/*
	hipHostMallocDefault: This flag's value is defined to be 0 and causes hipHostAlloc() to emulate hipHostMalloc().
	hipHostMallocPortable: The memory returned by this call will be considered as pinned memory by all CUDA contexts, not just the one that performed the allocation.
	hipHostMallocMapped: Maps the allocation into the CUDA address space. The device pointer to the memory may be obtained by calling hipHostGetDevicePointer().
	hipHostMallocWriteCombined: Allocates the memory as write-combined (WC).
	WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read efficiently by most CPUs.
	WC memory is a good option for buffers that will be written by the CPU and read by the device via mapped pinned memory or host->device transfers.
	*/

	//--->PINNED MEMORY<--- (HOST ONLY)
	//+ makes memory transactions between host and device significantly faster
	//- however this will allocate on host's RAM memory (in this case it would be 640 * 480 * 4 bytes == 1.2288Mb)
	size = sizeof(unsigned int);
	CheckErrorCuda(hipHostMalloc((void**)&m_h_pFrameBuffer, width * height * size));
	//CheckErrorCuda(hipHostAlloc((void**)&m_h_pFrameBuffer, width * height * size, hipHostMallocPortable));
	
	//host pinned memory without SDL window pixelbuffer
	//SDL allows random access to pixelbuffer, but cuda does not allowed host memory to be there

	//CUDAHOSTREGISTER FLAGS
	/*
	hipHostRegisterDefault: On a system with unified virtual addressing, the memory will be both mapped and portable. 
	On a system with no unified virtual addressing, the memory will be neither mapped nor portable.
	hipHostRegisterPortable: The memory returned by this call will be considered as pinned memory by all CUDA contexts, not just the one that performed the allocation.
	hipHostRegisterMapped: Maps the allocation into the CUDA address space. The device pointer to the memory may be obtained by calling hipHostGetDevicePointer().
	hipHostRegisterIoMemory: The passed memory pointer is treated as pointing to some memory-mapped I/O space, 
	e.g. belonging to a third-party PCIe device, and it will marked as non cache-coherent and contiguous.
	hipHostRegisterReadOnly: The passed memory pointer is treated as pointing to memory that is considered read-only by the device.
	On platforms without hipDeviceAttributePageableMemoryAccessUsesHostPageTables, this flag is required in order to register memory mapped to the CPU as read-only.
	Support for the use of this flag can be queried from the device attribute cudaDeviceAttrReadOnlyHostRegisterSupported.
	Using this flag with a current context associated with a device that does not have this attribute set will cause hipHostRegister to error with hipErrorNotSupported.
	*/

	//Invalid Argument
	//SDL_LockSurface(m_WindowHelper.pBackBuffer);
	//size = sizeof(unsigned int);
	//CheckErrorCuda(hipHostRegister(m_WindowHelper.pBackBufferPixels, width * height * size, hipHostRegisterDefault));
	//SDL_UnlockSurface(m_WindowHelper.pBackBuffer);;

	size = sizeof(PixelShade);
	CheckErrorCuda(hipFree(dev_PixelShadeBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_PixelShadeBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_PixelShadeBuffer, 0, width * height * size));

	//The framebuffer in device memory
	size = sizeof(unsigned int);
	CheckErrorCuda(hipFree(dev_FrameBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_FrameBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_FrameBuffer, 0, width * height * size));

	size = sizeof(int);
	CheckErrorCuda(hipFree(dev_DepthBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_DepthBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_DepthBuffer, 0, width * height * size));

	size = sizeof(int);
	hipFree(dev_Mutex);
	hipMalloc((void**)&dev_Mutex, width * height * size);
	hipMemset(dev_Mutex, 0, width * height * size);

	//NOTE: can only set data PER BYTE
	//PROBLEM: setting each byte to UCHAR_MAX (255) is impossible, since floating point numbers work differently (-nan result)
	//	0		11111110	11111111111111111111111
	//	^			^				^
	//	sign	exponent		mantissa
	//			254 - 127     2 - 2 ^ (-23)
	// 340282346638528859811704183484516925440.0   // FLT_MAX
	// 340282366920938463463374607431768211456.0   // 2^128
	//https://stackoverflow.com/questions/16350955/interpreting-the-bit-pattern-of-flt-max
	//SOLUTION:
	//Option 1: allocate float[width*height] and initialize to FLT_MAX, then memcpy (wastes lots of memory)
	//Option 2: loop through entire dev_array and set each member to FLT_MAX (too many global accesses)
	//>Option 3<: interpret depth buffer invertedly, so a depthvalue of 1.f is closest, and 0.f is furthest away from camera
	//Option 4: initialize and reset depthbuffer through additional kernel call, however this would be a lot of global memory accesses
}

CPU_CALLABLE
void CUDARenderer::AllocateMeshBuffers(const size_t numVertices, const size_t numIndices, const size_t numTriangles, size_t meshIdx)
{
	//If no sufficient space in vector, enlarge
	const size_t newSize = meshIdx + 1;
	if (newSize > dev_IVertexBuffer.size())
	{
		//TODO: reserve
		dev_IVertexBuffer.resize(newSize);
		dev_IndexBuffer.resize(newSize);
		dev_OVertexBuffer.resize(newSize);
		dev_Triangles.resize(newSize);
	}

	//Free unwanted memory
	CheckErrorCuda(hipFree(dev_IVertexBuffer[meshIdx]));
	CheckErrorCuda(hipFree(dev_IndexBuffer[meshIdx]));
	CheckErrorCuda(hipFree(dev_OVertexBuffer[meshIdx]));
	CheckErrorCuda(hipFree(dev_Triangles[meshIdx]));

	//Allocate Input Vertex Buffer
	CheckErrorCuda(hipMalloc((void**)&dev_IVertexBuffer[meshIdx], numVertices * sizeof(IVertex_Point4)));
	//Allocate Index Buffer
	CheckErrorCuda(hipMalloc((void**)&dev_IndexBuffer[meshIdx], numIndices * sizeof(unsigned int)));
	//Allocate Ouput Vertex Buffer
	CheckErrorCuda(hipMalloc((void**)&dev_OVertexBuffer[meshIdx], numVertices * sizeof(OVertex)));
	//Allocate device memory for entire range of triangles
	CheckErrorCuda(hipMalloc((void**)&dev_Triangles[meshIdx], numTriangles * sizeof(TriangleIdx)));
}

CPU_CALLABLE
void CUDARenderer::CopyMeshBuffers(float* vertexBuffer, unsigned int numVertices, short stride, unsigned int* indexBuffer, unsigned int numIndices, size_t meshIdx)
{
	//Copy Input Vertex Buffer
	CheckErrorCuda(hipMemcpy(dev_IVertexBuffer[meshIdx], vertexBuffer, numVertices * stride, hipMemcpyHostToDevice));
	//Copy Index Buffer
	CheckErrorCuda(hipMemcpy(dev_IndexBuffer[meshIdx], indexBuffer, numIndices * sizeof(unsigned int), hipMemcpyHostToDevice));
}

CPU_CALLABLE
void CUDARenderer::LoadMeshTextures(const std::string texturePaths[4], size_t meshIdx)
{
	const size_t newSize = (meshIdx + 1);
	if (newSize > m_TextureObjects.size())
	{
		m_TextureObjects.resize(newSize);
	}
	if (newSize * NumTextures > dev_TextureData.size())
	{
		dev_TextureData.resize(newSize * NumTextures);
	}

	GPUTexturesCompact gpuTextures{};

	//0 DIFFUSE > 1 NORMAL > 2 SPECULAR > 3 GLOSSINESS
	for (int i{}; i < NumTextures; ++i)
	{
		GPUTextureCompact* gpuTexture;
		switch (i)
		{
		case 0:
			gpuTexture = &gpuTextures.Diff;
			break;
		case 1:
			gpuTexture = &gpuTextures.Norm;
			break;
		case 2:
			gpuTexture = &gpuTextures.Spec;
			break;
		case 3:
			gpuTexture = &gpuTextures.Gloss;
			break;
		}
		SDL_Surface* pSurface = IMG_Load(texturePaths[i].c_str());
		if (pSurface)
		{
			const unsigned int width = pSurface->w;
			const unsigned int height = pSurface->h;
			const unsigned int* pixels = (unsigned int*)pSurface->pixels;
			const int bpp = pSurface->format->BytesPerPixel;
			//const size_t sizeInBytes = width * height * bpp;
			const size_t textureIdx = meshIdx * NumTextures + i;

			//copy texture data to device
			CheckErrorCuda(hipFree(dev_TextureData[textureIdx]));
			size_t pitch{};
			CheckErrorCuda(hipMallocPitch((void**)&dev_TextureData[textureIdx], &pitch, width * bpp, height)); //2D array
			CheckErrorCuda(hipMemcpy2D(dev_TextureData[textureIdx], pitch, pixels, pitch, width * bpp, height, hipMemcpyHostToDevice));

			//hipChannelFormatDesc formatDesc = hipCreateChannelDesc<unsigned int>();

			hipResourceDesc resDesc{};
			resDesc.resType = hipResourceTypePitch2D;
			resDesc.res.pitch2D.devPtr = dev_TextureData[textureIdx];
			resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
			resDesc.res.pitch2D.desc.x = pSurface->format->BitsPerPixel;
			resDesc.res.pitch2D.width = width;
			resDesc.res.pitch2D.height = height;

			hipTextureDesc texDesc{};
			texDesc.normalizedCoords = true; //able to sample texture with normalized uv coordinates
			texDesc.filterMode = hipFilterModePoint; //linear only supports float (and double) type
			texDesc.readMode = hipReadModeElementType;

			hipTextureObject_t dev_TextureObject{};
			CheckErrorCuda(hipCreateTextureObject(&dev_TextureObject, &resDesc, &texDesc, nullptr));

			/*{
				hipArray* dev_array;
				hipChannelFormatDesc formatDesc{};
				formatDesc.f = hipChannelFormatKindUnsigned;
				formatDesc.x = 32;

				hipMallocArray(&dev_array, &formatDesc, width, height, hipArrayTextureGather); //2D array
				hipMallocArray(&dev_array, &formatDesc, width * height, 1, hipArrayTextureGather); //1D array
				hipMemcpyToArray(dev_array, 0, 0, dev_TextureData[textureIdx], width * height * bpp, hipMemcpyHostToDevice);

				hipResourceDesc desc{};
				desc.resType = hipResourceTypeArray;
				desc.res.array.array = dev_array;

				hipTextureObject_t dev_TextureObject{};
				CheckErrorCuda(hipCreateTextureObject(&dev_TextureObject, &desc, &texDesc, nullptr));


				hipFreeArray(dev_array);
			}*/

			gpuTexture->dev_pTex = dev_TextureObject;
			gpuTextures.w = width;
			gpuTextures.h = height;
			gpuTexture->dev_TextureData = dev_TextureData[textureIdx];

			/*DEPRECATED
				//bind texture
				textureReference texRef{};
				texRef.normalized = false;
				texRef.channelDesc = hipCreateChannelDesc<unsigned int>();
				texRef.channelDesc.x = bpp * 8;
				texRef.channelDesc.f = hipChannelFormatKindUnsigned; //unsigned int

				size_t offset{};
				CUDA32bTexture2D texRef{}; //IN STATIC GLOBAL MEMORY!
				CheckErrorCuda(hipBindTexture2D(&offset, &texRef, dev_texData2D, &texRef.channelDesc, width, height, pitch * bpp));

				if (offset != 0)
				{
					std::cout << "Texture Offset : " << offset << '\n';
					return;
				}
			*/

			//free data
			SDL_FreeSurface(pSurface);
			//!DO NOT FREE TEXTURE DATA, as this will render the texture object invalid!
		}
	}
	//store textures
	m_TextureObjects[meshIdx] = gpuTextures;
}

CPU_CALLABLE
void CUDARenderer::FreeTextures()
{
	//destroy all texture objects
	for (const GPUTextures& textures : m_TextureObjects)
	{
		CheckErrorCuda(hipDestroyTextureObject(textures.Diff.dev_pTex));
		CheckErrorCuda(hipDestroyTextureObject(textures.Norm.dev_pTex));
		CheckErrorCuda(hipDestroyTextureObject(textures.Spec.dev_pTex));
		CheckErrorCuda(hipDestroyTextureObject(textures.Gloss.dev_pTex));
	}
	m_TextureObjects.clear();
	//free texture data
	for (unsigned int* dev_texData : dev_TextureData)
	{
		CheckErrorCuda(hipFree(dev_texData));
	}
	dev_TextureData.clear();
}

CPU_CALLABLE
void CUDARenderer::FreeMeshBuffers()
{
	for (size_t i{}; i < m_MeshIdentifiers.size(); ++i)
	{
		CheckErrorCuda(hipFree(dev_IVertexBuffer[i]));
		dev_IVertexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_IndexBuffer[i]));
		dev_IndexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_OVertexBuffer[i]));
		dev_OVertexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_Triangles[i]));
		dev_Triangles[i] = nullptr;
	}
	m_MeshIdentifiers.clear();
	dev_IVertexBuffer.clear();
	dev_IndexBuffer.clear();
	dev_OVertexBuffer.clear();
	dev_Triangles.clear();
}

CPU_CALLABLE
void CUDARenderer::FreeCUDADeviceBuffers()
{
	//Free buffers
	CheckErrorCuda(hipFree(dev_FrameBuffer));
	dev_FrameBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_PixelShadeBuffer));
	dev_PixelShadeBuffer = nullptr;

	CheckErrorCuda(hipHostFree(m_h_pFrameBuffer));
	m_h_pFrameBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_DepthBuffer));
	dev_DepthBuffer = nullptr;

	hipFree(dev_Mutex);
	dev_Mutex = nullptr;

	FreeMeshBuffers();
	FreeTextures();
}

CPU_CALLABLE
void CUDARenderer::UpdateCameraDataAsync(const FPoint3& camPos, const FMatrix4& viewProjectionMatrix)
{
	//Update CamPos
	size_t numBytes = sizeof(camPos);
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_RenderData_const), camPos.data, numBytes, 0, hipMemcpyHostToDevice));
	//Update ViewProjectionMatrix
	const size_t numBytesOffset = numBytes;
	numBytes = sizeof(viewProjectionMatrix);
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_RenderData_const), viewProjectionMatrix.data, numBytes, numBytesOffset, hipMemcpyHostToDevice));

	//CameraDataRaw cameraData{};
	//cameraData.cameraData.camPos = camPos;
	//cameraData.cameraData.viewProjectionMatrix = viewProjectionMatrix;
	//const size_t numBytes = sizeof(CameraData);
	//void* dev_ptr{};
	//CheckErrorCuda(hipGetSymbolAddress((void**)&dev_ptr, dev_RenderData_const));
	//CheckErrorCuda(hipMemcpyAsync(dev_ptr, cameraData.data, numBytes, hipMemcpyHostToDevice));
}

CPU_CALLABLE
void CUDARenderer::UpdateWorldMatrixDataAsync(const FMatrix4& worldMatrix)
{
	const size_t numBytes = sizeof(FMatrix4);
	const size_t numBytesOffset = sizeof(CameraData);
	CheckErrorCuda(hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_RenderData_const), worldMatrix.data, numBytes, numBytesOffset, hipMemcpyHostToDevice));

	//void* dev_ptr{};
	//CheckErrorCuda(hipGetSymbolAddress((void**)&dev_ptr, dev_RenderData_const));
	//CheckErrorCuda(hipMemcpyAsync(dev_ptr, worldMatrix.data, numBytes, hipMemcpyHostToDevice));
}

CPU_CALLABLE
int CUDARenderer::EnterValidRenderingState()
{
	//https://wiki.libsdl.org/SDL_LockSurface
	int state = SDL_LockSurface(m_WindowHelper.pBackBuffer); //Set up surface for directly accessing the pixels
	//Clear screen and reset buffers
	Clear();
	return state;
}

CPU_CALLABLE
void CUDARenderer::Present()
{
	//TODO: have Vertex Shader and Rasterizer run in parallel with cudamemcpy()
	const size_t size = m_WindowHelper.Width * m_WindowHelper.Height * sizeof(unsigned int);
	CheckErrorCuda(hipMemcpy(m_WindowHelper.pBackBufferPixels, dev_FrameBuffer, size, hipMemcpyDeviceToHost)); //We can directly read/write from pixelbuffer
	//memcpy(m_WindowHelper.pBackBufferPixels, m_WindowHelper.h_BackBufferPixels, size);
	SDL_UnlockSurface(m_WindowHelper.pBackBuffer); //Release a surface after directly accessing the pixels.
	SDL_BlitSurface(m_WindowHelper.pBackBuffer, 0, m_WindowHelper.pFrontBuffer, 0); //Copy the window surface to the screen.
	SDL_UpdateWindowSurface(m_WindowHelper.pWindow); //Update Window's surface
}

#pragma endregion

#pragma region GPU HELPER FUNCTIONS

GPU_CALLABLE static
void MultiplyMatrix(const float* matA, const float* matB, float* matC, unsigned int matSize)
{
	const unsigned int col = blockIdx.x * matSize + threadIdx.x;
	const unsigned int row = blockIdx.y * matSize + threadIdx.y;

	if (col < matSize && row < matSize)
	{
		float sum{};
		for (unsigned int i{}; i < matSize; ++i)
		{
			//matA goes from left to right in memory (row)
			//matB goes from top to bottom in memory (column)
			sum += matA[(threadIdx.y * matSize) + i] * matB[(i * matSize) + threadIdx.x];
		}
		__syncthreads();
		matC[col + row * matSize] = sum;
	}
}

GPU_CALLABLE static
void MultiplyMatrixVector(const float* mat, const float* p, float* output, unsigned int matSize)
{
	const unsigned int idx = threadIdx.x % 4;

	float sum{};
	for (unsigned int i{}; i < matSize; ++i)
	{
		//mat goes from left to right, every column per row
		//p goes from left to right, every column for 1 row
		sum += mat[(idx * matSize) + i] * p[threadIdx.x];
	}
	__syncthreads();
	output[threadIdx.x] = sum;
}

GPU_CALLABLE static
void CalculateIVertexPositionToNDC(float* iPositions, float* oPositions)
{
	//TODO: store members of vertices in separate arrays
	//		kernel does 1 operation at a time, with same shared memory
	//		e.g. load shared memory block with FPoint3 IVertex data
	//		calculate each element with each thread
	//		(store FMatrix4 in first row of shared memory, broadcast memory access since threads need same data)

	const float* WVPMatrix = reinterpret_cast<float*>(dev_WVPMatrix_const);

	//TODO: define size of dynamic shared memory by checking whether 
	//		amount of vertices >= 65535 (max number of blocks) / amount of vertices that are able to be processed in 1 CTA/block (8 here)
	//		this way we can have CTAs process double the amount of vertices without bank conflicts (slower, but good scaling method (for now))
	__shared__ float buffer[32]; //32 banks of 4 bytes per block

	//coalesced load into shared memory
	//buffer[threadIdx.x] = iPositions[threadIdx.x - (threadIdx.x / 3)]; //non-coalesced global read
	//if we skip the 4th value, we can have a strided access
	buffer[threadIdx.x] = iPositions[threadIdx.x]; //resorted to converting and storing IPositions into FPoint4

	__syncthreads();

	//calculate NDC (WVP * v.p)
	MultiplyMatrixVector(WVPMatrix, buffer, buffer, 4);

	__syncthreads();

	//divide xyz by w
	buffer[threadIdx.x] /= buffer[(threadIdx.x / 4) * 4 + 3]; //idx of w per 4 => 3 - 7 - 11 - 15 - ...

	__syncthreads();

	//coalesced store into global memory
	oPositions[threadIdx.x] = buffer[threadIdx.x];
}

GPU_CALLABLE static
void CalculateIVertexWorldPosition(float* iPositions, float* oWorldPos)
{
	const float* camPos = reinterpret_cast<float*>(dev_CameraPos_const);
	const float* worldMatrix = reinterpret_cast<float*>(dev_WorldMatrix_const);
	const float* rotationMatrix = reinterpret_cast<float*>(dev_RotationMatrix_const);

	__shared__ float buffer[32]; //32 banks of 4 bytes per block

	//coalesced load into shared memory
	buffer[threadIdx.x] = iPositions[threadIdx.x];

	__syncthreads();

	MultiplyMatrixVector(worldMatrix, buffer, buffer, 4); //calculate worldposition

	__syncthreads();

	oWorldPos[threadIdx.x] = buffer[threadIdx.x];

	__syncthreads();

	MultiplyMatrixVector(worldMatrix, buffer, buffer, 4); //calculate viewdirection

	MultiplyMatrixVector(rotationMatrix, buffer, buffer, 3); //calculate normal
	MultiplyMatrixVector(rotationMatrix, buffer, buffer, 3); //calculate tangent

	//TODO: copy uv and colour
}

//DEPRECATED
GPU_CALLABLE static
OVertex GetNDCVertex(const IVertex& __restrict__ iVertex)
{
	OVertex oVertex;

	const FPoint3& camPos = reinterpret_cast<FPoint3&>(dev_CameraPos_const);
	const FMatrix4& WVPMatrix = reinterpret_cast<FMatrix4&>(dev_WVPMatrix_const);
	const FMatrix4& worldMatrix = reinterpret_cast<FMatrix4&>(dev_WorldMatrix_const);
	const FMatrix3& rotationMatrix = reinterpret_cast<FMatrix3&>(dev_RotationMatrix_const);

	const FPoint3 worldPosition{ worldMatrix * FPoint4{ iVertex.p } };

	oVertex.p = WVPMatrix * FPoint4{ iVertex.p };
	oVertex.p.x /= oVertex.p.w;
	oVertex.p.y /= oVertex.p.w;
	oVertex.p.z /= oVertex.p.w;
	oVertex.uv = iVertex.uv;
	oVertex.n = FVector3{ rotationMatrix * iVertex.n };
	oVertex.tan = FVector3{ rotationMatrix * iVertex.tan };
	oVertex.vd = FVector3{ GetNormalized(worldPosition - camPos) };
	oVertex.c = iVertex.c;

	return oVertex;
}

GPU_CALLABLE static
float EdgeFunction(const FPoint2& v, const FVector2& edge, const FPoint2& pixel)
{
	// clockwise
	const FVector2 vertexToPixel{ pixel - v };
	return Cross(vertexToPixel, edge);
}

GPU_CALLABLE static
bool IsPixelInTriangle(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2, const FPoint2& pixel, float weights[3])
{
	const FVector2 edgeA{ v1.xy - v0.xy };
	const FVector2 edgeB{ v2.xy - v1.xy };
	const FVector2 edgeC{ v0.xy - v2.xy };
	// clockwise
	//const FVector2 edgeA{ v0 - v1 };
	//const FVector2 edgeB{ v1 - v2 };
	//const FVector2 edgeC{ v2 - v0 };
	// counter-clockwise

	{
		//// edgeA
		//FVector2 vertexToPixel{ pixel - v0 };
		//float cross = Cross(edgeA, vertexToPixel);
		//isInTriangle &= cross < 0.f;
		//// weight2 == positive cross of 'previous' edge, for v2 this is edgeA (COUNTER-CLOCKWISE)
		//weights[2] = cross / totalArea;

		//// edgeB
		//vertexToPixel = { pixel - v1 };
		//cross = Cross(edgeB, vertexToPixel);
		//isInTriangle &= cross < 0.f;
		//// weight1 (for v1 this is edgeB)
		//weights[1] = cross / totalArea;

		//// edgeC
		//vertexToPixel = { pixel - v2 };
		//cross = Cross(edgeC, vertexToPixel);
		//isInTriangle &= cross < 0.f;
		//// weight0 (for v0 this is edgeC)
		//weights[0] = cross / totalArea;

		//weights == inverted negative cross of 'previous' edge
		//weights[0] = Cross(-vertexToPixel, edgeC) / totalArea;
		//weights[1] = Cross(-vertexToPixel, edgeB) / totalArea;
		//weights[2] = Cross(-vertexToPixel, edgeA) / totalArea;
		// gives positive results because counter-clockwise
		//const float total = weights[0] + weights[1] + weights[2]; // total result equals 1
	}

	weights[2] = EdgeFunction(v0.xy, edgeA, pixel);
	weights[0] = EdgeFunction(v1.xy, edgeB, pixel);
	weights[1] = EdgeFunction(v2.xy, edgeC, pixel);

	return weights[0] >= 0.f && weights[1] >= 0.f && weights[2] >= 0.f;
}

GPU_CALLABLE static
bool IsPixelInTriangle(const RasterTriangle& triangle, const FPoint2& pixel, float weights[3])
{
	return IsPixelInTriangle(triangle.v0, triangle.v1, triangle.v2, pixel, weights);
}

GPU_CALLABLE static
//UNUSED
void ModifyPixelShadeBuffer()
{

}

GPU_CALLABLE static
//DEPRECATED
//This will "block" the current thread into a while loop until depthtest is complete
bool IsDepthTestSucceeded(float dev_DepthBuffer[], int dev_Mutex[], const size_t pixelIdx, float zInterpolated)
{
	//TODO: shared memory
	
	//Depth Test with correct depth interpolation
	if (zInterpolated < 0 || zInterpolated > 1.f)
		return false;
	
	//Update depthbuffer atomically
	bool isDepthTestSucceeded = false;
	bool isDone = false;
	do
	{
		isDone = (atomicCAS(&dev_Mutex[pixelIdx], 0, 1) == 0);
		if (isDone)
		{
			//critical section
			if (zInterpolated > dev_DepthBuffer[pixelIdx]) //DEPTH BUFFER INVERTED INTERPRETATION
			{
				dev_DepthBuffer[pixelIdx] = zInterpolated;
				//TODO: (atomically) write to pixelshaderbuffer
				isDepthTestSucceeded = true;
			}
			dev_Mutex[pixelIdx] = 0;
			//end of critical section
		}
	} while (!isDone);
	return isDepthTestSucceeded;
	//atomicCAS
	/*
	//int atomicCAS(int* address, int compare, int val);
	//reads the 16 - bit, 32 - bit or 64 - bit word old located at the address address in global or shared memory, 
	//computes(old == compare ? val : old), and stores the result back to memory at the same address.
	//These three operations are performed in one atomic transaction.The function returns old(Compare And Swap).
	*/
}

GPU_CALLABLE static
bool IsAllXOutsideFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return	(v0.x < -1.f && v1.x < -1.f && v2.x < -1.f) ||
		(v0.x > 1.f && v1.x > 1.f && v2.x > 1.f);
}

GPU_CALLABLE static
bool IsAllYOutsideFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return	(v0.y < -1.f && v1.y < -1.f && v2.y < -1.f) ||
		(v0.y > 1.f && v1.y > 1.f && v2.y > 1.f);
}

GPU_CALLABLE static
bool IsAllZOutsideFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return	(v0.z < 0.f && v1.z < 0.f && v2.z < 0.f) ||
		(v0.z > 1.f && v1.z > 1.f && v2.z > 1.f);
}

GPU_CALLABLE static
bool IsTriangleVisible(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	// Solution to FrustumCulling bug
	//	   if (all x values are < -1.f or > 1.f) AT ONCE, cull
	//	|| if (all y values are < -1.f or > 1.f) AT ONCE, cull
	//	|| if (all z values are < 0.f or > 1.f) AT ONCE, cull
	return(!IsAllXOutsideFrustum(v0, v1, v0)
		&& !IsAllYOutsideFrustum(v0, v1, v0)
		&& !IsAllZOutsideFrustum(v0, v1, v0));
}

GPU_CALLABLE static
bool IsVertexInFrustum(const FPoint4& NDC)
{
	return!((NDC.x < -1.f || NDC.x > 1.f) || 
			(NDC.y < -1.f || NDC.y > 1.f) || 
			(NDC.z < 0.f || NDC.z > 1.f));
}

GPU_CALLABLE static
bool IsTriangleInFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return(IsVertexInFrustum(v0)
		|| IsVertexInFrustum(v1)
		|| IsVertexInFrustum(v2));
	//TODO: bug, triangles gets culled when zoomed in, aka all 3 vertices are outside of frustum
}

GPU_CALLABLE static
void NDCToScreenSpace(FPoint4& v0, FPoint4& v1, FPoint4& v2, const unsigned int width, const unsigned int height)
{
	v0.x = ((v0.x + 1) / 2) * width;
	v0.y = ((1 - v0.y) / 2) * height;
	v1.x = ((v1.x + 1) / 2) * width;
	v1.y = ((1 - v1.y) / 2) * height;
	v2.x = ((v2.x + 1) / 2) * width;
	v2.y = ((1 - v2.y) / 2) * height;
}

GPU_CALLABLE static
BoundingBox GetBoundingBox(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2, const unsigned int width, const unsigned int height)
{
	BoundingBox bb;
	bb.xMin = (short)GetMinElement(v0.x, v1.x, v2.x) - 1; // xMin
	bb.yMin = (short)GetMinElement(v0.y, v1.y, v2.y) - 1; // yMin
	bb.xMax = (short)GetMaxElement(v0.x, v1.x, v2.x) + 1; // xMax
	bb.yMax = (short)GetMaxElement(v0.y, v1.y, v2.y) + 1; // yMax

	if (bb.xMin < 0) bb.xMin = 0; //clamp minX to Left of screen
	if (bb.yMin < 0) bb.yMin = 0; //clamp minY to Bottom of screen
	if (bb.xMax > width) bb.xMax = width; //clamp maxX to Right of screen
	if (bb.yMax > height) bb.yMax = height; //clamp maxY to Top of screen

	return bb;
}

GPU_CALLABLE static
bool IsAllXOutsideFrustum(const RasterTriangle& triangle)
{
	return IsAllXOutsideFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsAllYOutsideFrustum(const RasterTriangle& triangle)
{
	return IsAllYOutsideFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsAllZOutsideFrustum(const RasterTriangle& triangle)
{
	return IsAllZOutsideFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsTriangleVisible(const RasterTriangle& triangle)
{
	return IsTriangleVisible(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsTriangleInFrustum(const RasterTriangle& triangle)
{
	return IsTriangleInFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
void NDCToScreenSpace(RasterTriangle& triangle, const unsigned int width, const unsigned int height)
{
	NDCToScreenSpace(triangle.v0, triangle.v1, triangle.v2, width, height);
}

GPU_CALLABLE static
BoundingBox GetBoundingBox(const RasterTriangle& triangle, const unsigned int width, const unsigned int height)
{
	return GetBoundingBox(triangle.v0, triangle.v1, triangle.v2, width, height);
}

GPU_CALLABLE GPU_INLINE static
RGBColor ShadePixel(const PixelShade& pixelShade, SampleState sampleState, bool isDepthColour)
{
	RGBColor finalColour{};
	if (isDepthColour)
	{
		//TODO: replace with depthbuffer kernel call?
		finalColour = RGBColor{ Remap(pixelShade.zInterpolated, 0.985f, 1.f), 0.f, 0.f }; // depth colour
	}
	else
	{
		//global settings
		bool isFlipGreenChannel = false;
		const RGBColor ambientColour{ 0.05f, 0.05f, 0.05f };
		const FVector3 lightDirection = { 0.577f, -0.577f, -0.577f };
		const float lightIntensity = 7.0f;

		// texture sampling
		const GPUTexturesCompact& textures = pixelShade.textures;
		if (textures.Diff.dev_pTex != 0)
		{
			const RGBColor diffuseSample = GPUTextureSampler::Sample(textures.Diff, textures.w, textures.h, pixelShade.uv, sampleState);

			if (textures.Norm.dev_pTex != 0)
			{
				const RGBColor normalSample = GPUTextureSampler::Sample(textures.Norm, textures.w, textures.h, pixelShade.uv, sampleState);

				// normal mapping
				FVector3 binormal = Cross(pixelShade.tan, pixelShade.n);
				if (isFlipGreenChannel)
					binormal = -binormal;
				const FMatrix3 tangentSpaceAxis{ pixelShade.tan, binormal, pixelShade.n };

				FVector3 finalNormal{ 2.f * normalSample.r - 1.f, 2.f * normalSample.g - 1.f, 2.f * normalSample.b - 1.f };
				finalNormal = tangentSpaceAxis * finalNormal;

				// light calculations
				float observedArea{ Dot(-finalNormal, lightDirection) };
				Clamp(observedArea, 0.f, observedArea);
				observedArea /= (float)PI;
				observedArea *= lightIntensity;
				const RGBColor diffuseColour = diffuseSample * observedArea;

				if (textures.Spec.dev_pTex != 0 && textures.Gloss.dev_pTex != 0)
				{
					const RGBColor specularSample = GPUTextureSampler::Sample(textures.Spec, textures.w, textures.h, pixelShade.uv, sampleState);
					const RGBColor glossSample = GPUTextureSampler::Sample(textures.Gloss, textures.w, textures.h, pixelShade.uv, sampleState);

					// phong specular
					const FVector3 reflectV{ Reflect(lightDirection, finalNormal) };
					float angle{ Dot(reflectV, pixelShade.vd) };
					Clamp(angle, 0.f, 1.f);
					const float shininess = 25.f;
					angle = powf(angle, glossSample.r * shininess);
					const RGBColor specularColour = specularSample * angle;

					// final
					finalColour = ambientColour + diffuseColour + specularColour;
					finalColour.ClampColor();
				}
				else
				{
					finalColour = diffuseColour;
				}
			}
			else
			{
				finalColour = diffuseSample;
			}
		}
		else
		{
			finalColour = GetRGBColor_SDL(pixelShade.colour);
		}
	}
	return finalColour;
}

#pragma endregion

#pragma region KERNELS
//Kernel launch params:	numBlocks, numThreadsPerBlock, numSharedMemoryBytes, stream

GPU_KERNEL
void ResetDepthBufferKernel(int* dev_DepthBuffer, int value, const unsigned int width, const unsigned int height)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_DepthBuffer[pixelIdx] = value;
	}
}

GPU_KERNEL
void ClearFrameBufferKernel(unsigned int* dev_FrameBuffer, const unsigned int width, const unsigned int height, unsigned int colour)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_FrameBuffer[pixelIdx] = colour;
	}
}

GPU_KERNEL
void ClearPixelShadeBufferKernel(PixelShade* dev_PixelShadeBuffer, const unsigned int size)
{
	//every thread sets 1 WORD of data
	const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{
		reinterpret_cast<unsigned int*>(dev_PixelShadeBuffer)[idx] = 0;
	}
}

GPU_KERNEL
void Clear()
{
	//TODO: clear depthbuffer, framebuffer and pixelshadebuffer
}

GPU_KERNEL
void VertexShaderKernel(const IVertex_Point4* __restrict__ dev_IVertices, OVertex* dev_OVertices, const size_t numVertices)
{
	//sizeof(IVertex_Point4) == //60 bytes, 15 floats
	//sizeof(OVertex) == 72 bytes, 18 floats
	__shared__ float iPositionsBuffer[32];
	const unsigned int vertexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (vertexIdx < numVertices)
	{
		//strided copy
		const unsigned int stridedIdx = threadIdx.x % 4 + (threadIdx.x / 4) * sizeof(IVertex_Point4); //== stride
		iPositionsBuffer[vertexIdx] = reinterpret_cast<const float*>(dev_IVertices)[stridedIdx];

		__syncthreads();

		CalculateIVertexPositionToNDC(iPositionsBuffer, dev_OPositions);

		__syncthreads();

		reinterpret_cast<float*>(&dev_OPositions)[vertexIdx] = iPositionsBuffer[vertexIdx];
	}
}

GPU_KERNEL
void TriangleAssemblerKernel(TriangleIdx* dev_Triangles, const unsigned int* __restrict__ const dev_IndexBuffer, const size_t numIndices, 
	OVertex* dev_OVertices, const PrimitiveTopology pt)
{
	//'talk about naming gore, eh?
	const unsigned int indexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (pt == PrimitiveTopology::TriangleList)
	{
		const unsigned int correctedIdx = (indexIdx * 3);
		if (correctedIdx < numIndices)
		{
			//Unnecessary for TriangleLists
			//TriangleIdx triangle;
			//triangle.idx0 = dev_IndexBuffer[correctedIdx];
			//triangle.idx1 = dev_IndexBuffer[correctedIdx + 1];
			//triangle.idx2 = dev_IndexBuffer[correctedIdx + 2];
			//triangle.isCulled = false;
			//dev_Triangles[indexIdx] = triangle;

			memcpy(&dev_Triangles[indexIdx], &dev_IndexBuffer[correctedIdx], sizeof(TriangleIdx));
			//atomically increment visible triangle count
			//atomicAdd(dev_VisibleNumTriangles, 1);
		}
	}
	else //if (pt == PrimitiveTopology::TriangleStrip)
	{
		if (indexIdx < numIndices - 2)
		{
			//Necessary for TriangleStrips
			TriangleIdx triangle;
			const bool isOdd = (indexIdx % 2);
			//triangle.idx0 = dev_IndexBuffer[indexIdx];
			//if (isOdd)
			//{
			//	triangle.idx1 = dev_IndexBuffer[indexIdx + 2];
			//	triangle.idx2 = dev_IndexBuffer[indexIdx + 1];
			//}
			//else
			//{
			//	triangle.idx1 = dev_IndexBuffer[indexIdx + 1];
			//	triangle.idx2 = dev_IndexBuffer[indexIdx + 2];
			//}
			//triangle.isCulled = false;

			memcpy(&triangle, &dev_IndexBuffer[indexIdx], sizeof(TriangleIdx));
			if (isOdd)
			{
				const unsigned int origIdx1 = triangle.idx1;
				triangle.idx1 = triangle.idx2;
				triangle.idx2 = origIdx1;
			}
			dev_Triangles[indexIdx] = triangle;

			//atomically increment visible triangle count
			//atomicAdd(dev_VisibleNumTriangles, 1);
		}
	}
}

/*
GPU_CALLABLE
void RasterizeTriangle()
{
	RasterTriangle rasterTriangle;
	rasterTriangle.v0 = v0.p;
	rasterTriangle.v1 = v1.p;
	rasterTriangle.v2 = v2.p;

	if (!IsTriangleVisible(rasterTriangle))
	{
		return;
	}

	NDCToScreenSpace(rasterTriangle, width, height);
	const BoundingBox bb = GetBoundingBox(rasterTriangle, width, height);
	//Rasterize Screenspace triangle

	const float v0InvDepth = 1.f / rasterTriangle.v0.w;
	const float v1InvDepth = 1.f / rasterTriangle.v1.w;
	const float v2InvDepth = 1.f / rasterTriangle.v2.w;

	//TODO: 1 thread per triangle is bad for performance, use binning
	//Loop over all pixels in bounding box
	for (unsigned short y = bb.yMin; y < bb.yMax; ++y)
	{
		for (unsigned short x = bb.xMin; x < bb.xMax; ++x)
		{
			const FPoint2 pixel{ float(x), float(y) };
			float weights[3];
			if (IsPixelInTriangle(rasterTriangle, pixel, weights))
			{
				const float totalArea = abs(Cross(rasterTriangle.v0.xy - rasterTriangle.v1.xy, rasterTriangle.v0.xy - rasterTriangle.v2.xy));
				weights[0] /= totalArea;
				weights[1] /= totalArea;
				weights[2] /= totalArea;

				const size_t pixelIdx = x + y * width;
				const float zInterpolated = (weights[0] * v0.p.z) + (weights[1] * v1.p.z) + (weights[2] * v2.p.z);

				//peform early depth test
				if (zInterpolated < 0 || zInterpolated > 1.f)
					continue;

				const float wInterpolated = 1.f / (v0InvDepth * weights[0] + v1InvDepth * weights[1] + v2InvDepth * weights[2]);

				//create pixelshade object (== fragment)
				PixelShade pixelShade;

				//depthbuffer visualisation
				pixelShade.zInterpolated = zInterpolated;
				pixelShade.wInterpolated = wInterpolated;

				//uv
				new (&pixelShade.uv) FVector2{
					weights[0] * (v0.uv.x * v0InvDepth) + weights[1] * (v1.uv.x * v1InvDepth) + weights[2] * (v2.uv.x * v2InvDepth),
					weights[0] * (v0.uv.y * v0InvDepth) + weights[1] * (v1.uv.y * v1InvDepth) + weights[2] * (v2.uv.y * v2InvDepth) };
				pixelShade.uv *= wInterpolated;

				//normal
				new (&pixelShade.n) FVector3{
						weights[0] * (v0.n.x * v0InvDepth) + weights[1] * (v1.n.x * v1InvDepth) + weights[2] * (v2.n.x * v2InvDepth),
						weights[0] * (v0.n.y * v0InvDepth) + weights[1] * (v1.n.y * v1InvDepth) + weights[2] * (v2.n.y * v2InvDepth),
						weights[0] * (v0.n.z * v0InvDepth) + weights[1] * (v1.n.z * v1InvDepth) + weights[2] * (v2.n.z * v2InvDepth) };
				pixelShade.n *= wInterpolated;

				//tangent
				new (&pixelShade.tan) FVector3{
					weights[0] * (v0.tan.x * v0InvDepth) + weights[1] * (v1.tan.x * v1InvDepth) + weights[2] * (v2.tan.x * v2InvDepth),
					weights[0] * (v0.tan.y * v0InvDepth) + weights[1] * (v1.tan.y * v1InvDepth) + weights[2] * (v2.tan.y * v2InvDepth),
					weights[0] * (v0.tan.z * v0InvDepth) + weights[1] * (v1.tan.z * v1InvDepth) + weights[2] * (v2.tan.z * v2InvDepth) };

				//view direction
				new (&pixelShade.vd) FVector3{
					weights[0] * (v0.vd.x * v0InvDepth) + weights[1] * (v1.vd.x * v1InvDepth) + weights[2] * (v2.vd.x * v2InvDepth),
					weights[0] * (v0.vd.y * v0InvDepth) + weights[1] * (v1.vd.y * v1InvDepth) + weights[2] * (v2.vd.y * v2InvDepth),
					weights[0] * (v0.vd.z * v0InvDepth) + weights[1] * (v1.vd.z * v1InvDepth) + weights[2] * (v2.vd.z * v2InvDepth) };
				Normalize(pixelShade.vd);

				//colour
				const RGBColor interpolatedColour{
					weights[0] * v0.c.r + weights[1] * v1.c.r + weights[2] * v2.c.r,
					weights[0] * v0.c.g + weights[1] * v1.c.g + weights[2] * v2.c.g,
					weights[0] * v0.c.b + weights[1] * v1.c.b + weights[2] * v2.c.b };
				pixelShade.colour = GetRGBA_SDL(interpolatedColour).colour;

				//store textures
				pixelShade.textures = textures;

				//multiplying z value by a INT_MAX because atomicCAS only accepts ints
				const int scaledZ = zInterpolated * INT_MAX;

				//Perform atomic depth test
				bool isDone = false;
				do
				{
					isDone = (atomicCAS(&dev_Mutex[pixelIdx], 0, 1) == 0);
					if (isDone)
					{
						//critical section
						if (scaledZ < dev_DepthBuffer[pixelIdx])
						{
							//update depthbuffer
							dev_DepthBuffer[pixelIdx] = scaledZ;
							//modify pixelshadebuffer
							dev_PixelShadeBuffer[pixelIdx] = pixelShade;
						}
						dev_Mutex[pixelIdx] = 0;
						//end of critical section
					}
				} while (!isDone);
			}
		}
	}
}
*/

GPU_KERNEL
void RasterizerKernel(const TriangleIdx* __restrict__ const dev_Triangles, const OVertex* __restrict__ const dev_OVertices, const size_t numTriangles,
	PixelShade* dev_PixelShadeBuffer, int* dev_DepthBuffer, int* dev_Mutex, GPUTexturesCompact textures,
	const FVector3 camFwd, const CullingMode cm, const unsigned int width, const unsigned int height)
{
	//TODO: use shared memory, then coalesced copy
	//e.g. single bin buffer in single shared memory
	//extern GPU_SHARED_MEMORY Triangle triangles[];
	//TODO: use binning, each bin their AABBs (and checks) (bin rasterizer)

	const unsigned int triangleIndex = threadIdx.x + blockIdx.x * blockDim.x;
	//if (triangleIndex < numTriangles)
	//{
	//	triangles[threadIdx.x] = dev_Triangles[triangleIndex];
	//}
	//__syncthreads();
	//https://stackoverflow.com/questions/6563261/how-to-use-coalesced-memory-access

	//Every thread processes 1 single triangle for now
	if (!(triangleIndex < numTriangles))
		return;

	const TriangleIdx triangleIdx = dev_Triangles[triangleIndex];

	//if (triangleIdx.isCulled)
	//	return;

	//constexpr size_t a = sizeof(OVertex) * 3;
	//constexpr size_t b = sizeof(FPoint4) * 3;

	OVertex v0 = dev_OVertices[triangleIdx.idx0];
	OVertex v1 = dev_OVertices[triangleIdx.idx1];
	OVertex v2 = dev_OVertices[triangleIdx.idx2];

	bool isDoubleSidedRendering = false;

	//is triangle visible according to cullingmode?
	if (cm == CullingMode::BackFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ v1.p - v0.p }, FVector3{ v2.p - v0.p }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue <= 0.f)
		{
			if (isDoubleSidedRendering)
			{
				OVertex origV1 = v1;
				v1 = v2;
				v2 = origV1;
			}
			else
			{
				return; //cull triangle
			}
		}
	}
	else if (cm == CullingMode::FrontFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ v1.p - v0.p }, FVector3{ v2.p - v0.p }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue >= 0.f)
		{
			if (isDoubleSidedRendering)
			{
				OVertex origV1 = v1;
				v1 = v2;
				v2 = origV1;
			}
			else
			{
				return; //cull triangle
			}
		}
	}
	//else if (cm == CullingMode::NoCulling)
	//{
	//}

	//RasterTriangle rasterTriangle;
	//rasterTriangle.v0 = v0.p;
	//rasterTriangle.v1 = v1.p;
	//rasterTriangle.v2 = v2.p;

	if (!IsTriangleVisible(v0.p, v1.p, v2.p))
	{
		return;
	}

	NDCToScreenSpace(v0.p, v1.p, v2.p, width, height);
	const BoundingBox bb = GetBoundingBox(v0.p, v1.p, v2.p, width, height);
	//Rasterize Screenspace triangle

	const float v0InvDepth = 1.f / v0.p.w;
	const float v1InvDepth = 1.f / v1.p.w;
	const float v2InvDepth = 1.f / v2.p.w;

	//TODO: 1 thread per triangle is bad for performance, use binning
	//Loop over all pixels in bounding box
	for (unsigned short y = bb.yMin; y < bb.yMax; ++y)
	{
		for (unsigned short x = bb.xMin; x < bb.xMax; ++x)
		{
			const FPoint2 pixel{ float(x), float(y) };
			float weights[3];
			if (IsPixelInTriangle(v0.p, v1.p, v2.p, pixel, weights))
			{
				const float totalArea = abs(Cross(v0.p.xy - v1.p.xy, v0.p.xy - v2.p.xy));
				weights[0] /= totalArea;
				weights[1] /= totalArea;
				weights[2] /= totalArea;

				const size_t pixelIdx = x + y * width;
				const float zInterpolated = (weights[0] * v0.p.z) + (weights[1] * v1.p.z) + (weights[2] * v2.p.z);

				//peform early depth test
				if (zInterpolated < 0 || zInterpolated > 1.f)
					continue;

				const float wInterpolated = 1.f / (v0InvDepth * weights[0] + v1InvDepth * weights[1] + v2InvDepth * weights[2]);

				//create pixelshade object (== fragment)
				PixelShade pixelShade;

				//depthbuffer visualisation
				pixelShade.zInterpolated = zInterpolated;
				pixelShade.wInterpolated = wInterpolated;

				//uv
				pixelShade.uv.x = weights[0] * (v0.uv.x * v0InvDepth) + weights[1] * (v1.uv.x * v1InvDepth) + weights[2] * (v2.uv.x * v2InvDepth);
				pixelShade.uv.y = weights[0] * (v0.uv.y * v0InvDepth) + weights[1] * (v1.uv.y * v1InvDepth) + weights[2] * (v2.uv.y * v2InvDepth);
				pixelShade.uv *= wInterpolated;

				//normal
				pixelShade.n.x = weights[0] * (v0.n.x * v0InvDepth) + weights[1] * (v1.n.x * v1InvDepth) + weights[2] * (v2.n.x * v2InvDepth);
				pixelShade.n.y = weights[0] * (v0.n.y * v0InvDepth) + weights[1] * (v1.n.y * v1InvDepth) + weights[2] * (v2.n.y * v2InvDepth);
				pixelShade.n.z = weights[0] * (v0.n.z * v0InvDepth) + weights[1] * (v1.n.z * v1InvDepth) + weights[2] * (v2.n.z * v2InvDepth);
				pixelShade.n *= wInterpolated;

				//tangent
				pixelShade.tan.x = weights[0] * (v0.tan.x * v0InvDepth) + weights[1] * (v1.tan.x * v1InvDepth) + weights[2] * (v2.tan.x * v2InvDepth);
				pixelShade.tan.y = weights[0] * (v0.tan.y * v0InvDepth) + weights[1] * (v1.tan.y * v1InvDepth) + weights[2] * (v2.tan.y * v2InvDepth);
				pixelShade.tan.z = weights[0] * (v0.tan.z * v0InvDepth) + weights[1] * (v1.tan.z * v1InvDepth) + weights[2] * (v2.tan.z * v2InvDepth);

				//view direction
				pixelShade.vd.x = weights[0] * (v0.vd.x * v0InvDepth) + weights[1] * (v1.vd.x * v1InvDepth) + weights[2] * (v2.vd.x * v2InvDepth);
				pixelShade.vd.y = weights[0] * (v0.vd.y * v0InvDepth) + weights[1] * (v1.vd.y * v1InvDepth) + weights[2] * (v2.vd.y * v2InvDepth);
				pixelShade.vd.z = weights[0] * (v0.vd.z * v0InvDepth) + weights[1] * (v1.vd.z * v1InvDepth) + weights[2] * (v2.vd.z * v2InvDepth);
				Normalize(pixelShade.vd);

				//colour
				const RGBColor interpolatedColour{
					weights[0] * v0.c.r + weights[1] * v1.c.r + weights[2] * v2.c.r,
					weights[0] * v0.c.g + weights[1] * v1.c.g + weights[2] * v2.c.g,
					weights[0] * v0.c.b + weights[1] * v1.c.b + weights[2] * v2.c.b };
				pixelShade.colour = GetRGBA_SDL(interpolatedColour).colour;

				//store textures
				pixelShade.textures = textures;
				
				//TODO: store texture w and height in const memory?

				//multiplying z value by a INT_MAX because atomicCAS only accepts ints
				const int scaledZ = zInterpolated * INT_MAX;
				
				//Perform atomic depth test
				bool isDone = false;
				do
				{
					isDone = (atomicCAS(&dev_Mutex[pixelIdx], 0, 1) == 0);
					if (isDone)
					{
						//critical section
						if (scaledZ < dev_DepthBuffer[pixelIdx])
						{
							//update depthbuffer
							dev_DepthBuffer[pixelIdx] = scaledZ;
							//modify pixelshadebuffer
							dev_PixelShadeBuffer[pixelIdx] = pixelShade;
						}
						dev_Mutex[pixelIdx] = 0;
						//end of critical section
					}
				} while (!isDone);
			}
		}
	}
}

GPU_KERNEL
void PixelShaderKernel(unsigned int* dev_FrameBuffer, const PixelShade* __restrict__ const dev_PixelShadeBuffer,
	SampleState sampleState, bool isDepthColour, const unsigned int width, const unsigned int height)
{
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		PixelShade pixelShade = dev_PixelShadeBuffer[pixelIdx];
		const RGBColor colour = ShadePixel(pixelShade, sampleState, isDepthColour);
		RGBA rgba{ colour };
		dev_FrameBuffer[pixelIdx] = rgba.colour;
	}
}

GPU_KERNEL
void TextureTestKernel(unsigned int* dev_FrameBuffer, GPUTexture texture, const unsigned int width, const unsigned int height)
{
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		float u = float(x) / width;
		float v = float(y) / height;
		//u *= texture.w;
		//v *= texture.h;
		//float uC = Clamp(float(x), 0.f, (float)texture.w);
		//float vC = Clamp(float(y), 0.f, (float)texture.h);
		//float sampleIdx = u + v * texture.w;
		//unsigned int sample = tex1Dfetch<unsigned int>(texture.dev_pTex, (int)sampleIdx);
		//remap uv's to stretch towards the window's dimensions
		unsigned int sample = tex2D<unsigned int>(texture.dev_pTex, u, v);
		RGBA rgba = sample;
		unsigned char b = rgba.values.b;
		rgba.values.b = rgba.values.r;
		rgba.values.r = b;
		dev_FrameBuffer[pixelIdx] = rgba.colour;
	}
}

GPU_KERNEL
void DrawTextureGlobalKernel(unsigned int* dev_FrameBuffer, GPUTexture texture, bool isStretchedToWindow,
	SampleState sampleState, const unsigned int width, const unsigned int height)
{
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		//remap uv's to stretch towards the window's dimensions
		FVector2 uv;
		uv.x = float(x);
		uv.y = float(y);
		if (isStretchedToWindow)
		{
			uv.x /= width;
			uv.y /= height;
		}
		else
		{
			uv.x /= texture.w;
			uv.y /= texture.h;
		}
		RGBColor sample = GPUTextureSampler::Sample(texture, uv, sampleState);
		RGBA rgba = sample;
		dev_FrameBuffer[pixelIdx] = rgba.colour;
	}
}

#pragma endregion

#pragma region KERNEL LAUNCHERS

CPU_CALLABLE
void CUDARenderer::Clear(const RGBColor& colour)
{
	{
		const dim3 numThreadsPerBlock{ 16, 16 };
		const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
		const int depthBufferResetValue = INT_MAX;
		ResetDepthBufferKernel << <numBlocks, numThreadsPerBlock >> >
			(dev_DepthBuffer, depthBufferResetValue, m_WindowHelper.Width, m_WindowHelper.Height);
	}

	{
		//const RGBA rgba{ colour };
		//ClearFrameBufferKernel<<<numBlocks, numThreadsPerBlock>>>
		//	(dev_FrameBuffer, m_WindowHelper.Width, m_WindowHelper.Height, rgba.colour);
	}

	{
		const size_t sizeInWords = m_WindowHelper.Width * m_WindowHelper.Height * (sizeof(PixelShade) / 4);
		const unsigned int numThreadsPerBlock = 1024;
		const unsigned int numBlocks = (unsigned int)(sizeInWords + numThreadsPerBlock - 1) / numThreadsPerBlock;
		ClearPixelShadeBufferKernel<<<numBlocks, numThreadsPerBlock>>>
			(dev_PixelShadeBuffer, sizeInWords);
	}

	{
		const unsigned int size = m_WindowHelper.Width * m_WindowHelper.Height * sizeof(int);
		//CheckErrorCuda(hipMemset(dev_FrameBuffer, UCHAR_MAX, size));
		CheckErrorCuda(hipMemsetAsync(dev_Mutex, 0, size));
		//CheckErrorCuda(hipMemsetAsync(dev_PixelShadeBuffer, 0, m_WindowHelper.Width * m_WindowHelper.Height * sizeof(PixelShade)));
	}
}

CPU_CALLABLE
void CUDARenderer::VertexShader(const MeshIdentifier& mi)
{
	const size_t numVertices = mi.pMesh->GetVertexAmount();
	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = (unsigned int)(numVertices + numThreadsPerBlock - 1) / numThreadsPerBlock;
	VertexShaderKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_IVertexBuffer[mi.Idx], dev_OVertexBuffer[mi.Idx], numVertices);
}

CPU_CALLABLE
void CUDARenderer::TriangleAssembler(MeshIdentifier& mi)
{
	//unsigned int* dev_VisibleNumTriangles;
	//CheckErrorCuda(hipMalloc((void**)&dev_VisibleNumTriangles, sizeof(unsigned int)));

	const size_t numIndices = mi.pMesh->GetIndexAmount();
	const PrimitiveTopology topology = mi.pMesh->GetTopology();

	const unsigned int numThreadsPerBlock = 256;
	unsigned int numBlocks = ((unsigned int)numIndices + numThreadsPerBlock - 1) / numThreadsPerBlock;;
	if (topology == PrimitiveTopology::TriangleList)
		numBlocks = ((unsigned int)numIndices / 3 + numThreadsPerBlock - 1) / numThreadsPerBlock;
	//OCCUPANCY does strange things (tiny performance loss on current setup)
	//For TriangleStrips it would waste 2 threads at max, not too big of an issue here
	//else if (topology == PrimitiveTopology::TriangleStrip)
		//numBlocks = ((unsigned int)numIndices - 2 + numThreadsPerBlock - 1) / numThreadsPerBlock;
	TriangleAssemblerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], dev_IndexBuffer[mi.Idx], numIndices, 
		dev_OVertexBuffer[mi.Idx], topology);

	//CheckErrorCuda(hipDeviceSynchronize());
	//CheckErrorCuda(hipMemcpy(&mi.VisibleNumTriangles, dev_VisibleNumTriangles, sizeof(unsigned int), hipMemcpyDeviceToHost));
	//m_TotalVisibleNumTriangles += mi.VisibleNumTriangles;

	//CheckErrorCuda(hipFree(dev_VisibleNumTriangles));
}

CPU_CALLABLE
void CUDARenderer::Rasterizer(const MeshIdentifier& mi, const FVector3& camFwd, const CullingMode cm)
{
	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = ((unsigned int)m_TotalNumTriangles - 1) / numThreadsPerBlock + 1;
	//const size_t numSharedMemoryBytesPerBlock = (sizeof(TriangleIdx) * m_TotalNumTriangles) / numBlocks;
	RasterizerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], dev_OVertexBuffer[mi.Idx], mi.TotalNumTriangles,
		dev_PixelShadeBuffer, dev_DepthBuffer, dev_Mutex, mi.Textures,
		camFwd, cm, m_WindowHelper.Width, m_WindowHelper.Height);
}

CPU_CALLABLE
void CUDARenderer::PixelShader(SampleState sampleState, bool isDepthColour)
{
	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	PixelShaderKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_FrameBuffer, dev_PixelShadeBuffer, sampleState, isDepthColour,
		m_WindowHelper.Width, m_WindowHelper.Height);
}

CPU_CALLABLE
void CUDARenderer::DrawTexture(char* tP)
{
	SDL_Surface* pS = IMG_Load(tP);

	int w = pS->w;
	int h = pS->h;
	int bpp = pS->format->BytesPerPixel;
	unsigned int* buffer;
	size_t pitch{};
	CheckErrorCuda(hipMallocPitch((void**)&buffer, &pitch, w * bpp, h)); //2D array
	CheckErrorCuda(hipMemcpy2D(buffer, pitch, buffer, pitch, w * bpp, h, hipMemcpyHostToDevice));

	//hipChannelFormatDesc formatDesc = hipCreateChannelDesc<unsigned int>();

	hipResourceDesc resDesc{};
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = buffer;
	resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
	resDesc.res.pitch2D.desc.x = pS->format->BitsPerPixel;
	resDesc.res.pitch2D.width = w;
	resDesc.res.pitch2D.height = h;
	resDesc.res.pitch2D.pitchInBytes = pitch;

	hipTextureDesc texDesc{};
	texDesc.normalizedCoords = true; //able to sample texture with normalized uv coordinates
	texDesc.filterMode = hipFilterModePoint; //linear only supports float (and double) type
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex{};
	CheckErrorCuda(hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr));

	GPUTexture texture{};
	texture.dev_pTex = tex;
	texture.w = w;
	texture.h = h;
	texture.dev_TextureData = buffer;

	EnterValidRenderingState();

	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	TextureTestKernel<<<numBlocks, numThreadsPerBlock>>>(dev_FrameBuffer, texture, m_WindowHelper.Width, m_WindowHelper.Height);

	Present();

	//destroy texture object
	CheckErrorCuda(hipDestroyTextureObject(tex));

	SDL_FreeSurface(pS);

	//do not free buffer if it is meant to be reused
	CheckErrorCuda(hipFree(buffer));
}

CPU_CALLABLE
void CUDARenderer::DrawTextureGlobal(char* tp, bool isStretchedToWindow, SampleState sampleState)
{
	SDL_Surface* pS = IMG_Load(tp);

	int w = pS->w;
	int h = pS->h;
	int N = w * h;
	unsigned int* buffer;
	hipMalloc(&buffer, N * sizeof(unsigned int));
	hipMemcpy(buffer, pS->pixels, N * sizeof(unsigned int), hipMemcpyHostToDevice);

	EnterValidRenderingState();

	GPUTexture gpuTexture{};
	gpuTexture.dev_pTex = 0; //none
	gpuTexture.dev_TextureData = buffer;
	gpuTexture.w = w;
	gpuTexture.h = h;

	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	DrawTextureGlobalKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_FrameBuffer, gpuTexture, isStretchedToWindow, 
		sampleState, m_WindowHelper.Width, m_WindowHelper.Height);

	Present();

	SDL_FreeSurface(pS);

	hipFree(buffer);
}

#pragma endregion

#pragma region PUBLIC FUNCTIONS

CPU_CALLABLE
void CUDARenderer::LoadScene(const SceneGraph* pSceneGraph)
{
	if (!pSceneGraph)
	{
		std::cout << "!CUDARenderer::LoadScene > Invalid scenegraph!\n";
		return;
	}
	m_TotalNumTriangles = 0;
	FreeMeshBuffers();
	const std::vector<Mesh*>& pMeshes = pSceneGraph->GetMeshes();
	for (const Mesh* pMesh : pMeshes)
	{
		MeshIdentifier mi{};
		mi.Idx = m_MeshIdentifiers.size();
		mi.pMesh = pMesh;
		size_t numTriangles{};

		float* vertexBuffer = pMesh->GetVertices();
		unsigned int* indexBuffer = pMesh->GetIndexes();
		const unsigned int numVertices = pMesh->GetVertexAmount();
		const unsigned int numIndices = pMesh->GetIndexAmount();
		const PrimitiveTopology topology = pMesh->GetTopology();
		const short vertexType = pMesh->GetVertexType();
		const short stride = pMesh->GetVertexStride();
		const FMatrix4& worldMat = pMesh->GetWorldMatrix();

		switch (topology)
		{
		case PrimitiveTopology::TriangleList:
			numTriangles += numIndices / 3;
			break;
		case PrimitiveTopology::TriangleStrip:
			numTriangles += numIndices - 2;
			break;
		}
		mi.TotalNumTriangles = numTriangles;

		AllocateMeshBuffers(numVertices, numIndices, numTriangles, mi.Idx);
		CopyMeshBuffers(vertexBuffer, numVertices, stride, indexBuffer, numIndices, mi.Idx);
		LoadMeshTextures(pMesh->GetTexPaths(), mi.Idx);
		mi.Textures = m_TextureObjects[mi.Idx];

		m_TotalNumTriangles += numTriangles;
		m_MeshIdentifiers.push_back(mi);
	}
}

CPU_CALLABLE
void CUDARenderer::Render(const SceneManager& sm, const Camera* pCamera)
{
	//Render Data
	const bool isDepthColour = sm.IsDepthColour();
	const SampleState sampleState = sm.GetSampleState();
	const CullingMode cm = sm.GetCullingMode();

	//Camera Data
	const FPoint3& camPos = pCamera->GetPos();
	const FVector3& camFwd = pCamera->GetForward();
	const FMatrix4 lookatMatrix = pCamera->GetLookAtMatrix();
	const FMatrix4 viewMatrix = pCamera->GetViewMatrix(lookatMatrix);
	const FMatrix4 projectionMatrix = pCamera->GetProjectionMatrix();
	const FMatrix4 viewProjectionMatrix = projectionMatrix * viewMatrix;

	//TODO: use renderdata as constant memory

	//TODO: random illegal memory access BUG
	//Update global memory for camera's matrices
	//UpdateCameraDataAsync(camPos, viewProjectionMatrix);

	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_CameraPos_const), camPos.data, sizeof(camPos), 0, hipMemcpyHostToDevice));

	//SceneGraph Data
	const SceneGraph* pSceneGraph = sm.GetSceneGraph();
	const std::vector<Mesh*>& pObjects = pSceneGraph->GetMeshes();

#ifdef BENCHMARK
	float vertexShadingMs{};
	float TriangleAssemblingMs{};
	float RasterizationMs{};
#endif

	m_TotalVisibleNumTriangles = 0;
	for (MeshIdentifier& mi : m_MeshIdentifiers)
	{
		//Mesh Data
		const Mesh* pMesh = pObjects[mi.Idx];
		const FMatrix4& worldMat = pMesh->GetWorldMatrix();
		const FMatrix4 worldViewProjectionMatrix = viewProjectionMatrix * worldMat;
		const FMatrix3 rotationMatrix = (FMatrix3)worldMat;

		//Update const data
		CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_WorldMatrix_const), worldMat.data, sizeof(worldMat), 0, hipMemcpyHostToDevice));
		CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_WVPMatrix_const), worldViewProjectionMatrix.data, sizeof(worldViewProjectionMatrix), 0, hipMemcpyHostToDevice));
		CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_RotationMatrix_const), rotationMatrix.data, sizeof(rotationMatrix), 0, hipMemcpyHostToDevice));
		hipDeviceSynchronize();

		//TODO: can async copy (parts of) mesh buffers H2D

#ifdef BENCHMARK
		StartTimer();
#endif
		//TODO: async & streams
		//TODO: find out what order is best, for cudaDevCpy and Malloc
		//---STAGE 1---:  Perform Output Vertex Assembling
		VertexShader(mi);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 1---
#ifdef BENCHMARK
		vertexShadingMs += StopTimer();
		StartTimer();
#endif
		//---STAGE 2---:  Perform Triangle Assembling
		TriangleAssembler(mi);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 2---
#ifdef BENCHMARK
		TriangleAssemblingMs += StopTimer();
		StartTimer();
#endif
		//---STAGE 3---: Peform Triangle Rasterization & Pixel Shading
		Rasterizer(mi, camFwd, cm);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 3---
#ifdef BENCHMARK
		RasterizationMs += StopTimer();
#endif
	}

#ifdef BENCHMARK
		StartTimer();
#endif
		//---STAGE 4---: Peform  Pixel Shading
		PixelShader(sampleState, isDepthColour);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 4---
#ifdef BENCHMARK
		float PixelShadingMs = StopTimer();
		std::cout << "VS: " << vertexShadingMs << "ms | TA: " << TriangleAssemblingMs << "ms | Raster: " << RasterizationMs << "ms | PS: " << PixelShadingMs << "ms\r";
#endif
}

CPU_CALLABLE
void CUDARenderer::RenderAuto(const SceneManager& sm, const Camera* pCamera)
{
#ifdef _DEBUG
	if (EnterValidRenderingState())
		exit(1);
#else
	EnterValidRenderingState();
#endif

	Render(sm, pCamera);

	//TODO: parallel copies (streams & async)
	//Swap out buffers and update window
	Present();
}

CPU_CALLABLE
void CUDARenderer::StartTimer()
{
	CheckErrorCuda(hipEventRecord(m_StartEvent));
}

CPU_CALLABLE
float CUDARenderer::StopTimer()
{
	CheckErrorCuda(hipEventRecord(m_StopEvent));
	CheckErrorCuda(hipEventSynchronize(m_StopEvent));
	CheckErrorCuda(hipEventElapsedTime(&m_TimerMs, m_StartEvent, m_StopEvent));
	return m_TimerMs;
}

CPU_CALLABLE
void CUDARenderer::WarmUp()
{
	ResetDepthBufferKernel<<<0, 0>>>(nullptr, 0, m_WindowHelper.Width, m_WindowHelper.Height);
	ClearFrameBufferKernel<<<0, 0>>>(nullptr, m_WindowHelper.Width, m_WindowHelper.Height, 0);
	ClearPixelShadeBufferKernel<<<0, 0>>>(nullptr, 0);
	VertexShaderKernel<<<0, 0>>>(nullptr, nullptr, 0);
	TriangleAssemblerKernel<<<0, 0>>>(nullptr, nullptr, 0, nullptr, (PrimitiveTopology)0);
	RasterizerKernel<<<0, 0>>>(nullptr, nullptr, 0, nullptr, nullptr, nullptr, {}, {}, (CullingMode)0, m_WindowHelper.Width, m_WindowHelper.Height);
	PixelShaderKernel<<<0, 0>>> (nullptr, nullptr, SampleState(0), false, m_WindowHelper.Width, m_WindowHelper.Height);
}

#pragma endregion