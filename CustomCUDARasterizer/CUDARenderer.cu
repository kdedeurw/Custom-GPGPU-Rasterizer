#include "hip/hip_runtime.h"
#include "PCH.h"
#include "CUDARenderer.h"
#include <vector>

//Project CUDA includes
#include "GPUTextureSampler.cuh"
#include "CUDAMatrixMath.cuh"
#include "RasterizerOperations.cu"

#pragma region STRUCT DECLARATIONS

struct CUDAMeshBuffers // size == 12
{
	FPoint4* pPositions;
	union
	{
		OVertexData* pVertexDatas;
		struct
		{
			FVector2* pUVs;
			FVector3* pNormals;
			FVector3* pTangents;
			FVector3* pViewDirections;
			RGBColor* pColours;
		};
	};
};

#pragma endregion

#pragma region GLOBAL VARIABLES

//CONST DEVICE MEMORY - Does NOT have to be allocated or freed
constexpr unsigned int ConstMemorySize = 256;
GPU_CONST_MEMORY float dev_ConstMemory[ConstMemorySize];
//GPU_CONST_MEMORY float dev_CameraPos_const[sizeof(FPoint3) / sizeof(float)];
//GPU_CONST_MEMORY float dev_WVPMatrix_const[sizeof(FMatrix4) / sizeof(float)];
//GPU_CONST_MEMORY float dev_WorldMatrix_const[sizeof(FMatrix4) / sizeof(float)];
//GPU_CONST_MEMORY float dev_RotationMatrix_const[sizeof(FMatrix3) / sizeof(float)];

constexpr int NumTextures = 4;

//DEVICE MEMORY - Does have to be allocated and freed
static unsigned int* dev_FrameBuffer{};
static int* dev_DepthBuffer{}; //defined as INTEGER type for atomicCAS to work properly
static int* dev_DepthMutexBuffer{};
static PixelShade* dev_PixelShadeBuffer{}; //(== fragmentbuffer)
static std::vector<IVertex*> dev_IVertexBuffer{};
static std::vector<unsigned int*> dev_IndexBuffer{};
static std::vector<OVertex*> dev_OVertexBuffer{};
static std::vector<TriangleIdx*> dev_Triangles{};
static std::vector<unsigned int*> dev_TextureData{};
static unsigned int* dev_NumVisibleTriangles{};
static unsigned int* dev_BinQueueSizes{};
static unsigned int* dev_BinQueues{};
static int* dev_BinQueueSizesMutexBuffer{}; //TODO: can be 16-bit to save memory?
/*
//DEPRECATED
//Texture references have to be statically declared in global memory and bound to CUDA texture memory
//They cannot be referenced in functions, nor used in arrays
typedef texture<unsigned int, hipTextureType2D, hipReadModeElementType> CUDA32bTexture2D;
static CUDA32bTexture2D dev_DiffuseTextureReference{};
static CUDA32bTexture2D dev_NormalTextureReference{};
static CUDA32bTexture2D dev_SpecularTextureReference{};
static CUDA32bTexture2D dev_GlossinessTextureReference{};
*/

#pragma endregion

//--------------------------

CPU_CALLABLE
CUDARenderer::CUDARenderer(const WindowHelper& windowHelper, IPoint2 numBins, IPoint2 binDim, unsigned int binQueueMaxSize)
	: m_WindowHelper{ windowHelper }
	, m_TotalNumTriangles{}
	, m_TotalVisibleNumTriangles{}
	, m_h_pFrameBuffer{}
	, m_MeshIdentifiers{}
	, m_TextureObjects{}
	, m_BenchMarker{}
	, m_BinDim{ binDim }
	, m_BinQueues{ (unsigned int)numBins.x, (unsigned int)numBins.y, binQueueMaxSize }
{
	InitCUDADeviceBuffers();
}

CPU_CALLABLE
CUDARenderer::~CUDARenderer()
{
	CheckErrorCuda(DeviceSynchroniseCuda());
	FreeCUDADeviceBuffers();
}

#pragma region CPU HELPER FUNCTIONS

#pragma region PUBLIC FUNCTIONS

CPU_CALLABLE
void CUDARenderer::DisplayGPUSpecs(int deviceId)
{
	std::string yn{};

	std::cout << "\n---General---\n";
	hipDeviceProp_t prop;
	CheckErrorCuda(hipGetDeviceProperties(&prop, deviceId));
	std::cout << "Device detected: " << prop.name << '\n';
	std::cout << "Compute Capability: " << prop.major << '.' << prop.minor << '\n';
	std::cout << "Compute Mode: ";
	switch (prop.computeMode)
	{
	case hipComputeModeDefault:
		std::cout << "Default\n";
		break;
	case hipComputeModeExclusive:
		std::cout << "Exclusive\n";
		break;
	case hipComputeModeProhibited:
		std::cout << "Prohibited\n";
		break;
	case hipComputeModeExclusiveProcess:
		std::cout << "ExclusiveProcess\n";
		break;
	default:
		std::cout << "Undefined\n";
		break;
	}
	if (prop.isMultiGpuBoard)
	{
		std::cout << "Multi GPU setup: Yes\n";
		std::cout << "Multi GPU boardgroup ID: " << prop.multiGpuBoardGroupID << '\n';
	}
	else
	{
		std::cout << "Multi GPU setup: No\n";
	}
	std::cout << "Async Engine (DMA) count: " << prop.asyncEngineCount << '\n';
	yn = prop.deviceOverlap ? "Yes\n" : "No\n";
	std::cout << "Can concurrently copy memory between host and device while executing kernel: " << yn;
	switch (prop.asyncEngineCount)
	{
	case 0:
		std::cout << "Device cannot concurrently copy memory between host and device while executing a kernel\n";
		break;
	case 1:
		std::cout << "Device can concurrently copy memory between host and device while executing a kernel\n";
		break;
	case 2:
		std::cout << "Device can concurrently copy memory between host and device in both directions and execute a kernel at the same time\n";
		break;
	default:
		break;
	}
	yn = prop.concurrentKernels ? "Yes\n" : "No\n";
	std::cout << "Device supports executing multiple kernels within the same context simultaneously: " << yn;
	yn = prop.integrated ? "Yes\n" : "No\n";
	std::cout << "Integrated Graphics: " << yn;

	std::cout << "\n---Memory---\n";
	std::cout << "Total amount of Global Memory: " << ToMbs(prop.totalGlobalMem) << '\n';
	std::cout << "Total amount of Const Memory: " << ToKbs(prop.totalConstMem) << '\n';
	//size_t free{}, total{};
	//hipMemGetInfo(&free, &total);
	//std::cout << "Total amount of VRAM: " << total << '\n';
	//std::cout << "Free amount of VRAM: " << free << '\n';
	std::cout << "Shared Memory per Multiprocessor: " << ToKbs(prop.sharedMemPerMultiprocessor) << '\n';
	std::cout << "Shared Memory per Block: " << ToKbs(prop.sharedMemPerBlock) << '\n';

	std::cout << "Shared Memory Reserved by CUDA driver per Block: " << prop.reservedSharedMemPerBlock << " bytes\n";

	yn = prop.unifiedAddressing ? "Yes\n" : "No\n";
	std::cout << "Unified Addressing supported: " << yn;
	yn = prop.managedMemory ? "Yes\n" : "No\n";
	std::cout << "Managed Memory supported: " << yn;
	yn = prop.pageableMemoryAccess ? "Yes\n" : "No\n";
	std::cout << "Device can coherently access Pageable Memory (non-pinned memory): " << yn;
	yn = prop.pageableMemoryAccessUsesHostPageTables ? "Yes\n" : "No\n";
	std::cout << "Device can access pageable memory via host's page tables: " << yn;
	yn = prop.canMapHostMemory ? "Yes\n" : "No\n";
	std::cout << "Can Map host memory: " << yn;

	std::cout << "\n---Memory - Caching---\n";
	yn = prop.globalL1CacheSupported ? "Yes\n" : "No\n";
	std::cout << "Global L1 Cache Supported: " << yn;
	yn = prop.localL1CacheSupported ? "Yes\n" : "No\n";
	std::cout << "Local L1 Cache Supported: " << yn;
	std::cout << "L2 Cache Size: " << ToKbs(prop.l2CacheSize) << '\n';;
	std::cout << "Persisting L2 Cache Max Size: " << prop.persistingL2CacheMaxSize << " bytes\n";

	std::cout << "\n---Other---\n";
	std::cout << "ClockRate: " << prop.clockRate / 1000 << "Khz\n";
	std::cout << "Memory ClockRate: " << prop.memoryClockRate / 1000 << "Khz\n";
	std::cout << "Memory Pitch: " << prop.memPitch << " bytes\n";
	std::cout << "Maximum number of 32-bit registers per Multiprocessor: " << prop.regsPerMultiprocessor << "\n";
	std::cout << "Maximum number of 32-bit registers per Block: " << prop.regsPerBlock << "\n";

	std::cout << "\n---Thread specifications---\n";
	std::cout << "Max threads per Multiprocessor: " << prop.maxThreadsPerMultiProcessor << '\n';
	std::cout << "Max threads per Block: " << prop.maxThreadsPerBlock << '\n';
	std::cout << "Max threads Dimensions: X: " << prop.maxThreadsDim[0] << ", Y: " << prop.maxThreadsDim[1] << ", Z: " << prop.maxThreadsDim[2] << '\n';
	std::cout << "Warp Size (in threads): " << prop.warpSize << '\n';
	std::cout << '\n';
}

CPU_CALLABLE
void CUDARenderer::LoadScene(const SceneGraph* pSceneGraph)
{
	if (!pSceneGraph)
	{
		std::cout << "!CUDARenderer::LoadScene > Invalid scenegraph!\n";
		return;
	}
	m_TotalNumTriangles = 0;
	FreeMeshBuffers();
	const std::vector<Mesh*>& pMeshes = pSceneGraph->GetMeshes();
	for (const Mesh* pMesh : pMeshes)
	{
		MeshIdentifier mi{};
		mi.Idx = m_MeshIdentifiers.size();
		mi.pMesh = pMesh;
		size_t numTriangles{};

		const std::vector<IVertex> vertexBuffer = pMesh->GetVertexBuffer();
		const std::vector<unsigned int> indexBuffer = pMesh->GetIndexBuffer();
		const unsigned int numVertices = pMesh->GetVertexAmount();
		const unsigned int numIndices = pMesh->GetIndexAmount();
		const PrimitiveTopology topology = pMesh->GetTopology();
		const short stride = pMesh->GetVertexStride();
		const FMatrix4& worldMat = pMesh->GetWorldMatrix();

		switch (topology)
		{
		case PrimitiveTopology::TriangleList:
			numTriangles += numIndices / 3;
			break;
		case PrimitiveTopology::TriangleStrip:
			numTriangles += numIndices - 2;
			break;
		}
		mi.TotalNumTriangles = numTriangles;

		AllocateMeshBuffers(numVertices, numIndices, numTriangles, stride, mi.Idx);
		const float* pVertices = reinterpret_cast<const float*>(vertexBuffer.data());
		const unsigned int* pIndices = reinterpret_cast<const unsigned int*>(indexBuffer.data());
		CopyMeshBuffers(pVertices, numVertices, stride, pIndices, numIndices, mi.Idx);
		if (!pMesh->GetTexPaths()->empty())
		{
			GPUTexturesCompact gpuTextures = LoadMeshTextures(pMesh->GetTexPaths(), mi.Idx);
			m_TextureObjects[mi.Idx] = gpuTextures;
			mi.Textures = gpuTextures;
		}

		m_TotalNumTriangles += numTriangles;
		m_MeshIdentifiers.push_back(mi);
	}
}

CPU_CALLABLE
void CUDARenderer::Render(const SceneManager& sm, const Camera* pCamera)
{
	//Render Data
	const bool isDepthColour = sm.IsDepthColour();
	const SampleState sampleState = sm.GetSampleState();
	const CullingMode cm = sm.GetCullingMode();

	//Camera Data
	const FPoint3& camPos = pCamera->GetPos();
	const FVector3& camFwd = pCamera->GetForward();
	const FMatrix4 lookatMatrix = pCamera->GetLookAtMatrix();
	const FMatrix4 viewMatrix = pCamera->GetViewMatrix(lookatMatrix);
	const FMatrix4 projectionMatrix = pCamera->GetProjectionMatrix();
	const FMatrix4 viewProjectionMatrix = projectionMatrix * viewMatrix;

	UpdateCameraDataAsync(camPos, camFwd);

	//SceneGraph Data
	const SceneGraph* pSceneGraph = sm.GetSceneGraph();
	const std::vector<Mesh*>& pMeshes = pSceneGraph->GetMeshes();

#ifdef BENCHMARK
	float VertexShadingMs{};
#ifdef BINNING
	float TriangleBinningMs{};
#endif
	float TriangleAssemblingMs{};
	float RasterizationMs{};
	float PixelShadingMs{};
#endif

	m_TotalVisibleNumTriangles = 0;
	for (MeshIdentifier& mi : m_MeshIdentifiers)
	{
		//Mesh Data
		const Mesh* pMesh = pMeshes[mi.Idx];
		//Transpose when using shared memory
		//const FMatrix4 worldMat = Transpose(pMesh->GetWorldMatrix());
		//const FMatrix4 worldViewProjectionMatrix = Transpose(viewProjectionMatrix * worldMat);
		//const FMatrix3 rotationMatrix = Transpose(pMesh->GetRotationMatrix());
		//Use normal when using Elite Math Library
		const FMatrix4 worldMat = pMesh->GetWorldMatrix();
		const FMatrix4 worldViewProjectionMatrix = viewProjectionMatrix * worldMat;
		const FMatrix3 rotationMatrix = pMesh->GetRotationMatrix();

		//Update const data
		UpdateWorldMatrixDataAsync(worldMat, worldViewProjectionMatrix, rotationMatrix);
		hipDeviceSynchronize();

		//TODO: can async copy (parts of) mesh buffers H2D
		//TODO: async & streams + find out what order is best, for cudaDevCpy and Malloc

#ifdef BENCHMARK
		StartTimer();
#endif

		//---STAGE 1---:  Perform Output Vertex Assembling
		VertexShader(mi);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 1---

#ifdef BENCHMARK
		VertexShadingMs += StopTimer();
		StartTimer();
#endif

		//Reset number of visible triangles
		CheckErrorCuda(hipMemset(dev_NumVisibleTriangles, 0, sizeof(unsigned int)));

		//---STAGE 2---:  Perform Triangle Assembling
		TriangleAssembler(mi, camFwd, cm);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 2---

		CheckErrorCuda(hipMemcpy(&mi.VisibleNumTriangles, dev_NumVisibleTriangles, 4, hipMemcpyDeviceToHost));
		m_TotalVisibleNumTriangles += mi.VisibleNumTriangles;

#ifdef BENCHMARK
		TriangleAssemblingMs += StopTimer();
		StartTimer();
#endif

#ifdef BINNING

		//---STAGE 3---:  Perform Output Vertex Assembling
		TriangleBinner(mi);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 3---

#ifdef BENCHMARK
		TriangleBinningMs += StopTimer();
		StartTimer();
#endif
#endif

		//TODO: not have it per-mesh, but globally per-scene
		//TRIANGLES GET PUT INTO BINS
		//RASTERIZER FETCHES TRIANGLES FROM BINS
		//THIS CONTINUES !PER-MESH! UNTIL ALL TRIANGLES ARE PROCESSED
		//THEN CONTINUE TO NEXT MESH

		//---STAGE 4---: Peform Triangle Rasterization & interpolated fragment buffering
		Rasterizer(mi, camFwd, cm);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 4---

#ifdef BENCHMARK
		RasterizationMs += StopTimer();
#endif
	}

#ifdef BENCHMARK
	StartTimer();
#endif

	//---STAGE 5---: Peform Pixel Shading
	PixelShader(sampleState, isDepthColour);
	CheckErrorCuda(hipDeviceSynchronize());

	//---END STAGE 5---
#ifdef BENCHMARK
	PixelShadingMs = StopTimer();
	std::cout << "VS: " << VertexShadingMs 
		<< "ms | TA: " << TriangleAssemblingMs 
#ifdef BINNING
		<< "ms | Bin: " << TriangleBinningMs 
#endif
		<< "ms | Raster: " << RasterizationMs 
		<< "ms | PS: " << PixelShadingMs << "ms\r";
#endif
}

CPU_CALLABLE
void CUDARenderer::RenderAuto(const SceneManager& sm, const Camera* pCamera)
{
#ifdef _DEBUG
	if (EnterValidRenderingState())
		exit(1);
#else
	EnterValidRenderingState();
#endif

	Render(sm, pCamera);

	//TODO: parallel copies (streams & async)
	//Swap out buffers and update window
	Present();
}

CPU_CALLABLE
int CUDARenderer::EnterValidRenderingState()
{
	//https://wiki.libsdl.org/SDL_LockSurface
	int state = SDL_LockSurface(m_WindowHelper.pBackBuffer); //Set up surface for directly accessing the pixels
	//Clear screen and reset buffers
	Clear();
	return state;
}

CPU_CALLABLE
void CUDARenderer::Present()
{
	//TODO: have Vertex Shader and Rasterizer run in parallel with cudamemcpy()
	const size_t size = m_WindowHelper.Width * m_WindowHelper.Height * sizeof(unsigned int);
	CheckErrorCuda(hipMemcpy(m_WindowHelper.pBackBufferPixels, dev_FrameBuffer, size, hipMemcpyDeviceToHost)); //We can directly read/write from pixelbuffer
	//memcpy(m_WindowHelper.pBackBufferPixels, m_WindowHelper.h_BackBufferPixels, size);
	SDL_UnlockSurface(m_WindowHelper.pBackBuffer); //Release a surface after directly accessing the pixels.
	SDL_BlitSurface(m_WindowHelper.pBackBuffer, 0, m_WindowHelper.pFrontBuffer, 0); //Copy the window surface to the screen.
	SDL_UpdateWindowSurface(m_WindowHelper.pWindow); //Update Window's surface
}

CPU_CALLABLE
void CUDARenderer::StartTimer()
{
	m_BenchMarker.StartTimer();
}

CPU_CALLABLE
float CUDARenderer::StopTimer()
{
	return m_BenchMarker.StopTimer();
}

#pragma endregion

#pragma region PRIVATE FUNCTIONS

CPU_CALLABLE
void CUDARenderer::InitCUDADeviceBuffers()
{
	size_t size{};
	const unsigned int width = m_WindowHelper.Width;
	const unsigned int height = m_WindowHelper.Height;

	//CUDAHOSTALLOC FLAGS
	/*
	hipHostMallocDefault: This flag's value is defined to be 0 and causes hipHostAlloc() to emulate hipHostMalloc().
	hipHostMallocPortable: The memory returned by this call will be considered as pinned memory by all CUDA contexts, not just the one that performed the allocation.
	hipHostMallocMapped: Maps the allocation into the CUDA address space. The device pointer to the memory may be obtained by calling hipHostGetDevicePointer().
	hipHostMallocWriteCombined: Allocates the memory as write-combined (WC).
	WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read efficiently by most CPUs.
	WC memory is a good option for buffers that will be written by the CPU and read by the device via mapped pinned memory or host->device transfers.
	*/

	//--->PINNED MEMORY<--- (HOST ONLY)
	//+ makes memory transactions between host and device significantly faster
	//- however this will allocate on host's RAM memory (in this case it would be 640 * 480 * 4 bytes == 1.2288Mb)
	//size = sizeof(unsigned int);
	//CheckErrorCuda(hipHostMalloc((void**)&m_h_pFrameBuffer, width * height * size));
	//CheckErrorCuda(hipHostAlloc((void**)&m_h_pFrameBuffer, width * height * size, hipHostMallocPortable));
	
	//host pinned memory without SDL window pixelbuffer
	//SDL allows random access to pixelbuffer, but cuda does not allowed host memory to be there

	//CUDAHOSTREGISTER FLAGS
	/*
	hipHostRegisterDefault: On a system with unified virtual addressing, the memory will be both mapped and portable. 
	On a system with no unified virtual addressing, the memory will be neither mapped nor portable.
	hipHostRegisterPortable: The memory returned by this call will be considered as pinned memory by all CUDA contexts, not just the one that performed the allocation.
	hipHostRegisterMapped: Maps the allocation into the CUDA address space. The device pointer to the memory may be obtained by calling hipHostGetDevicePointer().
	hipHostRegisterIoMemory: The passed memory pointer is treated as pointing to some memory-mapped I/O space, 
	e.g. belonging to a third-party PCIe device, and it will marked as non cache-coherent and contiguous.
	hipHostRegisterReadOnly: The passed memory pointer is treated as pointing to memory that is considered read-only by the device.
	On platforms without hipDeviceAttributePageableMemoryAccessUsesHostPageTables, this flag is required in order to register memory mapped to the CPU as read-only.
	Support for the use of this flag can be queried from the device attribute cudaDeviceAttrReadOnlyHostRegisterSupported.
	Using this flag with a current context associated with a device that does not have this attribute set will cause hipHostRegister to error with hipErrorNotSupported.
	*/

	//Invalid Argument
	//SDL_LockSurface(m_WindowHelper.pBackBuffer);
	//size = sizeof(unsigned int);
	//CheckErrorCuda(hipHostRegister(m_WindowHelper.pBackBufferPixels, width * height * size, hipHostRegisterDefault));
	//SDL_UnlockSurface(m_WindowHelper.pBackBuffer);;

	size = sizeof(PixelShade);
	CheckErrorCuda(hipFree(dev_PixelShadeBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_PixelShadeBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_PixelShadeBuffer, 0, width * height * size));

	//The framebuffer in device memory
	size = sizeof(unsigned int);
	CheckErrorCuda(hipFree(dev_FrameBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_FrameBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_FrameBuffer, 0, width * height * size));

	size = sizeof(int);
	CheckErrorCuda(hipFree(dev_DepthBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_DepthBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_DepthBuffer, 0, width * height * size));

	size = sizeof(int);
	hipFree(dev_DepthMutexBuffer);
	hipMalloc((void**)&dev_DepthMutexBuffer, width * height * size);
	hipMemset(dev_DepthMutexBuffer, 0, width * height * size);

	//NOTE: can only set data PER BYTE
	//PROBLEM: setting each byte to UCHAR_MAX (255) is impossible, since floating point numbers work differently (-nan result)
	//	0		11111110	11111111111111111111111
	//	^			^				^
	//	sign	exponent		mantissa
	//			254 - 127     2 - 2 ^ (-23)
	// 340282346638528859811704183484516925440.0   // FLT_MAX
	// 340282366920938463463374607431768211456.0   // 2^128
	//https://stackoverflow.com/questions/16350955/interpreting-the-bit-pattern-of-flt-max
	//SOLUTION:
	//Option 1: allocate float[width*height] and initialize to FLT_MAX, then memcpy (wastes lots of memory)
	//Option 2: loop through entire dev_array and set each member to FLT_MAX (too many global accesses)
	//Option 3: interpret depth buffer invertedly, so a depthvalue of 1.f is closest, and 0.f is furthest away from camera
	//>Option 4<: initialize and reset depthbuffer through additional kernel call, however this would be a lot of global memory accesses

	//BINNING
	size = sizeof(unsigned int);
	hipFree(dev_NumVisibleTriangles);
	hipMalloc((void**)&dev_NumVisibleTriangles, size);
	hipMemset(dev_NumVisibleTriangles, 0, size);

	//actual arrays of triangle indexes
	size = sizeof(unsigned int);
	hipFree(dev_BinQueues);
	hipMalloc((void**)&dev_BinQueues, m_BinQueues.NumQueuesX * m_BinQueues.NumQueuesY * m_BinQueues.QueueMaxSize * size);
	hipMemset(dev_BinQueues, 0, m_BinQueues.NumQueuesX * m_BinQueues.NumQueuesY * m_BinQueues.QueueMaxSize * size);

	//actual size of arrays
	size = sizeof(unsigned int);
	hipFree(dev_BinQueueSizes);
	hipMalloc((void**)&dev_BinQueueSizes, m_BinQueues.NumQueuesX * m_BinQueues.NumQueuesY * size);
	hipMemset(dev_BinQueueSizes, 0, m_BinQueues.NumQueuesX * m_BinQueues.NumQueuesY * size);

	//mutex buffer to alter bin queue sizes
	size = sizeof(int);
	hipFree(dev_BinQueueSizesMutexBuffer);
	hipMalloc((void**)&dev_BinQueueSizesMutexBuffer, m_BinQueues.NumQueuesX * m_BinQueues.NumQueuesY * size);
	hipMemset(dev_BinQueueSizesMutexBuffer, 0, m_BinQueues.NumQueuesX * m_BinQueues.NumQueuesY * size);
}

CPU_CALLABLE
void CUDARenderer::AllocateMeshBuffers(const size_t numVertices, const size_t numIndices, const size_t numTriangles, unsigned int stride, size_t meshIdx)
{
	const size_t newSize = meshIdx + 1;
	if (newSize > dev_IVertexBuffer.capacity())
	{
		//TODO: reserve
		dev_IVertexBuffer.resize(newSize);
		dev_IndexBuffer.resize(newSize);
		dev_OVertexBuffer.resize(newSize);
		dev_Triangles.resize(newSize);
	}
	else
	{
		//Free unwanted memory
		CheckErrorCuda(hipFree(dev_IVertexBuffer[meshIdx]));
		CheckErrorCuda(hipFree(dev_IndexBuffer[meshIdx]));
		CheckErrorCuda(hipFree(dev_OVertexBuffer[meshIdx]));
		CheckErrorCuda(hipFree(dev_Triangles[meshIdx]));
	}

	//Allocate Input Vertex Buffer
	IVertex* pDevIVertexBuffer;
	CheckErrorCuda(hipMalloc((void**)&pDevIVertexBuffer, numVertices * stride));
	dev_IVertexBuffer[meshIdx] = pDevIVertexBuffer;
	//Allocate Index Buffer
	unsigned int* pDevIndexBuffer;
	CheckErrorCuda(hipMalloc((void**)&pDevIndexBuffer, numIndices * sizeof(unsigned int)));
	dev_IndexBuffer[meshIdx] = pDevIndexBuffer;
	//Allocate Ouput Vertex Buffer
	OVertex* pDevOVertexBuffer;
	CheckErrorCuda(hipMalloc((void**)&pDevOVertexBuffer, numVertices * sizeof(OVertex)));
	dev_OVertexBuffer[meshIdx] = pDevOVertexBuffer;
	//Allocate device memory for entire range of triangles
	TriangleIdx* pDevTriangleIdxBuffer;
	CheckErrorCuda(hipMalloc((void**)&pDevTriangleIdxBuffer, numTriangles * sizeof(TriangleIdx)));
	dev_Triangles[meshIdx] = pDevTriangleIdxBuffer;
}

CPU_CALLABLE
void CUDARenderer::CopyMeshBuffers(const float* vertexBuffer, unsigned int numVertices, short stride, const unsigned int* indexBuffer, unsigned int numIndices, size_t meshIdx)
{
	//Copy Input Vertex Buffer
	CheckErrorCuda(hipMemcpy(dev_IVertexBuffer[meshIdx], vertexBuffer, numVertices * stride, hipMemcpyHostToDevice));
	//Copy Index Buffer
	CheckErrorCuda(hipMemcpy(dev_IndexBuffer[meshIdx], indexBuffer, numIndices * sizeof(unsigned int), hipMemcpyHostToDevice));
}

CPU_CALLABLE
GPUTexturesCompact CUDARenderer::LoadMeshTextures(const std::string texturePaths[4], size_t meshIdx)
{
	const size_t newSize = (meshIdx + 1);
	if (newSize > m_TextureObjects.size())
	{
		m_TextureObjects.resize(newSize);
	}
	if (newSize * NumTextures > dev_TextureData.size())
	{
		dev_TextureData.resize(newSize * NumTextures);
	}

	GPUTexturesCompact gpuTextures{};

	//0 DIFFUSE > 1 NORMAL > 2 SPECULAR > 3 GLOSSINESS
	for (int i{}; i < NumTextures; ++i)
	{
		if (texturePaths[i].empty())
			continue;

		GPUTextureCompact* gpuTexture;
		switch (i)
		{
		case 0:
			gpuTexture = &gpuTextures.Diff;
			break;
		case 1:
			gpuTexture = &gpuTextures.Norm;
			break;
		case 2:
			gpuTexture = &gpuTextures.Spec;
			break;
		case 3:
			gpuTexture = &gpuTextures.Gloss;
			break;
		}

		const unsigned int textureIdx = meshIdx * NumTextures + i;
		const GPUTexture tex = LoadGPUTexture(texturePaths[i], textureIdx);

		gpuTextures.w = tex.w;
		gpuTextures.h = tex.h;
		gpuTexture->dev_pTex = tex.dev_pTex;
		gpuTexture->dev_TextureData = tex.dev_TextureData;
	}
	return gpuTextures;
}

CPU_CALLABLE
GPUTexture CUDARenderer::LoadGPUTexture(const std::string texturePath, unsigned int textureIdx)
{
	GPUTexture gpuTexture{};

	SDL_Surface* pSurface = IMG_Load(texturePath.c_str());
	if (pSurface)
	{
		const unsigned int width = pSurface->w;
		const unsigned int height = pSurface->h;
		const unsigned int* pixels = (unsigned int*)pSurface->pixels;
		const int bpp = pSurface->format->BytesPerPixel;
		//const size_t sizeInBytes = width * height * bpp;

		//copy texture data to device
		CheckErrorCuda(hipFree(dev_TextureData[textureIdx]));
		size_t pitch{};
		CheckErrorCuda(hipMallocPitch((void**)&dev_TextureData[textureIdx], &pitch, width * bpp, height)); //2D array
		CheckErrorCuda(hipMemcpy2D(dev_TextureData[textureIdx], pitch, pixels, pitch, width * bpp, height, hipMemcpyHostToDevice));

		//hipChannelFormatDesc formatDesc = hipCreateChannelDesc<unsigned int>();

		hipResourceDesc resDesc{};
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = dev_TextureData[textureIdx];
		resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
		resDesc.res.pitch2D.desc.x = pSurface->format->BitsPerPixel;
		resDesc.res.pitch2D.width = width;
		resDesc.res.pitch2D.height = height;

		hipTextureDesc texDesc{};
		texDesc.normalizedCoords = true; //able to sample texture with normalized uv coordinates
		texDesc.filterMode = hipFilterModePoint; //linear only supports float (and double) type
		texDesc.readMode = hipReadModeElementType;

		hipTextureObject_t dev_TextureObject{};
		CheckErrorCuda(hipCreateTextureObject(&dev_TextureObject, &resDesc, &texDesc, nullptr));

		/*{
			hipArray* dev_array;
			hipChannelFormatDesc formatDesc{};
			formatDesc.f = hipChannelFormatKindUnsigned;
			formatDesc.x = 32;

			hipMallocArray(&dev_array, &formatDesc, width, height, hipArrayTextureGather); //2D array
			hipMallocArray(&dev_array, &formatDesc, width * height, 1, hipArrayTextureGather); //1D array
			hipMemcpyToArray(dev_array, 0, 0, dev_TextureData[textureIdx], width * height * bpp, hipMemcpyHostToDevice);

			hipResourceDesc desc{};
			desc.resType = hipResourceTypeArray;
			desc.res.array.array = dev_array;

			hipTextureObject_t dev_TextureObject{};
			CheckErrorCuda(hipCreateTextureObject(&dev_TextureObject, &desc, &texDesc, nullptr));


			hipFreeArray(dev_array);
		}*/

		gpuTexture.dev_pTex = dev_TextureObject;
		gpuTexture.w = width;
		gpuTexture.h = height;
		gpuTexture.dev_TextureData = dev_TextureData[textureIdx];

		/*DEPRECATED
			//bind texture
			textureReference texRef{};
			texRef.normalized = false;
			texRef.channelDesc = hipCreateChannelDesc<unsigned int>();
			texRef.channelDesc.x = bpp * 8;
			texRef.channelDesc.f = hipChannelFormatKindUnsigned; //unsigned int

			size_t offset{};
			CUDA32bTexture2D texRef{}; //IN STATIC GLOBAL MEMORY!
			CheckErrorCuda(hipBindTexture2D(&offset, &texRef, dev_texData2D, &texRef.channelDesc, width, height, pitch * bpp));

			if (offset != 0)
			{
				std::cout << "Texture Offset : " << offset << '\n';
				return;
			}
		*/

		//free data
		SDL_FreeSurface(pSurface);
		//!DO NOT FREE TEXTURE DATA, as this will render the texture object invalid!
	}

	return gpuTexture;
}

CPU_CALLABLE
void CUDARenderer::FreeTextures()
{
	//destroy all texture objects
	for (const GPUTexturesCompact& textures : m_TextureObjects)
	{
		CheckErrorCuda(hipDestroyTextureObject(textures.Diff.dev_pTex));
		CheckErrorCuda(hipDestroyTextureObject(textures.Norm.dev_pTex));
		CheckErrorCuda(hipDestroyTextureObject(textures.Spec.dev_pTex));
		CheckErrorCuda(hipDestroyTextureObject(textures.Gloss.dev_pTex));
	}
	m_TextureObjects.clear();
	//free texture data
	for (unsigned int* dev_texData : dev_TextureData)
	{
		CheckErrorCuda(hipFree(dev_texData));
	}
	dev_TextureData.clear();
}

CPU_CALLABLE
void CUDARenderer::FreeMeshBuffers()
{
	for (size_t i{}; i < m_MeshIdentifiers.size(); ++i)
	{
		CheckErrorCuda(hipFree(dev_IVertexBuffer[i]));
		dev_IVertexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_IndexBuffer[i]));
		dev_IndexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_OVertexBuffer[i]));
		dev_OVertexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_Triangles[i]));
		dev_Triangles[i] = nullptr;
	}
	m_MeshIdentifiers.clear();
	dev_IVertexBuffer.clear();
	dev_IndexBuffer.clear();
	dev_OVertexBuffer.clear();
	dev_Triangles.clear();
}

CPU_CALLABLE
void CUDARenderer::FreeCUDADeviceBuffers()
{
	//Free buffers
	CheckErrorCuda(hipFree(dev_BinQueueSizesMutexBuffer));
	dev_BinQueueSizesMutexBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_BinQueueSizes));
	dev_BinQueueSizes = nullptr;

	CheckErrorCuda(hipFree(dev_BinQueues));
	dev_BinQueues = nullptr;

	CheckErrorCuda(hipFree(dev_NumVisibleTriangles));
	dev_NumVisibleTriangles = nullptr;

	CheckErrorCuda(hipFree(dev_DepthMutexBuffer));
	dev_DepthMutexBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_DepthBuffer));
	dev_DepthBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_FrameBuffer));
	dev_FrameBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_PixelShadeBuffer));
	dev_PixelShadeBuffer = nullptr;

	//not allocated, but extra safety
	CheckErrorCuda(hipHostFree(m_h_pFrameBuffer));
	m_h_pFrameBuffer = nullptr;

	FreeMeshBuffers();
	FreeTextures();
}

CPU_CALLABLE
void CUDARenderer::UpdateCameraDataAsync(const FPoint3& camPos, const FVector3& camFwd)
{
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_ConstMemory), camPos.data, sizeof(camPos)));
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_ConstMemory), camFwd.data, sizeof(camFwd), 3 * 4));
}

CPU_CALLABLE
void CUDARenderer::UpdateWorldMatrixDataAsync(const FMatrix4& worldMatrix, const FMatrix4& wvpMat, const FMatrix3& rotationMat)
{
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_ConstMemory), worldMatrix.data, sizeof(worldMatrix), 6 * 4));
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_ConstMemory), wvpMat.data, sizeof(wvpMat), 22 * 4));
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_ConstMemory), rotationMat.data, sizeof(rotationMat), 38 * 4));
}

#pragma endregion

#pragma endregion

#pragma region KERNELS

//Kernel launch params:	numBlocks, numThreadsPerBlock, numSharedMemoryBytes, stream

#pragma region Clearing

GPU_KERNEL
void ClearDepthBufferKernel(int* dev_DepthBuffer, int value, const unsigned int width, const unsigned int height)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_DepthBuffer[pixelIdx] = value;
	}
}

GPU_KERNEL
void ClearFrameBufferKernel(unsigned int* dev_FrameBuffer, const unsigned int width, const unsigned int height, unsigned int colour32)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_FrameBuffer[pixelIdx] = colour32;
	}
}

GPU_KERNEL
void ClearScreenKernel(PixelShade* dev_PixelShadeBuffer, const unsigned int width, const unsigned int height, unsigned int colour32)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_PixelShadeBuffer[pixelIdx].colour = colour32;
	}
}

GPU_KERNEL
void ClearPixelShadeBufferKernel(PixelShade* dev_PixelShadeBuffer, const unsigned int sizeInWords)
{
	//every thread sets 1 WORD of data
	const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < sizeInWords)
	{
		reinterpret_cast<float*>(dev_PixelShadeBuffer)[idx] = 0.f;
	}
}

GPU_KERNEL
void ClearDepthMutexBufferKernel(int* dev_MutexBuffer, const unsigned int width, const unsigned int height)
{
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_MutexBuffer[pixelIdx] = 0;
	}
}

GPU_KERNEL
void Clear()
{
	//TODO: clear depthbuffer, framebuffer and pixelshadebuffer
}

#pragma endregion

GPU_KERNEL
void VertexShaderKernelNaive(const IVertex* __restrict__ dev_IVertices, OVertex* dev_OVertices, unsigned int numVertices)
{
	//TODO: store matrix in top of shared memory for faster access
	//and offset shared memory access for threads
	//Potential problem: first warp might encounter bank conflicts?

	//The use of shared memory is not applicable here, (even though on-chip memory is faster)
	//- since the memory is not actually shared between threads
	//- most calculations are done within the thread's registers themselves (only 28 or 29 needed => < 32)
	//- the memory is not used multiple times

	//constexpr unsigned int paddedSizeOfIVertex = sizeof(IVertex) / 4 + 1;
	//extern GPU_SHARED_MEMORY float sharedMemoryBuffer[];

	const FPoint3& camPos = reinterpret_cast<const FPoint3&>(dev_ConstMemory[0]);
	const FMatrix4& worldMatrix = reinterpret_cast<const FMatrix4&>(dev_ConstMemory[6]);
	const FMatrix4& WVPMatrix = reinterpret_cast<const FMatrix4&>(dev_ConstMemory[22]);
	const FMatrix3& rotationMatrix = reinterpret_cast<const FMatrix3&>(dev_ConstMemory[38]);

	const unsigned int vertexIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (vertexIdx < numVertices)
	{
		const IVertex& iVertex = dev_IVertices[vertexIdx];
		OVertex oVertex = GetNDCVertex(iVertex, WVPMatrix, worldMatrix, rotationMatrix, camPos);
		dev_OVertices[vertexIdx] = oVertex;
	}
}

GPU_KERNEL
void TriangleAssemblerKernel(TriangleIdx* dev_Triangles, const unsigned int* __restrict__ const dev_IndexBuffer, unsigned int numIndices,
	unsigned int* dev_NumVisibleTriangles,
	const OVertex* dev_OVertices, const PrimitiveTopology pt, const CullingMode cm, const FVector3 camFwd, 
	unsigned int width, unsigned int height)
{
	//advantage of TriangleAssembly: each thread stores 1 triangle
	//many threads == many triangles processed and/or culled at once

	//TODO: use shared memory to copy faster
	//data size of 9 shows no bank conflicts!
	//TriangleIdx can stay in local memory (registers)
	//DEPENDS ON REGISTER USAGE

	TriangleIdx triangle;

	//'talk about naming gore, eh?
	const unsigned int indexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (pt == PrimitiveTopology::TriangleList)
	{
		const unsigned int correctedIdx = (indexIdx * 3);
		if (correctedIdx < numIndices)
		{
			memcpy(&triangle, &dev_IndexBuffer[correctedIdx], sizeof(TriangleIdx));
		}
		else
		{
			return;
		}
	}
	//TODO: get rid of these nasty if-statements
	else //if (pt == PrimitiveTopology::TriangleStrip)
	{
		if (indexIdx < numIndices - 2)
		{
			TriangleIdx triangle;
			memcpy(&triangle, &dev_IndexBuffer[indexIdx], sizeof(TriangleIdx));
			const bool isOdd = (indexIdx % 2);
			if (isOdd)
			{
				//swap without temp
				//TODO: what about indexes at UINT_MAX? => would just overflow and underflow back
				triangle.idx1 = triangle.idx1 + triangle.idx2;
				triangle.idx2 = triangle.idx1 - triangle.idx2;
				triangle.idx1 = triangle.idx1 - triangle.idx2;
			}
		}
		else
		{
			return;
		}
	}

	FPoint3 p0 = dev_OVertices[triangle.idx0].p.xyz;
	FPoint3 p1 = dev_OVertices[triangle.idx1].p.xyz;
	FPoint3 p2 = dev_OVertices[triangle.idx2].p.xyz;

	//PERFORM CULLING
	if (cm == CullingMode::BackFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ p1 - p0 }, FVector3{ p2 - p0 }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue <= 0.f)
		{
			return;
		}
	}
	else if (cm == CullingMode::FrontFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ p1 - p0 }, FVector3{ p2 - p0 }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue >= 0.f)
		{
			return; //cull triangle
		}
	}
	//else if (cm == CullingMode::NoCulling)
	//{
	//}

	//PERFORM CLIPPING
	if (!IsTriangleVisible(p0, p1, p2))
	{
		return;
	}
	
	const float totalArea = abs(Cross(p0.xy - p1.xy, p2.xy - p0.xy));
	if (totalArea <= 0.f)
	{
		return; //cull away triangle
	}

	const unsigned int triangleIdx = atomicAdd(dev_NumVisibleTriangles, 1); //returns old value
	memcpy(&dev_Triangles[triangleIdx], &triangle, sizeof(TriangleIdx));
}

GPU_KERNEL
void RasterizerPerTriangleKernel(const TriangleIdx* __restrict__ const dev_Triangles, const OVertex* __restrict__ const dev_OVertices, 
	unsigned int numTriangles, PixelShade* dev_PixelShadeBuffer, int* dev_DepthBuffer, int* dev_DepthMutexBuffer, 
	GPUTexturesCompact textures, const unsigned int width, const unsigned int height)
{
	//constexpr unsigned int triangleSize = sizeof(OVertex) * 3 / 4;
	//extern GPU_SHARED_MEMORY float sharedMemoryBuffer[];

	//Each thread processes 1 triangle
	const unsigned int globalTriangleIndex = threadIdx.x + blockIdx.x * blockDim.x;
	if (!(globalTriangleIndex < numTriangles))
		return;

	const TriangleIdx triangleIdx = dev_Triangles[globalTriangleIndex];

	//Shared memory is laid out in a big row-list
	//const unsigned int triangleMemoryIdx = threadIdx.x * triangleSize;
	//OVertex& v0 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx]);
	//OVertex& v1 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx + (sizeof(OVertex) / 4)]);
	//OVertex& v2 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx + (sizeof(OVertex) / 4) * 2]);

	//memcpy(&v0, &dev_OVertices[triangleIdx.idx0], sizeof(OVertex));
	//memcpy(&v1, &dev_OVertices[triangleIdx.idx1], sizeof(OVertex));
	//memcpy(&v2, &dev_OVertices[triangleIdx.idx2], sizeof(OVertex));

	OVertex v0 = dev_OVertices[triangleIdx.idx0];
	OVertex v1 = dev_OVertices[triangleIdx.idx1];
	OVertex v2 = dev_OVertices[triangleIdx.idx2];

	NDCToScreenSpace(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
	const BoundingBox bb = GetBoundingBox(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
	//Rasterize Screenspace triangle
	RasterizeTriangle(bb, v0, v1, v2, dev_DepthMutexBuffer, dev_DepthBuffer, dev_PixelShadeBuffer, width, textures);
}

GPU_KERNEL
void PixelShaderKernel(unsigned int* dev_FrameBuffer, const PixelShade* __restrict__ const dev_PixelShadeBuffer,
	SampleState sampleState, bool isDepthColour, const unsigned int width, const unsigned int height)
{
	//Notes: PixelShade has size of 32, but bank conflicts
	//TODO: store PixelShade data column-based to avoid bank conflicts, but faster access?
	//GPU_SHARED_MEMORY PixelShade pixelShadeSharedMemory[width * height];

	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	//TODO: if-statement is not necessary
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		RGBA rgba;
		if (isDepthColour)
		{
			rgba.r8 = 0; //For SDL: R and B values are swapped
			rgba.g8 = 0;
			rgba.b8 = (unsigned char)(Remap(dev_PixelShadeBuffer[pixelIdx].zInterpolated, 0.99f, 1.f) * 255);
			rgba.a8 = 0;
			dev_FrameBuffer[pixelIdx] = rgba.colour32;
		}
		else
		{
			const PixelShade& pixelShade = dev_PixelShadeBuffer[pixelIdx];
			if (pixelShade.textures.Diff.dev_pTex != 0)
			{
				RGBColor colour = ShadePixel(pixelShade.textures, pixelShade.uv, pixelShade.n, pixelShade.tan, pixelShade.vd, sampleState);
				rgba = colour; //== GetRGBAFromColour()
				dev_FrameBuffer[pixelIdx] = rgba.colour32;

				//normal visualisation
				//colour.r = pixelShade.n.x;
				//colour.g = pixelShade.n.y;
				//colour.b = pixelShade.n.z;
			}
			else
			{
				dev_FrameBuffer[pixelIdx] = pixelShade.colour;
			}
		}
	}
}

#pragma region Binning

GPU_KERNEL
void TriangleAssemblerAndBinnerKernel(TriangleIdx* dev_Triangles, const unsigned int* __restrict__ const dev_IndexBuffer, unsigned int numIndices,
	unsigned int* dev_NumVisibleTriangles, unsigned int* dev_BinQueueSizes, unsigned int* dev_BinQueues, int* dev_BinQueueSizesMutexBuffer,
	const OVertex* dev_OVertices, const PrimitiveTopology pt, const CullingMode cm, const FVector3 camFwd,
	IPoint2 numBins, IPoint2 binDim, unsigned int binQueueMaxSize, unsigned int width, unsigned int height)
{
	//advantage of TriangleAssembly: each thread stores 1 triangle
	//many threads == many triangles processed and/or culled at once

	//TODO: use shared memory to copy faster
	//data size of 9 shows no bank conflicts!
	//TriangleIdx can stay in local memory (registers)
	//DEPENDS ON REGISTER USAGE

	TriangleIdx triangle;

	//'talk about naming gore, eh?
	const unsigned int indexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (pt == PrimitiveTopology::TriangleList)
	{
		const unsigned int correctedIdx = (indexIdx * 3);
		if (correctedIdx < numIndices)
		{
			memcpy(&triangle, &dev_IndexBuffer[correctedIdx], sizeof(TriangleIdx));
		}
		else
		{
			return;
		}
	}
	else //if (pt == PrimitiveTopology::TriangleStrip)
	{
		if (indexIdx < numIndices - 2)
		{
			TriangleIdx triangle;
			memcpy(&triangle, &dev_IndexBuffer[indexIdx], sizeof(TriangleIdx));
			const bool isOdd = (indexIdx % 2);
			if (isOdd)
			{
				//swap without temp
				//TODO: what about indexes at UINT_MAX? => would just overflow and back
				triangle.idx1 = triangle.idx1 + triangle.idx2;
				triangle.idx2 = triangle.idx1 - triangle.idx2;
				triangle.idx1 = triangle.idx1 - triangle.idx2;
			}
		}
		else
		{
			return;
		}
	}

	FPoint3 p0 = dev_OVertices[triangle.idx0].p.xyz;
	FPoint3 p1 = dev_OVertices[triangle.idx1].p.xyz;
	FPoint3 p2 = dev_OVertices[triangle.idx2].p.xyz;

	//PERFORM CULLING
	if (cm == CullingMode::BackFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ p1 - p0 }, FVector3{ p2 - p0 }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue <= 0.f)
		{
			return;
		}
	}
	else if (cm == CullingMode::FrontFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ p1 - p0 }, FVector3{ p2 - p0 }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue >= 0.f)
		{
			return; //cull triangle
		}
	}
	//else if (cm == CullingMode::NoCulling)
	//{
	//}

	//PERFORM CLIPPING
	if (!IsTriangleVisible(p0, p1, p2))
	{
		return;
	}

	const float totalArea = abs(Cross(p0.xy - p1.xy, p2.xy - p0.xy));
	if (totalArea <= 0.f)
	{
		return; //cull away triangle
	}

	const unsigned int triangleIdx = atomicAdd(dev_NumVisibleTriangles, 1); //returns old value
	memcpy(&dev_Triangles[triangleIdx], &triangle, sizeof(TriangleIdx));

	//BINNING
	//TODO: make separate kernel bc register usage is too high

	//assign to correct bin(s), with globalTriangleIdx corresponding to its bin
	//each bin is a part of the window (have multiple atomic buffers for each bin)

	NDCToScreenSpace(p0.xy, p1.xy, p2.xy, width, height);
	const BoundingBox triangleBb = GetBoundingBox(p0.xy, p1.xy, p2.xy, width, height);

	int binMinX = triangleBb.xMin / binDim.x; //most left bin
	int binMinY = triangleBb.yMin / binDim.y; //most bottom bin
	int binMaxX = triangleBb.xMax / binDim.x; //most right bin
	int binMaxY = triangleBb.yMax / binDim.y; //most top bin
	binMinX = ClampFast(binMinX, 0, numBins.x);
	binMinY = ClampFast(binMinY, 0, numBins.y);
	binMaxX = ClampFast(binMaxX, 0, numBins.x - 1);
	binMaxY = ClampFast(binMaxY, 0, numBins.y - 1);
	//This creates a grid of bins that overlap with triangle boundingbox

	//TODO: get all middle bin points in triangle polygon
	//TODO: get all intersecting rectangles of bins from all 3 triangle edges
	//https://stackoverflow.com/questions/16203760/how-to-check-if-line-segment-intersects-a-rectangle

	for (int y{ binMinY }; y <= binMaxY; ++y)
	{
		for (int x{ binMinX }; x <= binMaxX; ++x)
		{
			//atomically add triangle to bin queue
			const unsigned int binIdx = x + y * numBins.x;

			bool isDone = false;
			do
			{
				isDone = (atomicCAS(&dev_BinQueueSizesMutexBuffer[binIdx], 0, 1) == 0);
				if (isDone)
				{
					//critical section
					const unsigned int currQueueSize = dev_BinQueueSizes[binIdx];
					if (currQueueSize < binQueueMaxSize)
					{
						//insert triangle Idx in queue
						dev_BinQueues[binIdx * binQueueMaxSize + currQueueSize] = triangleIdx;
						//increase bin's queue size
						++dev_BinQueueSizes[binIdx];
					}
					dev_BinQueueSizesMutexBuffer[binIdx] = 0;
					//end of critical section
				}
			} while (!isDone);
		}
	}
}

GPU_KERNEL
void TriangleBinnerKernel(TriangleIdx* dev_Triangles, unsigned int dev_NumVisibleTriangles, 
	unsigned int* dev_BinQueueSizes, unsigned int* dev_BinQueues, int* dev_BinQueueSizesMutexBuffer,
	const OVertex* dev_OVertices, 
	IPoint2 numBins, IPoint2 binDim, unsigned int binQueueMaxSize, unsigned int width, unsigned int height)
{
	//TODO: use shared memory to copy faster
	//data size of 9 shows no bank conflicts!
	//TriangleIdx can stay in local memory (registers)
	//DEPENDS ON REGISTER USAGE

	const unsigned int triangleIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (triangleIdx > dev_NumVisibleTriangles)
		return;

	const TriangleIdx triangle = dev_Triangles[triangleIdx];

	FPoint2 p0 = dev_OVertices[triangle.idx0].p.xy;
	FPoint2 p1 = dev_OVertices[triangle.idx1].p.xy;
	FPoint2 p2 = dev_OVertices[triangle.idx2].p.xy;

	//assign to correct bin(s), with globalTriangleIdx corresponding to its bin
	//each bin is a part of the window (have multiple atomic buffers for each bin)

	NDCToScreenSpace(p0, p1, p2, width, height);
	const BoundingBox triangleBb = GetBoundingBox(p0, p1, p2, width, height);

	int binMinX = triangleBb.xMin / binDim.x; //most left bin
	int binMinY = triangleBb.yMin / binDim.y; //most bottom bin
	int binMaxX = triangleBb.xMax / binDim.x; //most right bin
	int binMaxY = triangleBb.yMax / binDim.y; //most top bin
	binMinX = ClampFast(binMinX, 0, numBins.x);
	binMinY = ClampFast(binMinY, 0, numBins.y);
	binMaxX = ClampFast(binMaxX, 0, numBins.x - 1);
	binMaxY = ClampFast(binMaxY, 0, numBins.y - 1);
	//This creates a grid of bins that overlap with triangle boundingbox

	//TODO: get all middle bin points in triangle polygon
	//TODO: get all intersecting rectangles of bins from all 3 triangle edges
	//https://stackoverflow.com/questions/16203760/how-to-check-if-line-segment-intersects-a-rectangle

	for (int y{ binMinY }; y <= binMaxY; ++y)
	{
		for (int x{ binMinX }; x <= binMaxX; ++x)
		{
			//atomically add triangle to bin queue
			const unsigned int binIdx = x + y * numBins.x;
	
			bool isDone = false;
			do
			{
				isDone = (atomicCAS(&dev_BinQueueSizesMutexBuffer[binIdx], 0, 1) == 0);
				if (isDone)
				{
					//critical section
					const unsigned int currQueueSize = dev_BinQueueSizes[binIdx];
					if (currQueueSize < binQueueMaxSize)
					{
						//insert triangle Idx in queue
						dev_BinQueues[binIdx * binQueueMaxSize + currQueueSize] = triangleIdx;
						//increase bin's queue size
						++dev_BinQueueSizes[binIdx];
					}
					dev_BinQueueSizesMutexBuffer[binIdx] = 0;
					//end of critical section
				}
			} while (!isDone);
		}
	}
}

GPU_KERNEL
void RasterizerPerBinKernel(const TriangleIdx* __restrict__ const dev_Triangles, const OVertex* __restrict__ const dev_OVertices,
	PixelShade* dev_PixelShadeBuffer, int* dev_DepthBuffer, int* dev_DepthMutexBuffer, GPUTexturesCompact textures,
	unsigned int* dev_BinQueues, unsigned int* dev_BinQueueSizes, int* dev_BinQueueSizesMutexBuffer,
	IPoint2 binDim, unsigned int binQueueMaxSize, const unsigned int width, const unsigned int height)
{
	//each thread block has the same size of the queue size of a bin
	//each thread block processes 1 bin
	const unsigned int binIdx = blockIdx.x + blockIdx.y * gridDim.x;

	//PROGNOSIS: this will be slower and more threads will be "wasted" on the same potential triangle
	//BUT the main advantage is that big triangles will be eliminated and split up into smaller binned ones

	unsigned int queuedTriangleIdx;

	//TODO: since this is synchronised per block,
	//use syncthreads() instead of atomics

	//TODO: thread block should be size of queueSize
	//every thread can non-atomically fetch triangles from queue with thread index
	//they can atomically flag whether a triangle idx is valid or not
	//Problem: loop through entire queue array for every thread in Triangle Assembler
	//OR would not prove interesting bc TA uses FILO queue method
	//Other "Solution": every thread in TA can also only store triangle in their designated thread.idx spot in the queue array
	//This would mean that there's always only 2 threads atomically reading and writing 1 element of the queue (interesting enough???)
	//Don't forget about the dirty flagging thing tho

	//TODO: global/constant value that can be changed by host/CPU
	bool isQueueEmpty = false;

	bool isDone = false;
	do
	{
		isDone = (atomicCAS(&dev_BinQueueSizesMutexBuffer[binIdx], 0, 1) == 0);
		if (isDone)
		{
			//critical section
			const unsigned int currQueueSize = dev_BinQueueSizes[binIdx];
			//1: check whether the bin queue index is greater than 0
			if (currQueueSize > 0)
			{
				//2: fetch this index & decrease bin queue size
				queuedTriangleIdx = dev_BinQueues[binIdx * binQueueMaxSize + currQueueSize - 1];
				--dev_BinQueueSizes[binIdx];
			}
			else
			{
				isQueueEmpty = true;
			}
			dev_BinQueueSizesMutexBuffer[binIdx] = 0;
			//end of critical section
		}
	} while (!isDone);

	if (isQueueEmpty)
		return;

	//TODO: threads cooperatively store in shared memory
	//extern GPU_SHARED_MEMORY float sharedMemoryBuffer[];

	const TriangleIdx triangleIdx = dev_Triangles[queuedTriangleIdx];
	OVertex v0 = dev_OVertices[triangleIdx.idx0];
	OVertex v1 = dev_OVertices[triangleIdx.idx1];
	OVertex v2 = dev_OVertices[triangleIdx.idx2];

	//TODO: this is still set up for single-threaded rendering
	//3: rasterize triangle with binned bounding box (COARSE)

	NDCToScreenSpace(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
	const unsigned int minX = blockIdx.x * binDim.x;
	const unsigned int minY = blockIdx.y * binDim.y;
	const unsigned int maxX = minX + binDim.x;
	const unsigned int maxY = minY + binDim.y;
	const BoundingBox bb = GetBoundingBoxTiled(v0.p.xy, v1.p.xy, v2.p.xy, minX, minY, maxX, maxY);
	RasterizeTriangle(bb, v0, v1, v2, dev_DepthMutexBuffer, dev_DepthBuffer, dev_PixelShadeBuffer, width, textures);

	//BoundingBox bb;
	//bb.xMin = blockIdx.x * binDim.x;
	//bb.yMin = blockIdx.y * binDim.y;
	//bb.xMax = bb.xMin + binDim.x;
	//bb.yMax = bb.yMin + binDim.y;
	//RasterizeTriangle(bb, v0, v1, v2, dev_DepthMutexBuffer, dev_DepthBuffer, dev_PixelShadeBuffer, width, textures);

	//TODO: every thread in CTA processes a NxN tile of triangle in bin (fine rasterizer)
	//4: each thread in block does a 8x8 pixel area of triangle
	//each thread block does 1 triangle instead of each block does sizeofbinqueue triangles
}

GPU_KERNEL
void RasterizerPerTile(const TriangleIdx* __restrict__ const dev_Triangles, const OVertex* __restrict__ const dev_OVertices, unsigned int numTriangles,
	PixelShade* dev_PixelShadeBuffer, int* dev_DepthBuffer, int* dev_MutexBuffer, GPUTexturesCompact textures,
	const FVector3 camFwd, const CullingMode cm, const unsigned int width, const unsigned int height)
{
	//COARSE BINNER:
	//TODO: define coarse bins
	//=> screen divided up into pieces of same size
	//TODO: every bin has a fixed size array buffer of N indexes (triangle idxs)
	//TODO: process triangle and put into bin (atomically)
	//=> get boundingbox and % / xy values
	//TODO: every thread fetches a triangleIdx from bin queue (atomically)
	//=> check every bin if there's work to do

	//FINE BINNER:
	//TODO: define fine bins
	//=> subdivide coarse bins into smaller parts
	//TODO: every thread processes triangleIdx from coarse bin
	//TODO: coarse bin but just finer with more arrays???

	//TODO: put in LL
	//Every triangle has a (fixed) list of bin ids?
	//Every bin has a (fixed) list of triangleIds?

	constexpr unsigned int triangleSize = sizeof(OVertex) * 3 / 4;
	GPU_SHARED_MEMORY float sharedMemoryBuffer[224 * triangleSize]; //12096 elements == 48384 bytes

	//Every thread processes 1 single triangle for now
	const unsigned int globalTriangleIndex = threadIdx.x + blockIdx.x * blockDim.x;
	if (!(globalTriangleIndex < numTriangles))
		return;

	const TriangleIdx triangleIdx = dev_Triangles[globalTriangleIndex];

	//Shared memory is laid out in a big row-list
	const unsigned int triangleMemoryIdx = threadIdx.x * triangleSize;
	OVertex& v0 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx]);
	OVertex& v1 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx + sizeof(OVertex) / 4]);
	OVertex& v2 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx + sizeof(OVertex) / 4]);

	memcpy(&v0, &dev_OVertices[triangleIdx.idx0], sizeof(OVertex));
	memcpy(&v1, &dev_OVertices[triangleIdx.idx1], sizeof(OVertex));
	memcpy(&v2, &dev_OVertices[triangleIdx.idx2], sizeof(OVertex));

	const unsigned int minX = 0;
	const unsigned int minY = 0;
	const unsigned int maxX = 0;
	const unsigned int maxY = 0;

	NDCToScreenSpace(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
	const BoundingBox bb = GetBoundingBoxTiled(v0.p.xy, v1.p.xy, v2.p.xy, minX, minY, maxX, maxY);
	//Rasterize Screenspace triangle
	RasterizeTriangle(bb, v0, v1, v2, dev_MutexBuffer, dev_DepthBuffer, dev_PixelShadeBuffer, width, textures);
}

#pragma endregion

#pragma region DEPRECATED

GPU_KERNEL
void VertexShaderKernelPrototype(const IVertex* __restrict__ dev_IVertices, OVertex* dev_OVertices, unsigned int numVertices)
{
	//TODO: store matrix in top of shared memory for faster access
	//and offset shared memory access for threads
	//Potential problem: first warp might encounter bank conflicts?

	//OVERVIEW: each thread manages 1 attribute, this being a Vector3
	//The Output Position is being stored as a Vector3, with the W-elements stored in a separate shared memory row
	//32 x 3 = 96 => 32 vertex attributes per warp in a shared memory buffer of size 96
	//32 x 4 = 128 => 32 vertex OPositions per warp in a shared memory buffer of size 128

	//IVERTEX LAYOUT: POS3 - UV2 - NORM3 - TAN3 - COL3 (size: 14)
	//OVERTEX LAYOUT: POS4 - UV2 - NORM3 - TAN3 - VD3 - COL3 (size: 18)

	extern GPU_SHARED_MEMORY float sharedMemoryBuffer[];

	constexpr float* camPos = &dev_ConstMemory[0];
	constexpr float* worldMatrix = &dev_ConstMemory[6];
	constexpr float* WVPMatrix = &dev_ConstMemory[22];
	constexpr float* rotationMatrix = &dev_ConstMemory[38];

	//TODO: each thread should store 1 bank element at once for more coalesced access
	//instead of 1 thread storing 1 attribute from multiple banks to global memory

	//threadIdx.x: [0, 31]
	//threadIdx.y: [0, 7]
	const unsigned int vertexIdx = (blockIdx.x * (blockDim.x * blockDim.y)) + threadIdx.x + (threadIdx.y * blockDim.x);
	if (vertexIdx < numVertices)
	{
		const IVertex& iVertex = dev_IVertices[vertexIdx];
		OVertex* pOVertex = &dev_OVertices[vertexIdx];

		//TODO: store W component in local memory???
		//TODO: register usage is above 32, mainly due to matrixmath functions
		//also some used for shared memory and pointers

		// --- STEP 1 ---: Calculate Input Position to Ouput Position
		//for every 32 threads of vec3 (96 elements), a row of W elements is created (32 elements)

		//strided load into shared memory
		const unsigned int warpSharedMemIdx = threadIdx.y * 128;
		unsigned int sharedMemVecIdx = threadIdx.x * 3 + warpSharedMemIdx;
		float* pVecXYZ = &sharedMemoryBuffer[sharedMemVecIdx];
		const unsigned int sharedMemWIdx = threadIdx.x + 96 + warpSharedMemIdx;
		float* pVecW = &sharedMemoryBuffer[sharedMemWIdx];

		//memory is now coalesced
		memcpy(pVecXYZ, &iVertex.p, 12);
		//calculate NDC (WVP * v.p.xyzw / w)
		CalculateOutputPosXYZW(WVPMatrix, pVecXYZ, pVecW); //calculate NDC (WVPMat)
		//divide xyz by w
		pVecXYZ[0] /= *pVecW;
		pVecXYZ[1] /= *pVecW;
		pVecXYZ[2] /= *pVecW;

		//store into global memory
		memcpy(&pOVertex->p, pVecXYZ, 12); //copy vec3 elements
		pOVertex->p.w = *pVecW; //copy w element

		// --- STEP 2 ---: Calculate ViewDirection

		memcpy(pVecXYZ, &iVertex.p, 12);
		CalculateOutputPosXYZ(worldMatrix, pVecXYZ); //calculate worldposition (worldMat)

		pVecXYZ[0] -= camPos[0];
		pVecXYZ[1] -= camPos[1];
		pVecXYZ[2] -= camPos[2];
		Normalize(reinterpret_cast<FVector3&>(*pVecXYZ));

		memcpy(&pOVertex->vd, pVecXYZ, 12);
		__syncthreads(); //sync bc we don't use W value nomore

		//shared memory is now used 
		sharedMemVecIdx = threadIdx.x * 3 + threadIdx.y * 96;
		pVecXYZ = &sharedMemoryBuffer[sharedMemVecIdx];

		// --- STEP 3 ---: Calculate Input Normal to Output Normal

		memcpy(pVecXYZ, &iVertex.n, 12);
		MultiplyMatVec(rotationMatrix, pVecXYZ, 3, 3); //calculate normal
		memcpy(&pOVertex->n, pVecXYZ, 12);

		// --- STEP 4 ---: Calculate Input Tangent to Output Tangent

		memcpy(pVecXYZ, &iVertex.tan, 12);
		MultiplyMatVec(rotationMatrix, pVecXYZ, 3, 3); //calculate tangent
		memcpy(&pOVertex->tan, pVecXYZ, 12);

		// --- STEP 5 ---: Copy UV and Colour

		//COLOUR
		memcpy(pVecXYZ, &iVertex.c, 12);
		memcpy(&pOVertex->c, pVecXYZ, 12);

		//UV
		//pVecXYZ is "padded" UV to avoid bank conflicts
		memcpy(pVecXYZ, &iVertex.uv, 8);
		memcpy(&pOVertex->uv, pVecXYZ, 8);
	}
}

GPU_KERNEL
void TriangleAssemblerKernelOld(TriangleIdx* dev_Triangles, const unsigned int* __restrict__ const dev_IndexBuffer,
	unsigned int numIndices, const PrimitiveTopology pt)
{
	//TODO: perform culling/clipping etc.
	//advantage of TriangleAssembly: each thread stores 1 triangle
	//many threads == many triangles processed at once

	//TODO: global to shared to global?
	//TODO: local copy to global vs global to global?

	//'talk about naming gore, eh?
	const unsigned int indexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (pt == PrimitiveTopology::TriangleList)
	{
		const unsigned int correctedIdx = (indexIdx * 3);
		if (correctedIdx < numIndices)
		{
			memcpy(&dev_Triangles[indexIdx], &dev_IndexBuffer[correctedIdx], sizeof(TriangleIdx));
		}
	}
	else //if (pt == PrimitiveTopology::TriangleStrip)
	{
		if (indexIdx < numIndices - 2)
		{
			TriangleIdx triangle;
			memcpy(&triangle, &dev_IndexBuffer[indexIdx], sizeof(TriangleIdx));
			const bool isOdd = (indexIdx % 2);
			if (isOdd)
			{
				//swap without temp
				//TODO: what about indexes at UINT_MAX? => would just overflow and back
				triangle.idx1 = triangle.idx1 + triangle.idx2;
				triangle.idx2 = triangle.idx1 - triangle.idx2;
				triangle.idx1 = triangle.idx1 - triangle.idx2;
			}
			memcpy(&dev_Triangles[indexIdx], &triangle, sizeof(TriangleIdx));
		}
	}
}

GPU_KERNEL
void RasterizerPerPixelKernel(const TriangleIdx* __restrict__ dev_Triangles, const OVertex* __restrict__ const dev_OVertices, unsigned int numTriangles,
	PixelShade* dev_PixelShadeBuffer, int* dev_DepthBuffer, GPUTexturesCompact textures,
	const unsigned int width, const unsigned int height)
{
	//TODO: each thread represents a pixel
	//each thread loops through all triangles
	//triangles are stored in shared memory (broadcast)
	//advantage: thread only does 1 check per triangle w/o looping for all pixels 
	//=> O(n) n = numTriangles vs O(n^m) n = numTriangles m = numPixels
	//advantage: nomore atomic operations needed bc only 1 thread can write to 1 unique pixelIdx

	constexpr float* pCamFwd = &dev_ConstMemory[3];

	//extern GPU_SHARED_MEMORY float sharedMemoryBuffer[];

	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	const FPoint2 pixel{ float(x), float(y) };

	for (unsigned int i{}; i < numTriangles; ++i)
	{
		const TriangleIdx triangleIdx = dev_Triangles[i];

		//TODO: store in shared memory
		OVertex v0 = dev_OVertices[triangleIdx.idx0];
		OVertex v1 = dev_OVertices[triangleIdx.idx1];
		OVertex v2 = dev_OVertices[triangleIdx.idx2];

		if (!IsTriangleVisible(v0.p, v1.p, v2.p))
		{
			return;
		}

		NDCToScreenSpace(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
		const BoundingBox bb = GetBoundingBox(v0.p.xy, v1.p.xy, v2.p.xy, width, height);

		if (!IsPixelInBoundingBox(pixel, bb))
		{
			return;
		}

		//Rasterize pixel
		RasterizePixel(pixel, v0, v1, v2, dev_DepthBuffer, dev_PixelShadeBuffer, width, textures);
	}
}

GPU_KERNEL
void RasterizerPerTriangleKernelOld(const TriangleIdx* __restrict__ const dev_Triangles, const OVertex* __restrict__ const dev_OVertices, unsigned int numTriangles,
	PixelShade* dev_PixelShadeBuffer, int* dev_DepthBuffer, int* dev_MutexBuffer, GPUTexturesCompact textures,
	const FVector3 camFwd, const CullingMode cm, const unsigned int width, const unsigned int height)
{
	constexpr unsigned int triangleSize = sizeof(OVertex) * 3 / 4;
	extern GPU_SHARED_MEMORY float sharedMemoryBuffer[];

	//Every thread processes 1 single triangle for now
	const unsigned int globalTriangleIndex = threadIdx.x + blockIdx.x * blockDim.x;
	if (!(globalTriangleIndex < numTriangles))
		return;

	const TriangleIdx triangleIdx = dev_Triangles[globalTriangleIndex];

	//Shared memory is laid out in a big row-list
	//const unsigned int triangleMemoryIdx = threadIdx.x * triangleSize;
	//OVertex& v0 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx]);
	//OVertex& v1 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx + (sizeof(OVertex) / 4)]);
	//OVertex& v2 = reinterpret_cast<OVertex&>(sharedMemoryBuffer[triangleMemoryIdx + (sizeof(OVertex) / 4) * 2]);

	//memcpy(&v0, &dev_OVertices[triangleIdx.idx0], sizeof(OVertex));
	//memcpy(&v1, &dev_OVertices[triangleIdx.idx1], sizeof(OVertex));
	//memcpy(&v2, &dev_OVertices[triangleIdx.idx2], sizeof(OVertex));

	OVertex v0 = dev_OVertices[triangleIdx.idx0];
	OVertex v1 = dev_OVertices[triangleIdx.idx1];
	OVertex v2 = dev_OVertices[triangleIdx.idx2];

	//bool isDoubleSidedRendering = false;

	//is triangle visible according to cullingmode?
	if (cm == CullingMode::BackFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ v1.p - v0.p }, FVector3{ v2.p - v0.p }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue <= 0.f)
		{
			//if (isDoubleSidedRendering)
			//{
			//	OVertex origV1 = v1;
			//	v1 = v2;
			//	v2 = origV1;
			//}
			//else
			//{
			return; //cull triangle
		//}
		}
	}
	else if (cm == CullingMode::FrontFace)
	{
		const FVector3 faceNormal = GetNormalized(Cross(FVector3{ v1.p - v0.p }, FVector3{ v2.p - v0.p }));
		const float cullingValue = Dot(camFwd, faceNormal);
		if (cullingValue >= 0.f)
		{
			//if (isDoubleSidedRendering)
			//{
			//	OVertex origV1 = v1;
			//	v1 = v2;
			//	v2 = origV1;
			//}
			//else
			//{
			return; //cull triangle
		//}
		}
	}
	//else if (cm == CullingMode::NoCulling)
	//{
	//}

	if (!IsTriangleVisible(v0.p, v1.p, v2.p))
	{
		return;
	}

	NDCToScreenSpace(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
	const BoundingBox bb = GetBoundingBox(v0.p.xy, v1.p.xy, v2.p.xy, width, height);
	//Rasterize Screenspace triangle
	RasterizeTriangle(bb, v0, v1, v2, dev_MutexBuffer, dev_DepthBuffer, dev_PixelShadeBuffer, width, textures);
}

#pragma endregion

#pragma region TESTING

GPU_KERNEL
void TextureTestKernel(unsigned int* dev_FrameBuffer, GPUTexture texture, const unsigned int width, const unsigned int height)
{
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		float u = float(x) / width;
		float v = float(y) / height;
		//u *= texture.w;
		//v *= texture.h;
		//float uC = Clamp(float(x), 0.f, (float)texture.w);
		//float vC = Clamp(float(y), 0.f, (float)texture.h);
		//float sampleIdx = u + v * texture.w;
		//unsigned int sample = tex1Dfetch<unsigned int>(texture.dev_pTex, (int)sampleIdx);
		//remap uv's to stretch towards the window's dimensions
		unsigned int sample = tex2D<unsigned int>(texture.dev_pTex, u, v);
		RGBA rgba = sample;
		unsigned char b = rgba.b8;
		rgba.b8 = rgba.r8;
		rgba.r8 = b;
		dev_FrameBuffer[pixelIdx] = rgba.colour32;
	}
}

GPU_KERNEL
void DrawTextureGlobalKernel(unsigned int* dev_FrameBuffer, GPUTexture texture, bool isStretchedToWindow,
	SampleState sampleState, const unsigned int width, const unsigned int height)
{
	const unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		//remap uv's to stretch towards the window's dimensions
		FVector2 uv;
		uv.x = float(x);
		uv.y = float(y);
		if (isStretchedToWindow)
		{
			uv.x /= width;
			uv.y /= height;
		}
		else
		{
			uv.x /= texture.w;
			uv.y /= texture.h;
		}
		RGBColor sample = GPUTextureSampler::Sample(texture, uv, sampleState);
		RGBA rgba = sample;
		dev_FrameBuffer[pixelIdx] = rgba.colour32;
	}
}

#pragma endregion

#pragma endregion

#pragma region KERNEL LAUNCHERS

CPU_CALLABLE
void CUDARenderer::Clear(const RGBColor& colour)
{
	//TODO: async in stream
	{
		constexpr int depthBufferResetValue = INT_MAX;
		const dim3 numThreadsPerBlock{ 32, 32 };
		const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
		ClearDepthBufferKernel<<<numBlocks, numThreadsPerBlock>>>
			(dev_DepthBuffer, depthBufferResetValue, m_WindowHelper.Width, m_WindowHelper.Height);
	}

	{
		const size_t sizeInWords = m_WindowHelper.Width * m_WindowHelper.Height * (sizeof(PixelShade) / 4);
		constexpr unsigned int numThreadsPerBlock = 512;
		const unsigned int numBlocks = (unsigned int)(sizeInWords + numThreadsPerBlock - 1) / numThreadsPerBlock;
		ClearPixelShadeBufferKernel<<<numBlocks, numThreadsPerBlock>>>
			(dev_PixelShadeBuffer, sizeInWords);
	}

	{
		////UNNECESSARY STEP: pixelshade stage will overwrite this anyway + more kernel launch overhead
		//const RGBA rgba{ colour };
		//const dim3 numThreadsPerBlock{ 32, 32 };
		//const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
		//Needs to be called after ClearPixelShadeBufferKernel
		//ClearScreenKernel<<<numBlocks, numThreadsPerBlock>>>
		//	(dev_PixelShadeBuffer, m_WindowHelper.Width, m_WindowHelper.Height, rgba.colour32);
		////Not necessary, since we overwrite the entire buffer every frame anyway
		//ClearFrameBufferKernel<<<numBlocks, numThreadsPerBlock>>>
		//	(dev_FrameBuffer, m_WindowHelper.Width, m_WindowHelper.Height, rgba.colour32);
	}

	{
		////UNNECESSARY STEP: mutexbuffer should always revert to base initialized state, otherwise deadlocks would occur
		//const dim3 numThreadsPerBlock{ 32, 32 };
		//const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
		//ClearDepthMutexBufferKernel<<<numBlocks, numThreadsPerBlock>>>
		//	(dev_DepthMutexBuffer, m_WindowHelper.Width, m_WindowHelper.Height);
	}
}

CPU_CALLABLE
void CUDARenderer::VertexShader(const MeshIdentifier& mi)
{
	const unsigned int numVertices = mi.pMesh->GetVertexAmount();

	//constexpr unsigned int paddedSizeOfIVertex = sizeof(OVertex) / 4 + 1;
	//constexpr unsigned int sharedMemoryNeededPerThread = paddedSizeOfIVertex * 4;
	//constexpr unsigned int sharedMemoryNeededPerWarp = sharedMemoryNeededPerThread * 32;
	constexpr unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = (numVertices + (numThreadsPerBlock - 1)) / numThreadsPerBlock;
	//const unsigned int numSharedMemory = numThreadsPerBlock * sharedMemoryNeededPerThread;
	VertexShaderKernelNaive<<<numBlocks, numThreadsPerBlock>>>(
		dev_IVertexBuffer[mi.Idx], dev_OVertexBuffer[mi.Idx], numVertices);

	//NOTE: NOT FOR COMPUTE CAPABILITY 6.1, stats may be higher
	//Max amount of shared memory per block: 49152 (48Kbs)
	//Max amount of threads per block/CTA: 2048
	//Max amount of blocks (dim.x): 2^31 - 1
	//Max amount of blocks (dim.yz): 65535
	//Max amount of blocks per SM: 8

	////constexpr unsigned int maxSharedMemoryPerBlock = 49152;
	//constexpr unsigned int sharedMemoryNeededPerThread = 4 * sizeof(float);
	//constexpr unsigned int sharedMemoryNeededPerWarp = sharedMemoryNeededPerThread * 32;
	//const dim3 numThreadsPerBlock{ 32, 8 }; //256
	//const unsigned int numThreadsPerBlockTotal = numThreadsPerBlock.x * numThreadsPerBlock.y;
	//const unsigned int numBlocks = (numVertices + (numThreadsPerBlockTotal - 1)) / numThreadsPerBlockTotal;
	//const unsigned int numSharedMemory = numThreadsPerBlock.y * sharedMemoryNeededPerWarp;
	//VertexShaderKernelPrototype<<<numBlocks, numThreadsPerBlock, numSharedMemory>>>(
	//	dev_IVertexBuffer[mi.Idx], dev_OVertexBuffer[mi.Idx], numVertices);
}

CPU_CALLABLE
void CUDARenderer::TriangleAssembler(MeshIdentifier& mi, const FVector3& camFwd, const CullingMode cm)
{
	const unsigned int numTriangles = mi.TotalNumTriangles;
	const unsigned int numIndices = mi.pMesh->GetIndexAmount();
	const PrimitiveTopology topology = mi.pMesh->GetTopology();

	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = (numTriangles + (numThreadsPerBlock - 1)) / numThreadsPerBlock;
	TriangleAssemblerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], dev_IndexBuffer[mi.Idx], numIndices, dev_NumVisibleTriangles,
		dev_OVertexBuffer[mi.Idx], topology, cm, camFwd,
		m_WindowHelper.Width, m_WindowHelper.Height);

	//TriangleAssemblerKernelOld<<<numBlocks, numThreadsPerBlock>>>(
	//	dev_Triangles[mi.Idx], dev_IndexBuffer[mi.Idx], 
	//	numIndices, topology);
}

CPU_CALLABLE
void CUDARenderer::TriangleBinner(MeshIdentifier& mi)
{
	const unsigned int numTriangles = mi.VisibleNumTriangles;
	const IPoint2 numBins = { (int)m_BinQueues.NumQueuesX, (int)m_BinQueues.NumQueuesY };

	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = (numTriangles + (numThreadsPerBlock - 1)) / numThreadsPerBlock;
	TriangleBinnerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], mi.VisibleNumTriangles,
		dev_BinQueueSizes, dev_BinQueues, dev_BinQueueSizesMutexBuffer,
		dev_OVertexBuffer[mi.Idx],
		numBins, m_BinDim, m_BinQueues.QueueMaxSize, m_WindowHelper.Width, m_WindowHelper.Height);
}

CPU_CALLABLE
void CUDARenderer::TriangleAssemblerAndBinner(MeshIdentifier& mi, const FVector3& camFwd, const CullingMode cm)
{
	const unsigned int numIndices = mi.pMesh->GetIndexAmount();
	const PrimitiveTopology topology = mi.pMesh->GetTopology();
	const unsigned int numTriangles = mi.TotalNumTriangles;
	const IPoint2 numBins = { (int)m_BinQueues.NumQueuesX, (int)m_BinQueues.NumQueuesY };

	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = (numTriangles + (numThreadsPerBlock - 1)) / numThreadsPerBlock;
	TriangleAssemblerAndBinnerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], dev_IndexBuffer[mi.Idx], numIndices, dev_NumVisibleTriangles,
		dev_BinQueueSizes, dev_BinQueues, dev_BinQueueSizesMutexBuffer,
		dev_OVertexBuffer[mi.Idx], topology, cm, camFwd, 
		numBins, m_BinDim, m_BinQueues.QueueMaxSize, m_WindowHelper.Width, m_WindowHelper.Height);
}

CPU_CALLABLE
void CUDARenderer::Rasterizer(const MeshIdentifier& mi, const FVector3& camFwd, const CullingMode cm)
{
	const GPUTexturesCompact& textures = mi.Textures;

	constexpr unsigned int numThreadsPerBlock = 256;

#ifdef BINNING
	const dim3 numBlocks = { m_BinQueues.NumQueuesX, m_BinQueues.NumQueuesY };
	RasterizerPerBinKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], dev_OVertexBuffer[mi.Idx],
		dev_PixelShadeBuffer, dev_DepthBuffer, dev_DepthMutexBuffer, textures,
		dev_BinQueues, dev_BinQueueSizes, dev_BinQueueSizesMutexBuffer, m_BinDim,
		m_BinQueues.QueueMaxSize, m_WindowHelper.Width, m_WindowHelper.Height);
#else
	const unsigned int numTriangles = mi.VisibleNumTriangles;
	const unsigned int numBlocks = (numTriangles + (numThreadsPerBlock - 1)) / numThreadsPerBlock;
	RasterizerPerTriangleKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles[mi.Idx], dev_OVertexBuffer[mi.Idx], numTriangles,
		dev_PixelShadeBuffer, dev_DepthBuffer, dev_DepthMutexBuffer, textures,
		m_WindowHelper.Width, m_WindowHelper.Height);
#endif

	//SHARED MEMORY ATTEMPT
	//shared memory needed per triangle = 3 * OVertex (216 bytes)
	//shared memory per block: 49152 bytes
	//Thus 227.555 triangles per 48Kbs
	//32 * 7 = 224 (only 3.555 difference)
	//224 * 216 = 48384 bytes (768 bytes 'wasted')

	//constexpr unsigned int sizeOfTriangle = sizeof(OVertex) * 3;
	//constexpr unsigned int maxSharedMemoryPerBlock = 49152;
	//constexpr unsigned int numThreadsPerBlock = (maxSharedMemoryPerBlock / sizeOfTriangle) - (maxSharedMemoryPerBlock / sizeOfTriangle) % 32;
	//constexpr unsigned int numThreadsPerBlock = 128;
	//constexpr unsigned int numSharedMemory = numThreadsPerBlock * sizeOfTriangle;
	//const unsigned int numBlocks = (numTriangles + (numThreadsPerBlock - 1)) / numThreadsPerBlock;
	//RasterizerPerTriangleKernel<<<numBlocks, numThreadsPerBlock, numSharedMemory>>>(
	//	dev_Triangles[mi.Idx], dev_OVertexBuffer[mi.Idx], numTriangles,
	//	dev_PixelShadeBuffer, dev_DepthBuffer, dev_MutexBuffer, textures,
	//	m_WindowHelper.Width, m_WindowHelper.Height);

	//RasterizerPerTriangleKernelOld<<<numBlocks, numThreadsPerBlock, numSharedMemory>>>(
	//	dev_Triangles[mi.Idx], dev_OVertexBuffer[mi.Idx], numTriangles,
	//	dev_PixelShadeBuffer, dev_DepthBuffer, dev_MutexBuffer, textures,
	//	camFwd, cm, m_WindowHelper.Width, m_WindowHelper.Height);

	//const dim3 numThreadsPerBlock = { 16, 16 };
	//const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	//RasterizerPerPixelKernel<<<numBlocks, numThreadsPerBlock>>>(
	//	dev_Triangles[mi.Idx], dev_OVertexBuffer[mi.Idx], numTriangles,
	//	dev_PixelShadeBuffer, dev_DepthBuffer, textures,
	//	m_WindowHelper.Width, m_WindowHelper.Height);
}

CPU_CALLABLE
void CUDARenderer::PixelShader(SampleState sampleState, bool isDepthColour)
{
	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	PixelShaderKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_FrameBuffer, dev_PixelShadeBuffer, sampleState, isDepthColour,
		m_WindowHelper.Width, m_WindowHelper.Height);
}

CPU_CALLABLE
void CUDARenderer::DrawTexture(char* tP)
{
	SDL_Surface* pS = IMG_Load(tP);

	int w = pS->w;
	int h = pS->h;
	int bpp = pS->format->BytesPerPixel;
	unsigned int* buffer;
	size_t pitch{};
	CheckErrorCuda(hipMallocPitch((void**)&buffer, &pitch, w * bpp, h)); //2D array
	CheckErrorCuda(hipMemcpy2D(buffer, pitch, buffer, pitch, w * bpp, h, hipMemcpyHostToDevice));

	//hipChannelFormatDesc formatDesc = hipCreateChannelDesc<unsigned int>();

	hipResourceDesc resDesc{};
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = buffer;
	resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
	resDesc.res.pitch2D.desc.x = pS->format->BitsPerPixel;
	resDesc.res.pitch2D.width = w;
	resDesc.res.pitch2D.height = h;
	resDesc.res.pitch2D.pitchInBytes = pitch;

	hipTextureDesc texDesc{};
	texDesc.normalizedCoords = true; //able to sample texture with normalized uv coordinates
	texDesc.filterMode = hipFilterModePoint; //linear only supports float (and double) type
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex{};
	CheckErrorCuda(hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr));

	GPUTexture texture{};
	texture.dev_pTex = tex;
	texture.w = w;
	texture.h = h;
	texture.dev_TextureData = buffer;

	EnterValidRenderingState();

	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	TextureTestKernel<<<numBlocks, numThreadsPerBlock>>>(dev_FrameBuffer, texture, m_WindowHelper.Width, m_WindowHelper.Height);

	Present();

	//destroy texture object
	CheckErrorCuda(hipDestroyTextureObject(tex));

	SDL_FreeSurface(pS);

	//do not free buffer if it is meant to be reused
	CheckErrorCuda(hipFree(buffer));
}

CPU_CALLABLE
void CUDARenderer::DrawTextureGlobal(char* tp, bool isStretchedToWindow, SampleState sampleState)
{
	SDL_Surface* pS = IMG_Load(tp);

	int w = pS->w;
	int h = pS->h;
	int N = w * h;
	unsigned int* buffer;
	hipMalloc(&buffer, N * sizeof(unsigned int));
	hipMemcpy(buffer, pS->pixels, N * sizeof(unsigned int), hipMemcpyHostToDevice);

	EnterValidRenderingState();

	GPUTexture gpuTexture{};
	gpuTexture.dev_pTex = 0; //none
	gpuTexture.dev_TextureData = buffer;
	gpuTexture.w = w;
	gpuTexture.h = h;

	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	DrawTextureGlobalKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_FrameBuffer, gpuTexture, isStretchedToWindow, 
		sampleState, m_WindowHelper.Width, m_WindowHelper.Height);

	Present();

	SDL_FreeSurface(pS);

	hipFree(buffer);
}

CPU_CALLABLE
void CUDARenderer::WarmUp()
{
	ClearDepthBufferKernel<<<0, 0>>>(nullptr, 0, 0, 0);
	ClearFrameBufferKernel<<<0, 0>>>(nullptr, 0, 0, 0);
	ClearPixelShadeBufferKernel<<<0, 0>>>(nullptr, 0);
	ClearDepthMutexBufferKernel <<<0, 0>>>(nullptr, 0, 0);
	VertexShaderKernelNaive<<<0, 0>>>(nullptr, nullptr, 0);
	TriangleAssemblerKernelOld<<<0, 0>>>(nullptr, nullptr, 0, (PrimitiveTopology)0);
	RasterizerPerTriangleKernelOld<<<0, 0>>>(nullptr, nullptr, 0, nullptr, nullptr, nullptr, {}, {}, (CullingMode)0, 0, 0);
	PixelShaderKernel<<<0, 0>>>(nullptr, nullptr, SampleState(0), false, 0, 0);
}

#pragma endregion