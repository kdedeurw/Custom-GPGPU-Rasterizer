#include "hip/hip_runtime.h"
#include "PCH.h"
#include "CUDARenderer.cuh"
#include <vector>

//Project CUDA includes
#include "GPUTextureSampler.cuh"

//Project includes
#include "WindowHelper.h"
#include "SceneManager.h"
#include "SceneGraph.h"
#include "Mesh.h"
#include "Camera.h"
#include "Vertex.h"
#include "BoundingBox.h"
#include "GPUTextures.h"
#include "RGBRaw.h"

#pragma region STRUCT DECLARATIONS

struct RenderData
{
	FPoint3 camPos;
	FMatrix4 viewProjectionMatrix;
	FMatrix4 worldMatrix;
};

union RenderDataRaw
{
	//compiler BUG: attempting to reference a deleted function?
	float data[];
	RenderData renderData;
};

struct Triangle
{
	OVertex v0;
	OVertex v1;
	OVertex v2;
};

struct TrianglePtr
{
	OVertex* pV0;
	OVertex* pV1;
	OVertex* pV2;
};

#pragma endregion

#pragma region GLOBAL VARIABLES

//CONST DEVICE MEMORY - Does NOT have to be allocated or freed
GPU_CONST_MEMORY float dev_RenderData_const[sizeof(RenderData) / sizeof(float)]{};
//NOTE: cannot contain anything else besides primitive variables (int, float, etc.)

//DEVICE MEMORY - Does have to be allocated and freed
unsigned int* dev_FrameBuffer{};
float* dev_DepthBuffer{};
Triangle* dev_Triangles{};
int* dev_Mutex{};
std::vector<IVertex*> dev_IVertexBuffer{};
std::vector<unsigned int*> dev_IndexBuffer{};
std::vector<OVertex*> dev_OVertexBuffer{};

#pragma endregion

//TODO: use TrianglePtr again

//TODO: to counter global memory access sequencing
//allocate buffers/dynamic pools per BLOCK (per screen bin)
//Same goes with shared memory, but has bank conflicts

//--------------------------

CPU_CALLABLE CUDARenderer::CUDARenderer(const WindowHelper& windowHelper)
	: m_WindowHelper{ windowHelper }
	, m_NumTriangles{}
	, m_h_pFrameBuffer{}
	, m_MeshIdentifiers{}
{
	InitCUDARasterizer();
}

CPU_CALLABLE CUDARenderer::~CUDARenderer()
{
	CheckErrorCuda(DeviceSynchroniseCuda());
	FreeCUDARasterizer();
}

#pragma region MISC HELPER FUNCTIONS

CPU_CALLABLE std::string ToKbs(size_t bytes)
{
	const size_t toKbs = 1024;
	std::string output{ std::to_string(bytes / toKbs) + "Kb" };
	return output;
}

CPU_CALLABLE std::string ToMbs(size_t bytes)
{
	const size_t toMBs = 1024 * 1024;
	std::string output{ std::to_string(bytes / toMBs) + "Mb" };
	return output;
}

CPU_CALLABLE std::string ToGbs(size_t bytes)
{
	const size_t toGBs = 1024 * 1024 * 1024;
	std::string output{ std::to_string(bytes / toGBs) + "Gb" };
	return output;
}

BOTH_CALLABLE float GetMinElement(float val0, float val1, float val2)
{
	float min = val0;
	if (val1 < min)
		min = val1;
	if (val2 < min)
		min = val2;
	return min;
}

BOTH_CALLABLE float GetMaxElement(float val0, float val1, float val2)
{
	float max = val0;
	if (val1 > max)
		max = val1;
	if (val2 > max)
		max = val2;
	return max;
}

#pragma endregion

#pragma region CPU HELPER FUNCTIONS

CPU_CALLABLE void CUDARenderer::DisplayGPUSpecs(int deviceId)
{
	std::string yn{};

	std::cout << "\n---General---\n";
	hipDeviceProp_t prop;
	CheckErrorCuda(hipGetDeviceProperties(&prop, deviceId));
	std::cout << "Device detected: " << prop.name << '\n';
	std::cout << "Compute Capability: " << prop.major << '.' << prop.minor << '\n';
	std::cout << "Compute Mode: ";
	switch (prop.computeMode)
	{
	case hipComputeModeDefault:
		std::cout << "Default\n";
		break;
	case hipComputeModeExclusive:
		std::cout << "Exclusive\n";
		break;
	case hipComputeModeProhibited:
		std::cout << "Prohibited\n";
		break;
	case hipComputeModeExclusiveProcess:
		std::cout << "ExclusiveProcess\n";
		break;
	default:
		std::cout << "Undefined\n";
		break;
	}
	if (prop.isMultiGpuBoard)
	{
		std::cout << "Multi GPU setup: Yes\n";
		std::cout << "Multi GPU boardgroup ID: " << prop.multiGpuBoardGroupID << '\n';
	}
	else
	{
		std::cout << "Multi GPU setup: No\n";
	}
	std::cout << "Async Engine (DMA) count: " << prop.asyncEngineCount << '\n';
	yn = prop.deviceOverlap ? "Yes\n" : "No\n";
	std::cout << "Can concurrently copy memory between host and device while executing kernel: " << yn;
	switch (prop.asyncEngineCount)
	{
	case 0:
		std::cout << "Device cannot concurrently copy memory between host and device while executing a kernel\n";
		break;
	case 1:
		std::cout << "Device can concurrently copy memory between host and device while executing a kernel\n";
		break;
	case 2:
		std::cout << "Device can concurrently copy memory between host and device in both directions and execute a kernel at the same time\n";
		break;
	default:
		break;
	}
	yn = prop.concurrentKernels ? "Yes\n" : "No\n";
	std::cout << "Device supports executing multiple kernels within the same context simultaneously: " << yn;
	yn = prop.integrated ? "Yes\n" : "No\n";
	std::cout << "Integrated Graphics: " << yn;

	std::cout << "\n---Memory---\n";
	std::cout << "Total amount of Global Memory: " << ToMbs(prop.totalGlobalMem) << '\n';
	std::cout << "Total amount of Const Memory: " << ToKbs(prop.totalConstMem) << '\n';
	//size_t free{}, total{};
	//hipMemGetInfo(&free, &total);
	//std::cout << "Total amount of VRAM: " << total << '\n';
	//std::cout << "Free amount of VRAM: " << free << '\n';
	std::cout << "Shared Memory per Multiprocessor: " << ToKbs(prop.sharedMemPerMultiprocessor) << '\n';
	std::cout << "Shared Memory per Block: " << ToKbs(prop.sharedMemPerBlock) << '\n';

	std::cout << "Shared Memory Reserved by CUDA driver per Block: " << prop.reservedSharedMemPerBlock << " bytes\n";

	yn = prop.unifiedAddressing ? "Yes\n" : "No\n";
	std::cout << "Unified Addressing supported: " << yn;
	yn = prop.managedMemory ? "Yes\n" : "No\n";
	std::cout << "Managed Memory supported: " << yn;
	yn = prop.pageableMemoryAccess ? "Yes\n" : "No\n";
	std::cout << "Device can coherently access Pageable Memory (non-pinned memory): " << yn;
	yn = prop.pageableMemoryAccessUsesHostPageTables ? "Yes\n" : "No\n";
	std::cout << "Device can access pageable memory via host's page tables: " << yn;
	yn = prop.canMapHostMemory ? "Yes\n" : "No\n";
	std::cout << "Can Map host memory: " << yn;

	std::cout << "\n---Memory - Caching---\n";
	yn = prop.globalL1CacheSupported ? "Yes\n" : "No\n";
	std::cout << "Global L1 Cache Supported: " << yn;
	yn = prop.localL1CacheSupported ? "Yes\n" : "No\n";
	std::cout << "Local L1 Cache Supported: " << yn;
	std::cout << "L2 Cache Size: " << ToKbs(prop.l2CacheSize) << '\n';;
	std::cout << "Persisting L2 Cache Max Size: " << prop.persistingL2CacheMaxSize << " bytes\n";

	std::cout << "\n---Other---\n";
	std::cout << "ClockRate: " << prop.clockRate / 1000 << "Khz\n";
	std::cout << "Memory ClockRate: " << prop.memoryClockRate / 1000 << "Khz\n";
	std::cout << "Memory Pitch: " << prop.memPitch << " bytes\n";
	std::cout << "Maximum number of 32-bit registers per Multiprocessor: " << prop.regsPerMultiprocessor << "\n";
	std::cout << "Maximum number of 32-bit registers per Block: " << prop.regsPerBlock << "\n";

	std::cout << "\n---Thread specifications---\n";
	std::cout << "Max threads per Multiprocessor: " << prop.maxThreadsPerMultiProcessor << '\n';
	std::cout << "Max threads per Block: " << prop.maxThreadsPerBlock << '\n';
	std::cout << "Max threads Dimensions: X: " << prop.maxThreadsDim[0] << ", Y: " << prop.maxThreadsDim[1] << ", Z: " << prop.maxThreadsDim[2] <<'\n';
	std::cout << "Warp Size (in threads): " << prop.warpSize << '\n';
	std::cout << '\n';
}

CPU_CALLABLE void CUDARenderer::InitCUDARasterizer()
{
#ifdef _DEBUG
	DisplayGPUSpecs(0);
#endif

	size_t size{};
	const unsigned int width = m_WindowHelper.Width;
	const unsigned int height = m_WindowHelper.Height;

	//CUDAHOSTALLOC FLAGS
	/*
	hipHostMallocDefault: This flag's value is defined to be 0 and causes hipHostAlloc() to emulate hipHostMalloc().
	hipHostMallocPortable: The memory returned by this call will be considered as pinned memory by all CUDA contexts, not just the one that performed the allocation.
	hipHostMallocMapped: Maps the allocation into the CUDA address space. The device pointer to the memory may be obtained by calling hipHostGetDevicePointer().
	hipHostMallocWriteCombined: Allocates the memory as write-combined (WC).
	WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read efficiently by most CPUs.
	WC memory is a good option for buffers that will be written by the CPU and read by the device via mapped pinned memory or host->device transfers.
	*/

	//--->PINNED MEMORY<--- (HOST ONLY)
	//+ makes memory transactions between host and device significantly faster
	//- however this will allocate on host's RAM memory (in this case it would be 640 * 480 * 4 bytes == 1.2288Mb)
	size = sizeof(unsigned int);
	CheckErrorCuda(hipHostMalloc((void**)&m_h_pFrameBuffer, width * height * size));
	//CheckErrorCuda(hipHostAlloc((void**)&m_h_pFrameBuffer, width * height * size, hipHostMallocPortable));

	//CUDAHOSTREGISTER FLAGS
	/*
	hipHostRegisterDefault: On a system with unified virtual addressing, the memory will be both mapped and portable. 
	On a system with no unified virtual addressing, the memory will be neither mapped nor portable.
	hipHostRegisterPortable: The memory returned by this call will be considered as pinned memory by all CUDA contexts, not just the one that performed the allocation.
	hipHostRegisterMapped: Maps the allocation into the CUDA address space. The device pointer to the memory may be obtained by calling hipHostGetDevicePointer().
	hipHostRegisterIoMemory: The passed memory pointer is treated as pointing to some memory-mapped I/O space, 
	e.g. belonging to a third-party PCIe device, and it will marked as non cache-coherent and contiguous.
	hipHostRegisterReadOnly: The passed memory pointer is treated as pointing to memory that is considered read-only by the device.
	On platforms without hipDeviceAttributePageableMemoryAccessUsesHostPageTables, this flag is required in order to register memory mapped to the CPU as read-only.
	Support for the use of this flag can be queried from the device attribute cudaDeviceAttrReadOnlyHostRegisterSupported.
	Using this flag with a current context associated with a device that does not have this attribute set will cause hipHostRegister to error with hipErrorNotSupported.
	*/

	//Invalid Argument
	//SDL_LockSurface(m_WindowHelper.pBackBuffer);
	//size = sizeof(unsigned int);
	//CheckErrorCuda(hipHostRegister(m_WindowHelper.pBackBufferPixels, width * height * size, hipHostRegisterDefault));
	//SDL_UnlockSurface(m_WindowHelper.pBackBuffer);;

	//The framebuffer in device memory
	size = sizeof(unsigned int);
	CheckErrorCuda(hipFree(dev_FrameBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_FrameBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_FrameBuffer, 0, width * height * size));

	size = sizeof(float);
	CheckErrorCuda(hipFree(dev_DepthBuffer));
	CheckErrorCuda(hipMalloc((void**)&dev_DepthBuffer, width * height * size));
	CheckErrorCuda(hipMemset(dev_DepthBuffer, 0, width * height * size));

	size = sizeof(int);
	hipFree(dev_Mutex);
	hipMalloc((void**)&dev_Mutex, width * height * size);
	hipMemset(dev_Mutex, 0, width * height * size);

	//NOTE: can only set data PER BYTE
	//PROBLEM: setting each byte to UCHAR_MAX (255) is impossible, since floating point numbers work differently (-nan result)
	//	0		11111110	11111111111111111111111
	//	^			^				^
	//	sign	exponent		mantissa
	//			254 - 127     2 - 2 ^ (-23)
	// 340282346638528859811704183484516925440.0   // FLT_MAX
	// 340282366920938463463374607431768211456.0   // 2^128
	//https://stackoverflow.com/questions/16350955/interpreting-the-bit-pattern-of-flt-max
	//SOLUTION:
	//Option 1: allocate float[width*height] and initialize to FLT_MAX, then memcpy (wastes lots of memory)
	//Option 2: loop through entire dev_array and set each member to FLT_MAX (too many global accesses)
	//>Option 3<: interpret depth buffer invertedly, so a depthvalue of 1.f is closest, and 0.f is furthest away from camera
	//Option 4: initialize and reset depthbuffer through additional kernel call, however this would be a lot of global memory accesses
}

CPU_CALLABLE void CUDARenderer::AllocateMeshBuffers(const size_t numVertices, const size_t numIndices, int meshIdx)
{
	//If no sufficient space in vector, enlarge
	const size_t newSize = meshIdx + 1;
	if (newSize > dev_IVertexBuffer.size())
	{
		dev_IVertexBuffer.resize(newSize);
		dev_IndexBuffer.resize(newSize);
		dev_OVertexBuffer.resize(newSize);
	}

	//Free unwanted memory
	CheckErrorCuda(hipFree(dev_IVertexBuffer[meshIdx]));
	CheckErrorCuda(hipFree(dev_IndexBuffer[meshIdx]));
	CheckErrorCuda(hipFree(dev_OVertexBuffer[meshIdx]));

	//Allocate Input Vertex Buffer
	CheckErrorCuda(hipMalloc((void**)&dev_IVertexBuffer[meshIdx], numVertices * sizeof(IVertex)));
	//Allocate Index Buffer
	CheckErrorCuda(hipMalloc((void**)&dev_IndexBuffer[meshIdx], numIndices * sizeof(unsigned int)));
	//Allocate Ouput Vertex Buffer
	CheckErrorCuda(hipMalloc((void**)&dev_OVertexBuffer[meshIdx], numVertices * sizeof(OVertex)));
}

CPU_CALLABLE void CUDARenderer::CopyMeshBuffers(const std::vector<IVertex>& vertexBuffer, const std::vector<unsigned int>& indexBuffer, int meshIdx)
{
	//Copy Input Vertex Buffer
	CheckErrorCuda(hipMemcpy(dev_IVertexBuffer[meshIdx], &vertexBuffer[0], vertexBuffer.size() * sizeof(IVertex), hipMemcpyHostToDevice));
	//Copy Index Buffer
	CheckErrorCuda(hipMemcpy(dev_IndexBuffer[meshIdx], &indexBuffer[0], indexBuffer.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
}

CPU_CALLABLE void CUDARenderer::FreeMeshBuffers()
{
	for (size_t i{}; i < m_MeshIdentifiers.size(); ++i)
	{
		CheckErrorCuda(hipFree(dev_IVertexBuffer[i]));
		dev_IVertexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_IndexBuffer[i]));
		dev_IndexBuffer[i] = nullptr;
		CheckErrorCuda(hipFree(dev_OVertexBuffer[i]));
		dev_OVertexBuffer[i] = nullptr;
	}
	m_MeshIdentifiers.clear();
}

CPU_CALLABLE void CUDARenderer::FreeCUDARasterizer()
{
	//Free buffers
	CheckErrorCuda(hipFree(dev_FrameBuffer));
	dev_FrameBuffer = nullptr;

	//CheckErrorCuda(hipHostFree(m_WindowHelper.pBackBufferPixels));
	//m_WindowHelper.pBackBufferPixels = nullptr;

	CheckErrorCuda(hipHostFree(m_h_pFrameBuffer));
	m_h_pFrameBuffer = nullptr;

	CheckErrorCuda(hipFree(dev_DepthBuffer));
	dev_DepthBuffer = nullptr;

	hipFree(dev_Mutex);
	dev_Mutex = nullptr;

	CheckErrorCuda(hipFree(dev_Triangles));
	dev_Triangles = nullptr;

	FreeMeshBuffers();
}

CPU_CALLABLE void CUDARenderer::UpdateCameraDataAsync(const FPoint3& camPos, const FMatrix4& viewProjectionMatrix)
{
	//Update CamPos
	size_t numBytes = sizeof(camPos);
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_RenderData_const), camPos.data, numBytes, 0, hipMemcpyHostToDevice));
	//Update ViewProjectionMatrix
	const size_t numBytesOffset = numBytes;
	numBytes = sizeof(viewProjectionMatrix);
	CheckErrorCuda(hipMemcpyToSymbol(HIP_SYMBOL(dev_RenderData_const), viewProjectionMatrix.data, numBytes, numBytesOffset, hipMemcpyHostToDevice));

	//CameraDataRaw cameraData{};
	//cameraData.cameraData.camPos = camPos;
	//cameraData.cameraData.viewProjectionMatrix = viewProjectionMatrix;
	//const size_t numBytes = sizeof(CameraData);
	//void* dev_ptr{};
	//CheckErrorCuda(hipGetSymbolAddress((void**)&dev_ptr, dev_RenderData_const));
	//CheckErrorCuda(hipMemcpyAsync(dev_ptr, cameraData.data, numBytes, hipMemcpyHostToDevice));
}

CPU_CALLABLE void CUDARenderer::UpdateWorldMatrixDataAsync(const FMatrix4& worldMatrix)
{
	const size_t numBytes = sizeof(FMatrix4);
	const size_t numBytesOffset = sizeof(CameraData);
	CheckErrorCuda(hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_RenderData_const), worldMatrix.data, numBytes, numBytesOffset, hipMemcpyHostToDevice));

	//void* dev_ptr{};
	//CheckErrorCuda(hipGetSymbolAddress((void**)&dev_ptr, dev_RenderData_const));
	//CheckErrorCuda(hipMemcpyAsync(dev_ptr, worldMatrix.data, numBytes, hipMemcpyHostToDevice));
}

CPU_CALLABLE int CUDARenderer::EnterValidRenderingState()
{
	//https://wiki.libsdl.org/SDL_LockSurface
	int state = SDL_LockSurface(m_WindowHelper.pBackBuffer); //Set up surface for directly accessing the pixels
	//Clear screen and reset buffers
	Clear();
	return state;
}

CPU_CALLABLE void CUDARenderer::Present()
{
	//TODO: have Vertex Shader and Rasterizer run in parallel with cudamemcpy()
	const size_t size = m_WindowHelper.Width * m_WindowHelper.Height * sizeof(unsigned int);
	CheckErrorCuda(hipMemcpy(m_WindowHelper.pBackBufferPixels, dev_FrameBuffer, size, hipMemcpyDeviceToHost)); //We can directly read/write from pixelbuffer
	//memcpy(m_WindowHelper.pBackBufferPixels, m_WindowHelper.h_BackBufferPixels, size);
	SDL_UnlockSurface(m_WindowHelper.pBackBuffer); //Release a surface after directly accessing the pixels.
	SDL_BlitSurface(m_WindowHelper.pBackBuffer, 0, m_WindowHelper.pFrontBuffer, 0);
	SDL_UpdateWindowSurface(m_WindowHelper.pWindow); //Copy the window surface to the screen.
}

#pragma endregion

#pragma region GPU HELPER FUNCTIONS

GPU_CALLABLE OVertex GetNDCVertex(const IVertex& iVertex, const FPoint3& camPos,
	const FMatrix4& viewProjectionMatrix, const FMatrix4& worldMatrix)
{
	OVertex oVertex;

	const FPoint3 worldPosition{ worldMatrix * FPoint4{ iVertex.p } };
	const FMatrix4 worldViewProjectionMatrix = viewProjectionMatrix * worldMatrix;
	//const FMatrix3 rotationMatrix = (FMatrix3)worldMatrix;

	new (&oVertex.p) FPoint4{ worldViewProjectionMatrix * FPoint4{ iVertex.p } };
	oVertex.p.x /= oVertex.p.w;
	oVertex.p.y /= oVertex.p.w;
	oVertex.p.z /= oVertex.p.w;

	new (&oVertex.vd) const FVector3{ GetNormalized(worldPosition - camPos) };
	new (&oVertex.n) const FVector3{ (FMatrix3)worldMatrix * iVertex.n };
	new (&oVertex.tan) const FVector3{ (FMatrix3)worldMatrix * iVertex.tan };

	oVertex.uv = iVertex.uv;
	oVertex.c = iVertex.c;

	return oVertex;
}

GPU_CALLABLE bool EdgeFunction(const FPoint2& v, const FVector2& edge, const FPoint2& pixel, float& weight)
{
	// counter-clockwise
	const FVector2 vertexToPixel{ pixel - v };
	const float cross = Cross(edge, vertexToPixel);
	weight = cross;
	return cross < 0.f;
}

GPU_CALLABLE bool IsPixelInTriangle(FPoint4 rasterCoords[3], const FPoint2& pixel, float weights[3])
{
	const FPoint2& v0 = rasterCoords[0].xy;
	const FPoint2& v1 = rasterCoords[1].xy;
	const FPoint2& v2 = rasterCoords[2].xy;

	const FVector2 edgeA{ v0 - v1 };
	const FVector2 edgeB{ v1 - v2 };
	const FVector2 edgeC{ v2 - v0 };
	// counter-clockwise

	const float totalArea = Cross(edgeA, edgeC);

	{
		//// edgeA
		//FVector2 vertexToPixel{ pixel - v0 };
		//float cross = Cross(edgeA, vertexToPixel);
		//isInTriangle &= cross < 0.f;
		//// weight2 == positive cross of 'previous' edge, for v2 this is edgeA (COUNTER-CLOCKWISE)
		//weights[2] = cross / totalArea;

		//// edgeB
		//vertexToPixel = { pixel - v1 };
		//cross = Cross(edgeB, vertexToPixel);
		//isInTriangle &= cross < 0.f;
		//// weight1 (for v1 this is edgeB)
		//weights[1] = cross / totalArea;

		//// edgeC
		//vertexToPixel = { pixel - v2 };
		//cross = Cross(edgeC, vertexToPixel);
		//isInTriangle &= cross < 0.f;
		//// weight0 (for v0 this is edgeC)
		//weights[0] = cross / totalArea;

		//weights == inverted negative cross of 'previous' edge
		//weights[0] = Cross(-vertexToPixel, edgeC) / totalArea;
		//weights[1] = Cross(-vertexToPixel, edgeB) / totalArea;
		//weights[2] = Cross(-vertexToPixel, edgeA) / totalArea;
		// gives positive results because counter-clockwise
		//const float total = weights[0] + weights[1] + weights[2]; // total result equals 1
	}

	if (EdgeFunction(v0, edgeA, pixel, weights[2])) return false;
	if (EdgeFunction(v1, edgeB, pixel, weights[1])) return false;
	if (EdgeFunction(v2, edgeC, pixel, weights[0])) return false;
	weights[0] /= totalArea;
	weights[1] /= totalArea;
	weights[2] /= totalArea;

	return true;
}

GPU_CALLABLE bool DepthTest(float dev_DepthBuffer[], int dev_Mutex[], const size_t pixelIdx, float weights[3], float zInterpolated)
{
	//TODO: shared memory
	
	//Update depthbuffer atomically
	bool isDone = false;
	do
	{
		isDone = (atomicCAS(&dev_Mutex[pixelIdx], 0, 1) == 0);
		if (isDone)
		{
			if (zInterpolated < dev_DepthBuffer[pixelIdx]) //DEPTH BUFFER INVERTED INTERPRETATION
			{
				dev_DepthBuffer[pixelIdx] = zInterpolated;
			}
			dev_Mutex[pixelIdx] = 0;
		}
	} while (!isDone);
	return true;
	//atomicCAS
	/*
	//int atomicCAS(int* address, int compare, int val);
	//reads the 16 - bit, 32 - bit or 64 - bit word old located at the address address in global or shared memory, 
	//computes(old == compare ? val : old), and stores the result back to memory at the same address.
	//These three operations are performed in one atomic transaction.The function returns old(Compare And Swap).
	*/
}

GPU_CALLABLE bool FrustumTestVertex(const FPoint4& NDC)
{
	bool isOutside = false;
	isOutside |= (NDC.x < -1.f || NDC.x > 1.f);
	isOutside |= (NDC.y < -1.f || NDC.y > 1.f);
	isOutside |= (NDC.z < 0.f || NDC.z > 1.f);
	return isOutside;
}

GPU_CALLABLE bool FrustumTest(FPoint4 NDC[3])
{
	bool isOutside = false;
	isOutside |= FrustumTestVertex(NDC[0]);
	isOutside |= FrustumTestVertex(NDC[1]);
	isOutside |= FrustumTestVertex(NDC[2]);
	return isOutside;
}

GPU_CALLABLE void NDCToScreenSpace(FPoint4 rasterCoords[3], const unsigned int width, const unsigned int height)
{
	for (int i{}; i < 3; ++i)
	{
		rasterCoords[i].x = ((rasterCoords[i].x + 1) / 2) * width;
		rasterCoords[i].y = ((1 - rasterCoords[i].y) / 2) * height;
	}
}

GPU_CALLABLE BoundingBox GetBoundingBox(FPoint4 rasterCoords[3], const unsigned int width, const unsigned int height)
{
	BoundingBox bb;
	bb.xMin = (short)GetMinElement(rasterCoords[0].x, rasterCoords[1].x, rasterCoords[2].x) - 1; // xMin
	bb.yMin = (short)GetMinElement(rasterCoords[0].y, rasterCoords[1].y, rasterCoords[2].y) - 1; // yMin
	bb.xMax = (short)GetMaxElement(rasterCoords[0].x, rasterCoords[1].x, rasterCoords[2].x) + 1; // xMax
	bb.yMax = (short)GetMaxElement(rasterCoords[0].y, rasterCoords[1].y, rasterCoords[2].y) + 1; // yMax

	if (bb.xMin < 0) bb.xMin = 0; //clamp minX to Left of screen
	if (bb.yMin < 0) bb.yMin = 0; //clamp minY to Bottom of screen
	if (bb.xMax > width) bb.xMax = width; //clamp maxX to Right of screen
	if (bb.yMax > height) bb.yMax = height; //clamp maxY to Top of screen

	return bb;
}

GPU_CALLABLE GPU_INLINE RGBColor ShadePixel(const OVertex& oVertex, const GPUTextures& textures, SampleState sampleState, bool isDepthColour)
{
	return oVertex.c;

	RGBColor finalColour{};
	if (isDepthColour)
	{
		finalColour = RGBColor{ Remap(oVertex.p.z, 0.985f, 1.f), 0.f, 0.f }; // depth colour
		finalColour.ClampColor();
	}
	else
	{
		//TODO: textures lmao
		const RGBColor diffuseColour = GPUTextureSampler::Sample(textures.pDiff, oVertex.uv, sampleState);

		const RGBColor normalRGB = GPUTextureSampler::Sample(textures.pNorm, oVertex.uv, sampleState);
		FVector3 normal{ normalRGB.r, normalRGB.g, normalRGB.b };

		FVector3 binormal{ Cross(oVertex.tan, oVertex.n) };
		FMatrix3 tangentSpaceAxis{ oVertex.tan, binormal, oVertex.n };

		normal.x = 2.f * normal.x - 1.f;
		normal.y = 2.f * normal.y - 1.f;
		normal.z = 2.f * normal.z - 1.f;

		normal = tangentSpaceAxis * normal;

		//// light calculations
		//for (Light* pLight : sm.GetSceneGraph()->GetLights())
		//{
		//	const FVector3& lightDir{ pLight->GetDirection(FPoint3{}) };
		//	const float observedArea{ Dot(-normal, lightDir) };
		//
		//	if (observedArea < 0.f)
		//		continue;
		//
		//	const RGBColor biradiance{ pLight->GetBiradiance(FPoint3{}) };
		//	// swapped direction of lights
		//
		//	// phong
		//	const FVector3 reflectV{ Reflect(lightDir, normal) };
		//	//Normalize(reflectV);
		//	const float angle{ Dot(reflectV, oVertex.vd) };
		//	const RGBColor specularSample{ textures.pSpec->Sample(oVertex.uv, sampleState) };
		//	const RGBColor phongSpecularReflection{ specularSample * powf(angle, textures.pGloss->Sample(oVertex.uv, sampleState).r * 25.f) };
		//
		//	//const RGBColor lambertColour{ diffuseColour * (RGBColor{1.f,1.f,1.f} - specularSample) };
		//	//const RGBColor lambertColour{ (diffuseColour / float(E_PI)) * (RGBColor{1.f,1.f,1.f} - specularSample) };
		//	const RGBColor lambertColour{ (diffuseColour * specularSample) / float(E_PI) }; //severely incorrect result, using diffusecolour for now
		//	// Lambert diffuse == incoming colour multiplied by diffuse coefficient (1 in this case) divided by Pi
		//	finalColour += biradiance * (diffuseColour + phongSpecularReflection) * observedArea;
		//}
		finalColour.ClampColor();
	}
	return finalColour;
}

#pragma endregion

#pragma region KERNELS
//Kernel launch params:	numBlocks, numThreadsPerBlock, numSharedMemoryBytes, stream

GPU_KERNEL void ResetDepthBuffer(float dev_DepthBuffer[], const unsigned int width, const unsigned int height)
{
	//TODO: too many global accesses
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_DepthBuffer[pixelIdx] = 0.f; //FLT_MAX DEPTHBUFFER INVERTED INTERPRETATION
	}
}

GPU_KERNEL void ClearFrameBuffer(unsigned int dev_FrameBuffer[], const unsigned int width, const unsigned int height, unsigned int colour)
{
	//TODO: too many global accesses
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		dev_FrameBuffer[pixelIdx] = colour;
	}
}

GPU_KERNEL void VertexShaderKernel(IVertex dev_IVertices[], OVertex dev_OVertices[], const size_t numVertices,
	const FPoint3 camPos, const FMatrix4 viewProjectionMatrix, const FMatrix4 worldMatrix)
{
	const unsigned int vertexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vertexIdx < numVertices)
	{
		const IVertex& iV = dev_IVertices[vertexIdx];
		//TODO: store in shared memory
		const OVertex oV = GetNDCVertex(iV, camPos, viewProjectionMatrix, worldMatrix);
		dev_OVertices[vertexIdx] = std::move(oV);
	}
	//TODO: coalesced global memory copy

	//TODO: no data race here, find a way to make ALL THREADS write async
}

GPU_KERNEL void TriangleAssemblerKernel(Triangle dev_Triangles[], OVertex dev_OVertices[], unsigned int dev_IndexBuffer[], const size_t numIndices, 
	const PrimitiveTopology pt)
{
	//'talk about naming gore, eh?
	const unsigned int indexIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	//triangles usually exist out of 3 vertices (sarcasm)
	if (pt == PrimitiveTopology::TriangleList)
	{
		if (indexIdx < numIndices / 3)
		{
			Triangle triangle;
			//TODO: copy???
			const unsigned int correctedIdx = indexIdx + 2;
			//triangle.pV0 = &dev_OVertices[dev_IndexBuffer[correctedIdx - 2]];
			//triangle.pV1 = &dev_OVertices[dev_IndexBuffer[correctedIdx - 1]];
			//triangle.pV2 = &dev_OVertices[dev_IndexBuffer[correctedIdx]];
			triangle.v0 = dev_OVertices[dev_IndexBuffer[correctedIdx - 2]];
			triangle.v1 = dev_OVertices[dev_IndexBuffer[correctedIdx - 1]];
			triangle.v2 = dev_OVertices[dev_IndexBuffer[correctedIdx]];
			dev_Triangles[indexIdx] = triangle;
		}
	}
	else //if (pt == PrimitiveTopology::TriangleStrip)
	{
		if (indexIdx < numIndices - 2)
		{
			Triangle triangle;
			//TODO: copy???
			const bool isOdd = indexIdx % 2 != 0;
			const unsigned int idx0{ dev_IndexBuffer[indexIdx] };
			const unsigned int idx1 = isOdd ? dev_IndexBuffer[indexIdx + 2] : dev_IndexBuffer[indexIdx + 1];
			const unsigned int idx2 = isOdd ? dev_IndexBuffer[indexIdx + 1] : dev_IndexBuffer[indexIdx + 2];
			//triangle.pV0 = &dev_OVertices[dev_IndexBuffer[idx0]];
			//triangle.pV1 = &dev_OVertices[dev_IndexBuffer[idx1]];
			//triangle.pV2 = &dev_OVertices[dev_IndexBuffer[idx2]];
			triangle.v0 = dev_OVertices[dev_IndexBuffer[idx0]];
			triangle.v1 = dev_OVertices[dev_IndexBuffer[idx1]];
			triangle.v2 = dev_OVertices[dev_IndexBuffer[idx2]];
			dev_Triangles[indexIdx] = triangle;
		}
	}
}

GPU_KERNEL void RasterizerKernel(Triangle dev_Triangles[], const size_t numTriangles, unsigned int dev_FrameBuffer[], float dev_DepthBuffer[], int dev_Mutex[],
	GPUTextures& textures, SampleState sampleState, bool isDepthColour, const unsigned int width, const unsigned int height)
{
	//TODO: use shared memory, then coalescened copy
	//e.g. single bin buffer in single shared memory
	//GPU_SHARED_MEMORY float test[sizeof(RenderData)];

	//TODO: use binning, each bin their AABBs (and checks) (bin rasterizer)

	//Every thread processes 1 single triangle for now
	const unsigned int triangleIndex = threadIdx.x + blockIdx.x * blockDim.x;
	if (triangleIndex < numTriangles)
	{
		//TODO: remove copy?
		Triangle triangle = dev_Triangles[triangleIndex];
		//FPoint4 rasterCoords[3]{ triangle.pV0->v, triangle.pV1->v, triangle.pV2->v };
		FPoint4 rasterCoords[3]{ triangle.v0.p, triangle.v1.p, triangle.v2.p };

		//TODO: add early out in triangle assembler?
		//Or clip
		const bool isOutsideFrustum = FrustumTest(rasterCoords);
		if (isOutsideFrustum)
			return;

		NDCToScreenSpace(rasterCoords, width, height);
		const BoundingBox bb = GetBoundingBox(rasterCoords, width, height);
		//Rasterize Screenspace triangle
			 
		//TODO: 1 thread per triangle is bad for performance, use binning
		//Loop over all pixels in bounding box
		for (unsigned short y = bb.yMin; y < bb.yMax; ++y)
		{
			for (unsigned short x = bb.xMin; x < bb.xMax; ++x)
			{
				const FPoint2 pixel{ float(x), float(y) };
				float weights[3];
				if (IsPixelInTriangle(rasterCoords, pixel, weights))
				{
					const size_t pixelIdx = x + y * width;
					const float zInterpolated = (weights[0] * rasterCoords[0].z) + (weights[1] * rasterCoords[1].z) + (weights[2] * rasterCoords[2].z);
					//if (DepthTest(dev_DepthBuffer, dev_Mutex, pixelIdx, weights, zInterpolated))
					{
						OVertex oVertex;

						//const OVertex& v0 = *triangle.pV0;
						//const OVertex& v1 = *triangle.pV1;
						//const OVertex& v2 = *triangle.pV2;
						const OVertex& v0 = triangle.v0;
						const OVertex& v1 = triangle.v1;
						const OVertex& v2 = triangle.v2;

						const float wInterpolated = (weights[0] * v0.p.w) + (weights[1] * v1.p.w) + (weights[2] * v2.p.w);

						new (&oVertex.p) FPoint4{ pixel, zInterpolated, wInterpolated };

						new (&oVertex.uv) FVector2{
							weights[0] * (v0.uv.x / rasterCoords[0].w) + weights[1] * (v1.uv.x / rasterCoords[1].w) + weights[2] * (v2.uv.x / rasterCoords[2].w),
							weights[0] * (v0.uv.y / rasterCoords[0].w) + weights[1] * (v1.uv.y / rasterCoords[1].w) + weights[2] * (v2.uv.y / rasterCoords[2].w) };
						oVertex.uv *= wInterpolated;

						new (&oVertex.n) FVector3{
								weights[0] * (v0.n.x / rasterCoords[0].w) + weights[1] * (v1.n.x / rasterCoords[1].w) + weights[2] * (v2.n.x / rasterCoords[2].w),
								weights[0] * (v0.n.y / rasterCoords[0].w) + weights[1] * (v1.n.y / rasterCoords[1].w) + weights[2] * (v2.n.y / rasterCoords[2].w),
								weights[0] * (v0.n.z / rasterCoords[0].w) + weights[1] * (v1.n.z / rasterCoords[1].w) + weights[2] * (v2.n.z / rasterCoords[2].w) };
						oVertex.n *= wInterpolated;

						new (&oVertex.tan) const FVector3{
							weights[0] * (v0.tan.x / rasterCoords[0].w) + weights[1] * (v1.tan.x / rasterCoords[1].w) + weights[2] * (v2.tan.x / rasterCoords[2].w),
							weights[0] * (v0.tan.y / rasterCoords[0].w) + weights[1] * (v1.tan.y / rasterCoords[1].w) + weights[2] * (v2.tan.y / rasterCoords[2].w),
							weights[0] * (v0.tan.z / rasterCoords[0].w) + weights[1] * (v1.tan.z / rasterCoords[1].w) + weights[2] * (v2.tan.z / rasterCoords[2].w) };

						new (&oVertex.vd) FVector3{
						weights[0] * (v0.vd.y / rasterCoords[0].w) + weights[1] * (v1.vd.y / rasterCoords[1].w) + weights[2] * (v2.vd.y / rasterCoords[2].w),
						weights[0] * (v0.vd.x / rasterCoords[0].w) + weights[1] * (v1.vd.x / rasterCoords[1].w) + weights[2] * (v2.vd.x / rasterCoords[2].w),
						weights[0] * (v0.vd.z / rasterCoords[0].w) + weights[1] * (v1.vd.z / rasterCoords[1].w) + weights[2] * (v2.vd.z / rasterCoords[2].w) };
						Normalize(oVertex.vd);

						new (&oVertex.c) const RGBColor{
							weights[0] * (v0.c.r / rasterCoords[0].w) + weights[1] * (v1.c.r / rasterCoords[1].w) + weights[2] * (v2.c.r / rasterCoords[2].w),
							weights[0] * (v0.c.g / rasterCoords[0].w) + weights[1] * (v1.c.g / rasterCoords[1].w) + weights[2] * (v2.c.g / rasterCoords[2].w),
							weights[0] * (v0.c.b / rasterCoords[0].w) + weights[1] * (v1.c.b / rasterCoords[1].w) + weights[2] * (v2.c.b / rasterCoords[2].w) };

						//Pixel Shading
						//const RGBColor colour = ShadePixel(oVertex, textures, sampleState, isDepthColour);
						const RGBA rgba{ oVertex.c };
						dev_FrameBuffer[pixelIdx] += rgba.colour;
					}
				}
			}
		}
	}
}

#pragma region DEPRECATED

//DEPRECATED
GPU_KERNEL void PixelShaderKernel(unsigned int dev_FrameBuffer[], GPUTextures textures, SampleState sampleState, 
	bool isDepthColour, const unsigned int width, const unsigned int height)
{
	/*
	const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		const unsigned int pixelIdx = x + y * width;
		const OVertex& oVertex = dev_PixelShaderBuffer[pixelIdx]; //copy or ref?
		const RGBColor colour = ShadePixel(oVertex, textures, sampleState, isDepthColour);
		//store individual bytes from 32-bit format colour (RGBA)
		const unsigned char rgba[4] = { (unsigned char)(colour.r * 255.f), (unsigned char)(colour.g * 255.f), (unsigned char)(colour.b * 255.f), UCHAR_MAX };
		//convert to 4-byte RGBA value
		memcpy(&dev_FrameBuffer[pixelIdx], rgba, sizeof(unsigned int));
	}
	*/
}

#pragma endregion

#pragma endregion

#pragma region KERNEL LAUNCHERS

CPU_CALLABLE void CUDARenderer::Clear(const RGBColor& colour)
{
	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	ResetDepthBuffer<<<numBlocks, numThreadsPerBlock>>>
		(dev_DepthBuffer, m_WindowHelper.Width, m_WindowHelper.Height);
	const RGBA rgba{ colour };
	ClearFrameBuffer<<<numBlocks, numThreadsPerBlock>>>
		(dev_FrameBuffer, m_WindowHelper.Width, m_WindowHelper.Height, rgba.colour);
}

CPU_CALLABLE void CUDARenderer::VertexShader(const MeshIdentifier& mi, const FPoint3& camPos, const FMatrix4& viewProjectionMatrix, const FMatrix4& worldMatrix)
{
	const size_t numVertices = mi.pMesh->GetVertices().size();
	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocksForVertices = (numVertices + numThreadsPerBlock - 1) / numThreadsPerBlock;
	VertexShaderKernel<<<numBlocksForVertices, numThreadsPerBlock>>>(
		dev_IVertexBuffer[mi.Idx], dev_OVertexBuffer[mi.Idx], numVertices,
		camPos, viewProjectionMatrix, worldMatrix);
}

CPU_CALLABLE void CUDARenderer::TriangleAssembler(const MeshIdentifier& mi)
{
	const size_t numIndices = mi.pMesh->GetIndexes().size();
	const PrimitiveTopology topology = mi.pMesh->GetTopology();
	//TODO: change launch parameters
	//TIP: 1 thread per triangle, so # of threads =~= numIndices / 3
	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = ((unsigned int)numIndices + numThreadsPerBlock - 1) / numThreadsPerBlock;
	TriangleAssemblerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles, dev_OVertexBuffer[mi.Idx], dev_IndexBuffer[mi.Idx], numIndices, topology);
}

CPU_CALLABLE void CUDARenderer::Rasterizer(GPUTextures& textures, SampleState sampleState, bool isDepthColour)
{
	const unsigned int numThreadsPerBlock = 256;
	const unsigned int numBlocks = ((unsigned int)m_NumTriangles - 1) / numThreadsPerBlock + 1;
	//const size_t numSharedMemoryBytes = sizeof(RenderData);
	RasterizerKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_Triangles, m_NumTriangles,
		dev_FrameBuffer, dev_DepthBuffer, dev_Mutex,
		textures, sampleState, isDepthColour,
		m_WindowHelper.Width, m_WindowHelper.Height);
}

#pragma region DEPRECATED

CPU_CALLABLE void CUDARenderer::PixelShader(GPUTextures& textures, SampleState sampleState, bool isDepthColour)
{
	/*
	//TODO: if ever want to reuse again, get OVertex buffer
	const dim3 numThreadsPerBlock{ 16, 16 };
	const dim3 numBlocks{ m_WindowHelper.Width / numThreadsPerBlock.x, m_WindowHelper.Height / numThreadsPerBlock.y };
	PixelShaderKernel<<<numBlocks, numThreadsPerBlock>>>(
		dev_FrameBuffer, dev_PixelShaderBuffer, textures,
		sampleState, isDepthColour,
		m_WindowHelper.Width, m_WindowHelper.Height);
	*/
}

#pragma endregion

#pragma endregion

#pragma region PUBLIC FUNCTIONS

CPU_CALLABLE void CUDARenderer::LoadScene(const SceneGraph* pSceneGraph)
{
	m_NumTriangles = 0;
	FreeMeshBuffers(); //!must be called before MeshIdentifiers.clear()!
	const std::vector<Mesh*>& pMeshes = pSceneGraph->GetObjects();
	for (const Mesh* pMesh : pMeshes)
	{
		MeshIdentifier mi{};
		mi.Idx = m_MeshIdentifiers.size();
		mi.pMesh = pMesh;
		m_MeshIdentifiers.push_back(mi);

		const std::vector<IVertex>& vertexBuffer = pMesh->GetVertices();
		const std::vector<unsigned int>& indexBuffer = pMesh->GetIndexes();
		const size_t numVertices = vertexBuffer.size();
		const size_t numIndices = indexBuffer.size();
		const PrimitiveTopology topology = pMesh->GetTopology();
		const FMatrix4& worldMat = pMesh->GetWorldMatrix();

		AllocateMeshBuffers(numVertices, numIndices, mi.Idx);
		CopyMeshBuffers(vertexBuffer, indexBuffer, mi.Idx);

		switch (topology)
		{
		case PrimitiveTopology::TriangleList:
			m_NumTriangles += numIndices / 3;
			break;
		case PrimitiveTopology::TriangleStrip:
			m_NumTriangles += numIndices - 2;
			break;
		default:
			break;
		}
	}

	CheckErrorCuda(hipFree(dev_Triangles)); //Free unwanted memory
	//Allocate device memory for entire range of triangles
	CheckErrorCuda(hipMalloc((void**)&dev_Triangles, m_NumTriangles * sizeof(Triangle)));
}

CPU_CALLABLE void CUDARenderer::Render(const SceneManager& sm, const Camera* pCamera)
{
#ifdef _DEBUG
	if (EnterValidRenderingState())
		exit(1);
#else
	EnterValidRenderingState();
#endif

	//Render Data
	const bool isDepthColour = sm.IsDepthColour();
	const SampleState sampleState = sm.GetSampleState();

	//Camera Data
	const FPoint3& camPos = pCamera->GetPos();
	const FMatrix4 lookatMatrix = pCamera->GetLookAtMatrix();
	const FMatrix4 viewMatrix = pCamera->GetViewMatrix(lookatMatrix);
	const FMatrix4 projectionMatrix = pCamera->GetProjectionMatrix();
	const FMatrix4 viewProjectionMatrix = projectionMatrix * viewMatrix;

	//TODO: RENDERDATA NOT NEEDED, SINCE PASSED IN BY VALUE TO KERNEL
	//Otherwise every thread in a kernel needs to access its global device memory
	//RenderData renderData{};

	//TODO: random illegal memory access BUG
	//Update global memory for camera's matrices
	//UpdateCameraDataAsync(camPos, viewProjectionMatrix);

	//SceneGraph Data
	const SceneGraph* pSceneGraph = sm.GetSceneGraph();
	const std::vector<Mesh*>& pObjects = pSceneGraph->GetObjects();

	//TODO: create big coalesced memory array of buffer(s)?
	for (const MeshIdentifier& mi : m_MeshIdentifiers)
	{
		//Mesh Data
		const Mesh* pMesh = pObjects[mi.Idx];
		const FMatrix4& worldMat = pMesh->GetWorldMatrix();

		//TODO: random illegal memory access BUG
		//Update global memory for mesh's worldmatrix
		//UpdateWorldMatrixDataAsync(worldMat);
		//hipDeviceSynchronize();

		//---STAGE 1---:  Perform Output Vertex Assembling
		//TODO: async & streams
		//TODO: find out what order is best, for cudaDevCpy and Malloc
		VertexShader(mi, camPos, viewProjectionMatrix, worldMat);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 1---

		//---STAGE 2---:  Perform Triangle Assembling
		TriangleAssembler(mi);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 2---

		//---STAGE 3---: Peform Triangle Rasterization & Pixel Shading
		const Textures& textures = pMesh->GetTextures();
		GPUTextures gpuTextures{};
		Rasterizer(gpuTextures, sampleState, isDepthColour);
		CheckErrorCuda(hipDeviceSynchronize());
		//---END STAGE 3---
	}

	//TODO: parallel copies (streams & async)
	//Swap out buffers and update window
	Present();
}

#pragma endregion