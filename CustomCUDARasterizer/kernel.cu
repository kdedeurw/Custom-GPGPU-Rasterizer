﻿#include "PCH.h"

//External includes
#include "vld.h"
#include "SDL.h"
#include "SDL_surface.h"
#undef main

//Standard includes
#include <iostream>
#include <algorithm>

//Project includes
#include "ETimer.h"
#include "ERenderer.h"
#include "EMath.h"
#include "EMathUtilities.h"
#include "ERGBColor.h"
#include "Camera.h"
#include "EventManager.h"
#include "Vertex.h"
#include "Mesh.h"
#include "Texture.h"
#include "ObjParser.h"
#include "SceneManager.h"
#include "SceneGraph.h"
#include "DirectionalLight.h"

//Choose which GPU to run on, change this on a multi-GPU system. (Default is 0, for single-GPU systems)
hipError_t SetDeviceCuda(int deviceId = 0);
//Calls hipGetLastError, this checks for any errors launching the kernel
hipError_t CheckErrorCuda();
//Calls hipDeviceSynchronize, this waits for the kernel to finish, and returns any errors encountered during the launch.
hipError_t DeviceSynchroniseCuda();
//Calls hipDeviceReset, this must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
hipError_t DeviceResetCuda();

void Init()
{
	std::vector<SceneGraph*> pSceneGraphs{};

	{
		// SceneGraph 1
		SceneGraph* pSceneGraph = new SceneGraph{};
		{
			// Mesh 1
			std::vector<IVertex> vertices{
				IVertex{ FPoint3{ 0.f, 2.f, 0.f }, FVector2{} },
				IVertex{ FPoint3{ -1.f, 0.f, 0.f }, FVector2{}},
				IVertex{ FPoint3{ 1.f, 0.f, 0.f }, FVector2{} } };
			std::vector<int> indices{ 0, 1, 2 };
			const std::string texPaths[4]{ "", "", "", "" };
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleList };
			pSceneGraph->AddMesh(pMesh);
		}
		{
			// Mesh 2
			std::vector<IVertex> vertices{
				IVertex{ FPoint3{ 0.f, 4.f, -2.f }, FVector2{}, FVector3{1.f,1.f,1.f}, RGBColor{1.f, 0.f, 0.f} },
				IVertex{ FPoint3{ -3.f, -2.f, -2.f }, FVector2{}, FVector3{1.f,1.f,1.f}, RGBColor{0.f, 1.f, 0.f} },
				IVertex{ FPoint3{ 3.f, -2.f, -2.f }, FVector2{}, FVector3{1.f,1.f,1.f}, RGBColor{0.f, 0.f, 1.f} } };
			std::vector<int> indices{ 0, 1, 2 };
			const std::string texPaths[4]{ "", "", "", "" };
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleList };
			pSceneGraph->AddMesh(pMesh);
		}
		pSceneGraph->AddLight(new DirectionalLight{ RGBColor{1.f, 1.f, 1.f}, 2.f, FVector3{ 0.577f, -0.577f, -0.577f } });
		pSceneGraphs.push_back(pSceneGraph);
	}

	{
		// SceneGraph 2
		SceneGraph* pSceneGraph = new SceneGraph{};
		std::vector<IVertex> vertices{
			IVertex{FPoint3{-3, 3, -2}, FVector2{0, 0}}, IVertex{FPoint3{0, 3, -2}, FVector2{0.5f, 0}}, IVertex{FPoint3{3, 3, -2}, FVector2{1, 0}},
			IVertex{FPoint3{-3, 0, -2}, FVector2{0, 0.5f}}, IVertex{FPoint3{0, 0, -2}, FVector2{0.5f, 0.5f}}, IVertex{FPoint3{3, 0, -2}, FVector2{1, 0.5f}},
			IVertex{FPoint3{-3, -3, -2}, FVector2{0, 1}}, IVertex{FPoint3{0, -3, -2}, FVector2{0.5f, 1}}, IVertex{FPoint3{3, -3, -2}, FVector2{1, 1}} };
		// shared vertices among both quads (duh they're the same quad)
		const std::string texPaths[4]{ "Resources/uv_grid_2.png", "", "", "" };
		{
			// Mesh 1
			std::vector<int> indices{ 0, 3, 1,
										3, 4, 1,
										1, 4, 2,
										4, 5, 2,
										3, 6, 4,
										6, 7, 4,
										4, 7, 5,
										7, 8, 5, }; // obviously a list
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleList };
			//pSceneGraph->AddMesh(pMesh);
		}
		{
			// Mesh 2
			std::vector<int> indices{ 0, 3, 1, 4, 2, 5, 5, 3, 3, 6, 4, 7, 5, 8 }; // strip
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleStrip };
			pSceneGraph->AddMesh(pMesh);
			pSceneGraph->AddLight(new DirectionalLight{ RGBColor{1.f, 1.f, 1.f}, 2.f, FVector3{ 0.577f, -0.577f, -0.577f } });
		}
		pSceneGraphs.push_back(pSceneGraph);
	}

	{
		// SceneGraph 3 // TukTuk
		SceneGraph* pSceneGraph = new SceneGraph{};
		{
			// Mesh 1
			ObjParser parser{ "Resources/tuktuk.obj" };
			parser.SetInvertYAxis(true);
			parser.ReadFromObjFile();
			std::vector<IVertex> vertices{ *parser.GetVertexBuffer() };
			std::vector<int> indices{ parser.GetIndexBuffer() };
			const std::string texPaths[4]{ "Resources/tuktuk.png", "", "", "" };
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleList, 1.f };
			pSceneGraph->AddMesh(pMesh);
			pSceneGraph->AddLight(new DirectionalLight{ RGBColor{1.f, 1.f, 1.f}, 2.f, FVector3{ 0.577f, -0.577f, -0.577f } });
		}
		pSceneGraphs.push_back(pSceneGraph);
	}

	{
		// SceneGraph 4 // Bunny
		SceneGraph* pSceneGraph = new SceneGraph{};
		{
			// Mesh 1
			ObjParser parser{ "Resources/lowpoly_bunny.obj" };
			parser.ReadFromObjFile();
			std::vector<IVertex> vertices{ *parser.GetVertexBuffer() };
			std::vector<int> indices{ parser.GetIndexBuffer() };
			const std::string texPaths[4]{ "", "", "", "" };
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleList };
			pSceneGraph->AddMesh(pMesh);
			pSceneGraph->AddLight(new DirectionalLight{ RGBColor{1.f, 1.f, 1.f}, 2.f, FVector3{ 0.577f, -0.577f, -0.577f } });
		}
		pSceneGraphs.push_back(pSceneGraph);
	}

	{
		// SceneGraph 5 // Vehicle
		SceneGraph* pSceneGraph = new SceneGraph{};
		{
			// Mesh 1
			ObjParser parser{ "Resources/vehicle.obj" };
			parser.SetInvertYAxis(true);
			parser.ReadFromObjFile();
			std::vector<IVertex> vertices{ *parser.GetVertexBuffer() };
			std::vector<int> indices{ parser.GetIndexBuffer() };
			const std::string texPaths[4]{ "Resources/vehicle_diffuse.png", "Resources/vehicle_normal.png", "Resources/vehicle_specular.png", "Resources/vehicle_gloss.png" };
			Mesh* pMesh = new Mesh{ vertices, indices, texPaths, Mesh::PrimitiveTopology::TriangleList, 1.f };
			pSceneGraph->AddMesh(pMesh);
			pSceneGraph->AddLight(new DirectionalLight{ RGBColor{1.f, 1.f, 1.f}, 2.f, FVector3{ 0.577f, -0.577f, -0.577f } });
		}
		pSceneGraphs.push_back(pSceneGraph);
	}

	SceneManager& sm = *SceneManager::GetInstance();
	for (SceneGraph* pSceneGraph : pSceneGraphs)
	{
		sm.AddSceneGraph(pSceneGraph);
	}

	EventManager::GetInstance(); // initializing instance
}

void ShutDown(SDL_Window* pWindow)
{
	SDL_DestroyWindow(pWindow);
	SDL_Quit();
}

int main(int argc, char* args[])
{
    hipError_t cudaStatus{};
    cudaStatus = SetDeviceCuda();

	//Unreferenced parameters
	(void)argc;
	(void)args;

	//Create window + surfaces
	SDL_Init(SDL_INIT_VIDEO);

	const uint32_t width = 640;
	const uint32_t height = 480;
	SDL_Window* pWindow = SDL_CreateWindow(
		"Rasterizer - Kristof Dedeurwaerder",
		SDL_WINDOWPOS_UNDEFINED,
		SDL_WINDOWPOS_UNDEFINED,
		width, height, 0);

	if (!pWindow)
		return 1;

	//Initialize "framework"
	Elite::Timer* pTimer = new Elite::Timer();
	Elite::Renderer* pRenderer = new Elite::Renderer(pWindow);

	Camera::CreateInstance(FPoint3{ 0.f, 5.f, 65.f }, 45.f);
	Camera::GetInstance()->SetAspectRatio(float(width), float(height));
	EventManager& em = *EventManager::GetInstance();
	SceneManager& sm = *SceneManager::GetInstance();

	Init();

	//Start loop
	pTimer->Start();
	bool isLooping = true;
	bool takeScreenshot = false;
	float printTimer = 0.f;
	float deltaTime = 0.001f;
	while (isLooping)
	{
		//--------- Get input events ---------
		em.ProcessInputs(isLooping, takeScreenshot, pTimer->GetElapsed());

		//--------- Render ---------
		pRenderer->Render();

		//--------- Timer ---------
		pTimer->Update();
		printTimer += pTimer->GetElapsed();
		if (printTimer >= 1.f)
		{
			printTimer = 0.f;
			std::cout << "FPS: " << pTimer->GetFPS() << std::endl;
		}

		//--------- Update Meshes ---------
		sm.Update(pTimer->GetElapsed());

		//Save screenshot after full render
		if (takeScreenshot)
		{
			if (!pRenderer->SaveBackbufferToImage())
				std::cout << "Screenshot saved!" << std::endl;
			else
				std::cout << "Something went wrong. Screenshot not saved!" << std::endl;
			takeScreenshot = false;
		}
	}
	pTimer->Stop();

	//Shutdown "framework"
	delete pRenderer;
	delete pTimer;

	// delete singleton objects
	delete Camera::GetInstance();
	delete SceneManager::GetInstance();
	delete EventManager::GetInstance();

	ShutDown(pWindow);
	return 0;
}

hipError_t SetDeviceCuda(int deviceId)
{
    hipError_t cudaStatus = hipSetDevice(deviceId);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    return cudaStatus;
}

hipError_t CheckErrorCuda()
{
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "latest kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    return cudaStatus;
}

hipError_t DeviceSynchroniseCuda()
{
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d !\n", cudaStatus);
    }
    return cudaStatus;
}

hipError_t DeviceResetCuda()
{
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
    }
    return cudaStatus;
}