#include "hip/hip_runtime.h"
#include "PCH.h"

//Project CUDA includes
#include "GPUTextureSampler.cuh"

BOTH_CALLABLE static
float GetMinElement(float val0, float val1, float val2)
{
	float min = val0;
	if (val1 < min)
		min = val1;
	if (val2 < min)
		min = val2;
	return min;
}

BOTH_CALLABLE static
float GetMaxElement(float val0, float val1, float val2)
{
	float max = val0;
	if (val1 > max)
		max = val1;
	if (val2 > max)
		max = val2;
	return max;
}

GPU_CALLABLE GPU_INLINE static
float EdgeFunction(const FPoint2& v, const FVector2& edge, const FPoint2& pixel)
{
	// clockwise
	const FVector2 vertexToPixel{ pixel - v };
	return Cross(vertexToPixel, edge);
}

GPU_CALLABLE static
bool IsPixelInTriangle(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2, const FPoint2& pixel, float weights[3])
{
	const FVector2 edgeA{ v1.xy - v0.xy };
	const FVector2 edgeB{ v2.xy - v1.xy };
	const FVector2 edgeC{ v0.xy - v2.xy };
	// clockwise
	//const FVector2 edgeA{ v0.xy - v1.xy };
	//const FVector2 edgeB{ v1.xy - v2.xy };
	//const FVector2 edgeC{ v2.xy - v0.xy };
	// counter-clockwise

	weights[2] = EdgeFunction(v0.xy, edgeA, pixel);
	weights[0] = EdgeFunction(v1.xy, edgeB, pixel);
	weights[1] = EdgeFunction(v2.xy, edgeC, pixel);

	return weights[0] >= 0.f && weights[1] >= 0.f && weights[2] >= 0.f;
}

GPU_CALLABLE static
bool IsAllXOutsideFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return	(v0.x < -1.f && v1.x < -1.f && v2.x < -1.f) ||
		(v0.x > 1.f && v1.x > 1.f && v2.x > 1.f);
}

GPU_CALLABLE static
bool IsAllYOutsideFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return	(v0.y < -1.f && v1.y < -1.f && v2.y < -1.f) ||
		(v0.y > 1.f && v1.y > 1.f && v2.y > 1.f);
}

GPU_CALLABLE static
bool IsAllZOutsideFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return	(v0.z < 0.f && v1.z < 0.f && v2.z < 0.f) ||
		(v0.z > 1.f && v1.z > 1.f && v2.z > 1.f);
}

GPU_CALLABLE static
bool IsTriangleVisible(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	// Solution to FrustumCulling bug
	//	   if (all x values are < -1.f or > 1.f) AT ONCE, cull
	//	|| if (all y values are < -1.f or > 1.f) AT ONCE, cull
	//	|| if (all z values are < 0.f or > 1.f) AT ONCE, cull
	return(!IsAllXOutsideFrustum(v0, v1, v2)
		&& !IsAllYOutsideFrustum(v0, v1, v2)
		&& !IsAllZOutsideFrustum(v0, v1, v2));
}

GPU_CALLABLE static
bool IsVertexInFrustum(const FPoint4& NDC)
{
	return!((NDC.x < -1.f || NDC.x > 1.f) ||
		(NDC.y < -1.f || NDC.y > 1.f) ||
		(NDC.z < 0.f || NDC.z > 1.f));
}

GPU_CALLABLE static
bool IsTriangleInFrustum(const FPoint4& v0, const FPoint4& v1, const FPoint4& v2)
{
	return(IsVertexInFrustum(v0)
		|| IsVertexInFrustum(v1)
		|| IsVertexInFrustum(v2));
	//TODO: bug, triangles gets culled when zoomed in, aka all 3 vertices are outside of frustum
}

BOTH_CALLABLE static
BoundingBox GetBoundingBox(const FPoint2& v0, const FPoint2& v1, const FPoint2& v2, const unsigned int width, const unsigned int height)
{
	BoundingBox bb;
	bb.xMin = (short)GetMinElement(v0.x, v1.x, v2.x) - 1; // xMin
	bb.yMin = (short)GetMinElement(v0.y, v1.y, v2.y) - 1; // yMin
	bb.xMax = (short)GetMaxElement(v0.x, v1.x, v2.x) + 1; // xMax
	bb.yMax = (short)GetMaxElement(v0.y, v1.y, v2.y) + 1; // yMax

	if (bb.xMin < 0) bb.xMin = 0; //clamp minX to Left of screen
	if (bb.yMin < 0) bb.yMin = 0; //clamp minY to Bottom of screen
	if (bb.xMax > width) bb.xMax = width; //clamp maxX to Right of screen
	if (bb.yMax > height) bb.yMax = height; //clamp maxY to Top of screen

	return bb;
}

BOTH_CALLABLE GPU_INLINE static
void NDCToScreenSpace(FPoint2& v0, FPoint2& v1, FPoint2& v2, const unsigned int width, const unsigned int height)
{
	v0.x = ((v0.x + 1) / 2) * width;
	v0.y = ((1 - v0.y) / 2) * height;
	v1.x = ((v1.x + 1) / 2) * width;
	v1.y = ((1 - v1.y) / 2) * height;
	v2.x = ((v2.x + 1) / 2) * width;
	v2.y = ((1 - v2.y) / 2) * height;
}

GPU_CALLABLE GPU_INLINE static
OVertex GetNDCVertex(const IVertex& __restrict__ iVertex, const FMatrix4& wvpMat, const FMatrix4& worldMat, const FMatrix3& rotMat, const FPoint3& camPos)
{
	OVertex oVertex;
	oVertex.p = wvpMat * FPoint4{ iVertex.p };
	oVertex.p.x /= oVertex.p.w;
	oVertex.p.y /= oVertex.p.w;
	oVertex.p.z /= oVertex.p.w;

	oVertex.n = rotMat * iVertex.n;
	oVertex.tan = rotMat * iVertex.tan;

	const FPoint3 worldPosition{ worldMat * FPoint4{ iVertex.p } };
	oVertex.vd = GetNormalized(worldPosition - camPos);

	oVertex.uv = iVertex.uv;
	oVertex.c = iVertex.c;

	return oVertex;
}

#pragma region DEPRECATED

GPU_CALLABLE static
bool IsPixelInTriangle(const RasterTriangle& triangle, const FPoint2& pixel, float weights[3])
{
	return IsPixelInTriangle(triangle.v0, triangle.v1, triangle.v2, pixel, weights);
}

GPU_CALLABLE static
bool IsAllXOutsideFrustum(const RasterTriangle& triangle)
{
	return IsAllXOutsideFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsAllYOutsideFrustum(const RasterTriangle& triangle)
{
	return IsAllYOutsideFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsAllZOutsideFrustum(const RasterTriangle& triangle)
{
	return IsAllZOutsideFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsTriangleVisible(const RasterTriangle& triangle)
{
	return IsTriangleVisible(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
bool IsTriangleInFrustum(const RasterTriangle& triangle)
{
	return IsTriangleInFrustum(triangle.v0, triangle.v1, triangle.v2);
}

GPU_CALLABLE static
void NDCToScreenSpace(RasterTriangle& triangle, const unsigned int width, const unsigned int height)
{
	NDCToScreenSpace(triangle.v0.xy, triangle.v1.xy, triangle.v2.xy, width, height);
}

GPU_CALLABLE static
BoundingBox GetBoundingBox(const RasterTriangle& triangle, const unsigned int width, const unsigned int height)
{
	return GetBoundingBox(triangle.v0.xy, triangle.v1.xy, triangle.v2.xy, width, height);
}

#pragma endregion

GPU_CALLABLE static
bool IsPixelInBoundingBox(const FPoint2& pixel, const BoundingBox& bb)
{
	return pixel.x < bb.xMin || pixel.x > bb.xMax || pixel.y < bb.yMin || pixel.y > bb.yMax;
}

GPU_CALLABLE GPU_INLINE static
unsigned int GetStridedIdxByOffset(unsigned int globalDataIdx, unsigned int vertexStride, unsigned int valueStride, unsigned int offset = 0)
{
	//what value in row of [0, valueStride] + what vertex globally + element offset
	return (threadIdx.x % valueStride) + (globalDataIdx / valueStride) * vertexStride + offset;
}

GPU_CALLABLE static
void PerformDepthTestAtomic(int* dev_DepthBuffer, int* dev_DepthMutexBuffer, const unsigned int pixelIdx, float zInterpolated, PixelShade* dev_PixelShadeBuffer, const PixelShade& pixelShade)
{
	//Update depthbuffer atomically
	bool isDone = false;
	do
	{
		isDone = (atomicCAS(&dev_DepthMutexBuffer[pixelIdx], 0, 1) == 0);
		if (isDone)
		{
			//critical section
			if (zInterpolated < dev_DepthBuffer[pixelIdx])
			{
				dev_DepthBuffer[pixelIdx] = zInterpolated;
				dev_PixelShadeBuffer[pixelIdx] = pixelShade;
			}
			dev_DepthMutexBuffer[pixelIdx] = 0;
			//end of critical section
		}
	} while (!isDone);
}

GPU_CALLABLE static
void RasterizePixel(const FPoint2& pixel, const OVertex& v0, const OVertex& v1, const OVertex& v2,
	int* dev_DepthBuffer, PixelShade* dev_PixelShadeBuffer, unsigned int width, const GPUTexturesCompact& textures)
{
	const float v0InvDepth = 1.f / v0.p.w;
	const float v1InvDepth = 1.f / v1.p.w;
	const float v2InvDepth = 1.f / v2.p.w;

	float weights[3];
	if (IsPixelInTriangle(v0.p, v1.p, v2.p, pixel, weights))
	{
		const float totalArea = abs(Cross(v0.p.xy - v1.p.xy, v0.p.xy - v2.p.xy));
		weights[0] /= totalArea;
		weights[1] /= totalArea;
		weights[2] /= totalArea;

		const float zInterpolated = (weights[0] * v0.p.z) + (weights[1] * v1.p.z) + (weights[2] * v2.p.z);

		//peform early depth test
		if (zInterpolated < 0 || zInterpolated > 1.f)
			return;

		const float wInterpolated = 1.f / (v0InvDepth * weights[0] + v1InvDepth * weights[1] + v2InvDepth * weights[2]);

		//create pixelshade object (== fragment)
		PixelShade pixelShade;

		//depthbuffer visualisation
		pixelShade.zInterpolated = zInterpolated;
		pixelShade.wInterpolated = wInterpolated;

		//uv
		pixelShade.uv.x = weights[0] * (v0.uv.x * v0InvDepth) + weights[1] * (v1.uv.x * v1InvDepth) + weights[2] * (v2.uv.x * v2InvDepth);
		pixelShade.uv.y = weights[0] * (v0.uv.y * v0InvDepth) + weights[1] * (v1.uv.y * v1InvDepth) + weights[2] * (v2.uv.y * v2InvDepth);
		pixelShade.uv *= wInterpolated;

		//normal
		pixelShade.n.x = weights[0] * (v0.n.x * v0InvDepth) + weights[1] * (v1.n.x * v1InvDepth) + weights[2] * (v2.n.x * v2InvDepth);
		pixelShade.n.y = weights[0] * (v0.n.y * v0InvDepth) + weights[1] * (v1.n.y * v1InvDepth) + weights[2] * (v2.n.y * v2InvDepth);
		pixelShade.n.z = weights[0] * (v0.n.z * v0InvDepth) + weights[1] * (v1.n.z * v1InvDepth) + weights[2] * (v2.n.z * v2InvDepth);
		pixelShade.n *= wInterpolated;

		//tangent
		pixelShade.tan.x = weights[0] * (v0.tan.x * v0InvDepth) + weights[1] * (v1.tan.x * v1InvDepth) + weights[2] * (v2.tan.x * v2InvDepth);
		pixelShade.tan.y = weights[0] * (v0.tan.y * v0InvDepth) + weights[1] * (v1.tan.y * v1InvDepth) + weights[2] * (v2.tan.y * v2InvDepth);
		pixelShade.tan.z = weights[0] * (v0.tan.z * v0InvDepth) + weights[1] * (v1.tan.z * v1InvDepth) + weights[2] * (v2.tan.z * v2InvDepth);

		//view direction
		pixelShade.vd.x = weights[0] * (v0.vd.x * v0InvDepth) + weights[1] * (v1.vd.x * v1InvDepth) + weights[2] * (v2.vd.x * v2InvDepth);
		pixelShade.vd.y = weights[0] * (v0.vd.y * v0InvDepth) + weights[1] * (v1.vd.y * v1InvDepth) + weights[2] * (v2.vd.y * v2InvDepth);
		pixelShade.vd.z = weights[0] * (v0.vd.z * v0InvDepth) + weights[1] * (v1.vd.z * v1InvDepth) + weights[2] * (v2.vd.z * v2InvDepth);
		Normalize(pixelShade.vd);

		//colour
		const RGBColor interpolatedColour{
			weights[0] * v0.c.r + weights[1] * v1.c.r + weights[2] * v2.c.r,
			weights[0] * v0.c.g + weights[1] * v1.c.g + weights[2] * v2.c.g,
			weights[0] * v0.c.b + weights[1] * v1.c.b + weights[2] * v2.c.b };
		pixelShade.colour = RGBA::GetRGBAFromColour(interpolatedColour).colour32;

		//store textures
		pixelShade.textures = textures;

		//multiplying z value by a INT_MAX because atomicCAS only accepts ints
		const int scaledZ = zInterpolated * INT_MAX;

		const unsigned int pixelIdx = (unsigned int)pixel.x + (unsigned int)pixel.y * width;

		if (scaledZ < dev_DepthBuffer[pixelIdx])
		{
			dev_DepthBuffer[pixelIdx] = scaledZ;
			dev_PixelShadeBuffer[pixelIdx] = pixelShade;
		}
	}
}

GPU_CALLABLE static
void RasterizePixelAtomic(const FPoint2& pixel, const OVertex& v0, const OVertex& v1, const OVertex& v2,
	int* dev_DepthBuffer, int* dev_DepthMutexBuffer, PixelShade* dev_PixelShadeBuffer, unsigned int width, const GPUTexturesCompact& textures)
{
	const float v0InvDepth = 1.f / v0.p.w;
	const float v1InvDepth = 1.f / v1.p.w;
	const float v2InvDepth = 1.f / v2.p.w;

	float weights[3];
	if (IsPixelInTriangle(v0.p, v1.p, v2.p, pixel, weights))
	{
		const float totalArea = abs(Cross(v0.p.xy - v1.p.xy, v0.p.xy - v2.p.xy));
		weights[0] /= totalArea;
		weights[1] /= totalArea;
		weights[2] /= totalArea;

		const float zInterpolated = (weights[0] * v0.p.z) + (weights[1] * v1.p.z) + (weights[2] * v2.p.z);

		//peform early depth test
		if (zInterpolated < 0 || zInterpolated > 1.f)
			return;

		const float wInterpolated = 1.f / (v0InvDepth * weights[0] + v1InvDepth * weights[1] + v2InvDepth * weights[2]);

		//create pixelshade object (== fragment)
		PixelShade pixelShade;

		//depthbuffer visualisation
		pixelShade.zInterpolated = zInterpolated;
		pixelShade.wInterpolated = wInterpolated;

		//uv
		pixelShade.uv.x = weights[0] * (v0.uv.x * v0InvDepth) + weights[1] * (v1.uv.x * v1InvDepth) + weights[2] * (v2.uv.x * v2InvDepth);
		pixelShade.uv.y = weights[0] * (v0.uv.y * v0InvDepth) + weights[1] * (v1.uv.y * v1InvDepth) + weights[2] * (v2.uv.y * v2InvDepth);
		pixelShade.uv *= wInterpolated;

		//normal
		pixelShade.n.x = weights[0] * (v0.n.x * v0InvDepth) + weights[1] * (v1.n.x * v1InvDepth) + weights[2] * (v2.n.x * v2InvDepth);
		pixelShade.n.y = weights[0] * (v0.n.y * v0InvDepth) + weights[1] * (v1.n.y * v1InvDepth) + weights[2] * (v2.n.y * v2InvDepth);
		pixelShade.n.z = weights[0] * (v0.n.z * v0InvDepth) + weights[1] * (v1.n.z * v1InvDepth) + weights[2] * (v2.n.z * v2InvDepth);
		pixelShade.n *= wInterpolated;

		//tangent
		pixelShade.tan.x = weights[0] * (v0.tan.x * v0InvDepth) + weights[1] * (v1.tan.x * v1InvDepth) + weights[2] * (v2.tan.x * v2InvDepth);
		pixelShade.tan.y = weights[0] * (v0.tan.y * v0InvDepth) + weights[1] * (v1.tan.y * v1InvDepth) + weights[2] * (v2.tan.y * v2InvDepth);
		pixelShade.tan.z = weights[0] * (v0.tan.z * v0InvDepth) + weights[1] * (v1.tan.z * v1InvDepth) + weights[2] * (v2.tan.z * v2InvDepth);

		//view direction
		pixelShade.vd.x = weights[0] * (v0.vd.x * v0InvDepth) + weights[1] * (v1.vd.x * v1InvDepth) + weights[2] * (v2.vd.x * v2InvDepth);
		pixelShade.vd.y = weights[0] * (v0.vd.y * v0InvDepth) + weights[1] * (v1.vd.y * v1InvDepth) + weights[2] * (v2.vd.y * v2InvDepth);
		pixelShade.vd.z = weights[0] * (v0.vd.z * v0InvDepth) + weights[1] * (v1.vd.z * v1InvDepth) + weights[2] * (v2.vd.z * v2InvDepth);
		Normalize(pixelShade.vd);

		//colour
		const RGBColor interpolatedColour{
			weights[0] * v0.c.r + weights[1] * v1.c.r + weights[2] * v2.c.r,
			weights[0] * v0.c.g + weights[1] * v1.c.g + weights[2] * v2.c.g,
			weights[0] * v0.c.b + weights[1] * v1.c.b + weights[2] * v2.c.b };
		pixelShade.colour = RGBA::GetRGBAFromColour(interpolatedColour).colour32;

		//store textures
		pixelShade.textures = textures;

		//multiplying z value by a INT_MAX because atomicCAS only accepts ints
		const int scaledZ = zInterpolated * INT_MAX;

		const unsigned int pixelIdx = (unsigned int)pixel.x + (unsigned int)pixel.y * width;

		PerformDepthTestAtomic(dev_DepthBuffer, dev_DepthMutexBuffer, pixelIdx, scaledZ, dev_PixelShadeBuffer, pixelShade);
	}
}

GPU_CALLABLE static
void RasterizeTriangle(const BoundingBox& bb, const OVertex& v0, const OVertex& v1, const OVertex& v2,
	int* dev_DepthMutexBuffer, int* dev_DepthBuffer, PixelShade* dev_PixelShadeBuffer, unsigned int width, const GPUTexturesCompact& textures)
{
	//Loop over all pixels in bounding box
	for (unsigned short y = bb.yMin; y < bb.yMax; ++y)
	{
		for (unsigned short x = bb.xMin; x < bb.xMax; ++x)
		{
			const FPoint2 pixel{ float(x), float(y) };
			RasterizePixelAtomic(pixel, v0, v1, v2, dev_DepthBuffer, dev_DepthMutexBuffer, dev_PixelShadeBuffer, width, textures);
		}
	}
}

GPU_CALLABLE GPU_INLINE static
RGBColor ShadePixel(const GPUTexturesCompact& textures, const FVector2& uv, const FVector3& n, const FVector3& tan, const FVector3& vd,
	SampleState sampleState, bool isFlipGreenChannel = false)
{
	RGBColor finalColour{};

	//global settings
	const RGBColor ambientColour{ 0.05f, 0.05f, 0.05f };
	const FVector3 lightDirection = { 0.577f, -0.577f, -0.577f };
	const float lightIntensity = 7.0f;

	// texture sampling
	const RGBColor diffuseSample = GPUTextureSampler::Sample(textures.Diff, textures.w, textures.h, uv, sampleState);

	if (textures.Norm.dev_pTex != 0)
	{
		const RGBColor normalSample = GPUTextureSampler::Sample(textures.Norm, textures.w, textures.h, uv, sampleState);

		// normal mapping
		FVector3 binormal = Cross(tan, n);
		if (isFlipGreenChannel)
			binormal = -binormal;
		const FMatrix3 tangentSpaceAxis{ tan, binormal, n };

		FVector3 finalNormal{ 2.f * normalSample.r - 1.f, 2.f * normalSample.g - 1.f, 2.f * normalSample.b - 1.f };
		finalNormal = tangentSpaceAxis * finalNormal;

		// light calculations
		float observedArea{ Dot(-finalNormal, lightDirection) };
		Clamp(observedArea, 0.f, observedArea);
		observedArea /= (float)PI;
		observedArea *= lightIntensity;
		const RGBColor diffuseColour = diffuseSample * observedArea;

		if (textures.Spec.dev_pTex != 0 && textures.Gloss.dev_pTex != 0)
		{
			const RGBColor specularSample = GPUTextureSampler::Sample(textures.Spec, textures.w, textures.h, uv, sampleState);
			const RGBColor glossSample = GPUTextureSampler::Sample(textures.Gloss, textures.w, textures.h, uv, sampleState);

			// phong specular
			const FVector3 reflectV{ Reflect(lightDirection, finalNormal) };
			float angle{ Dot(reflectV, vd) };
			Clamp(angle, 0.f, 1.f);
			const float shininess = 25.f;
			angle = powf(angle, glossSample.r * shininess);
			const RGBColor specularColour = specularSample * angle;

			// final
			finalColour = ambientColour + diffuseColour + specularColour;
			finalColour.ClampColor();
		}
		else
		{
			finalColour = diffuseColour;
		}
	}
	else
	{
		finalColour = diffuseSample;
	}
	return finalColour;
}

GPU_CALLABLE GPU_INLINE static
void MultiplyMatVec(const float* pMat, float* pVec, unsigned int matSize, unsigned int vecSize)
{
	//thread goes through each element of vector
	float vec[4]{};
	for (unsigned int element{}; element < vecSize; ++element)
	{
		float sum{};
		for (unsigned int i{}; i < matSize; ++i)
		{
			sum += pMat[(element * matSize) + i] * pVec[i];
		}
		vec[element] = sum;
	}
	memcpy(pVec, vec, vecSize * 4);
}

GPU_CALLABLE GPU_INLINE static
void CalculateOutputPosXYZ(const float* pMat, float* pVec)
{
	constexpr unsigned int matSize = 4;
	constexpr unsigned int vecSize = 3;

	//thread goes through each element of vector
	float vec[3]{};
	for (unsigned int element{}; element < vecSize; ++element)
	{
		for (unsigned int i{}; i < vecSize; ++i)
		{
			vec[element] += pMat[(element * matSize) + i] * pVec[i];
		}
		vec[element] += pMat[(element * matSize) + 3]; // * pVec[w] == 1.f
	}
	memcpy(pVec, vec, 12);
}

GPU_CALLABLE GPU_INLINE static
void CalculateOutputPosXYZW(const float* pMat, float* pVec, float* pW)
{
	constexpr unsigned int matSize = 4;
	constexpr unsigned int vecSize = 3;

	//thread goes through each element of vector
	float vec[4]{};
	for (unsigned int element{}; element < vecSize; ++element)
	{
		for (unsigned int i{}; i < vecSize; ++i)
		{
			vec[element] += pMat[(element * matSize) + i] * pVec[i];
		}
		vec[element] += pMat[(element * matSize) + 3]; // * pVec[w] == 1.f
	}

	for (unsigned int i{}; i < vecSize; ++i)
	{
		vec[3] += pMat[12 + i] * pVec[i];
	}
	vec[3] += pMat[15]; // * pVec[w] == 1.f

	memcpy(pVec, vec, 12);
	*pW = vec[3];
}

//BINNING + TILING

GPU_CALLABLE static
BoundingBox GetBoundingBoxTiled(const FPoint2& v0, const FPoint2& v1, const FPoint2& v2,
	const unsigned int minX, const unsigned int minY, const unsigned int maxX, const unsigned int maxY)
{
	BoundingBox bb;
	bb.xMin = (short)GetMinElement(v0.x, v1.x, v2.x) - 1; // xMin
	bb.yMin = (short)GetMinElement(v0.y, v1.y, v2.y) - 1; // yMin
	bb.xMax = (short)GetMaxElement(v0.x, v1.x, v2.x) + 1; // xMax
	bb.yMax = (short)GetMaxElement(v0.y, v1.y, v2.y) + 1; // yMax

	if (bb.xMin < minX) bb.xMin = minX; //clamp minX to Left of screen
	if (bb.yMin < minY) bb.yMin = minY; //clamp minY to Bottom of screen
	if (bb.xMax > maxX) bb.xMax = maxX; //clamp maxX to Right of screen
	if (bb.yMax > maxY) bb.yMax = maxY; //clamp maxY to Top of screen

	return bb;
}

GPU_CALLABLE static
bool IsOverlapping(const BoundingBox& rect0, const BoundingBox& rect1)
{
	//if x is either left or right outside of other
	if (rect0.xMin > rect1.xMax || rect0.xMax < rect1.xMin)
		return false;

	//if y is either top or bottom outside of other
	if (rect0.yMin > rect1.yMax || rect0.yMax < rect1.yMin)
		return false;

	return true;
}