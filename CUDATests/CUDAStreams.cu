#include "hip/hip_runtime.h"
#include "PCH.h"

namespace CUDAStreams
{
	hipError_t StreamTestDONTCALL()
	{
		hipError_t cudaStatus{};
		const unsigned int N = 1024;
		float h_a[N]{};
		float* d_a{};

		//Synchronised calls
		hipMemcpy(d_a, h_a, N, hipMemcpyHostToDevice); //CPU will wait until this transfer is complete
		//(asynchronous) increment<<<1, N>>>(d_a)
		//myCpuFunction(b) (asynchronous && independent CPU code)
		hipMemcpy(h_a, d_a, N, hipMemcpyDeviceToHost); //CPU will wait until this transfer is complete

		//Create stream
		hipStream_t stream1;
		hipError_t result;
		result = hipStreamCreate(&stream1);
		result = hipStreamDestroy(stream1);

		//Asynchronised calls aka NON-BLOCKING the host
		result = hipMemcpyAsync(d_a, h_a, N, hipMemcpyHostToDevice, stream1);

		//increment<<<1, N, 0, stream1>>>(d_a)

		return cudaStatus;
	}

	__global__ void kernel(float* a, int offset)
	{
		int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
		float x = (float)i;
		float s = sinf(x);
		float c = cosf(x);
		a[i] = a[i] + sqrtf(s * s + c * c);
	}

	float maxError(float* a, int n)
	{
		float maxE = 0;
		for (int i = 0; i < n; i++)
		{
			float error = fabs(a[i] - 1.0f);
			if (error > maxE)
				maxE = error;
		}
		return maxE;
	}

	hipError_t StreamCuda()
	{
		hipError_t cudaStatus{};

		const int blockSize = 256, nStreams = 4;
		const int n = 4 * 1024 * blockSize * nStreams; //16MBs * 4 streams
		const int streamSize = n / nStreams; //16MBs
		const int nStreamBytes = streamSize * sizeof(float); //16MBs * 4
		const int nBytes = n * sizeof(float); //total amount of bytes

		hipDeviceProp_t prop;
		checkCuda(hipGetDeviceProperties(&prop, 0));
		printf("Device : %s\n", prop.name);
		checkCuda(hipSetDevice(0));

		// allocate pinned host memory and device memory
		float* h_a, * d_a;
		checkCuda(hipHostMalloc((void**)&h_a, nBytes));      // host pinned
		checkCuda(hipMalloc((void**)&d_a, nBytes)); // device

		float ms; // elapsed time in milliseconds

		// create events and streams
		hipEvent_t startEvent, stopEvent, dummyEvent;
		hipStream_t stream[nStreams];
		checkCuda(hipEventCreate(&startEvent));
		checkCuda(hipEventCreate(&stopEvent));
		checkCuda(hipEventCreate(&dummyEvent));
		for (int i = 0; i < nStreams; ++i)
		{
			checkCuda(hipStreamCreate(&stream[i]));
		}

		// baseline case - sequential transfer and execute
		memset(h_a, 0, nBytes);
		checkCuda(hipEventRecord(startEvent, 0));
		checkCuda(hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice));

		//		NumBlocks,		ThreadsPerBlock
		kernel<<<n / blockSize, blockSize>>>(d_a, 0);

		checkCuda(hipMemcpy(h_a, d_a, nBytes, hipMemcpyDeviceToHost));
		checkCuda(hipEventRecord(stopEvent, 0));
		checkCuda(hipEventSynchronize(stopEvent));
		checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

		printf("Time for sequential transfer and execute (ms): %f\n", ms);
		printf("  max error: %e\n", maxError(h_a, n));

		// asynchronous version 1: loop over {copy, kernel, copy}
		memset(h_a, 0, nBytes);
		checkCuda(hipEventRecord(startEvent, 0));

		for (int i = 0; i < nStreams; ++i)
		{
			int offset = i * streamSize;
			checkCuda(hipMemcpyAsync(&d_a[offset], &h_a[offset], nStreamBytes, hipMemcpyHostToDevice, stream[i]));

			kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);

			checkCuda(hipMemcpyAsync(&h_a[offset], &d_a[offset], nStreamBytes, hipMemcpyDeviceToHost,stream[i]));
		}

		checkCuda(hipEventRecord(stopEvent, 0));
		checkCuda(hipEventSynchronize(stopEvent));
		checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
		printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
		printf("  max error: %e\n", maxError(h_a, n));

		// asynchronous version 2: 
		// loop over copy, loop over kernel, loop over copy
		memset(h_a, 0, nBytes);
		checkCuda(hipEventRecord(startEvent, 0));

		for (int i = 0; i < nStreams; ++i)
		{
			int offset = i * streamSize;
			checkCuda(hipMemcpyAsync(&d_a[offset], &h_a[offset], nStreamBytes, hipMemcpyHostToDevice, stream[i]));
		}

		for (int i = 0; i < nStreams; ++i)
		{
			int offset = i * streamSize;
			kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
		}

		for (int i = 0; i < nStreams; ++i)
		{
			int offset = i * streamSize;
			checkCuda(hipMemcpyAsync(&h_a[offset], &d_a[offset], nStreamBytes, hipMemcpyDeviceToHost,stream[i]));
		}

		checkCuda(hipEventRecord(stopEvent, 0));
		checkCuda(hipEventSynchronize(stopEvent));
		checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
		printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
		printf("  max error: %e\n", maxError(h_a, n));

		// cleanup
		checkCuda(hipEventDestroy(startEvent));
		checkCuda(hipEventDestroy(stopEvent));
		checkCuda(hipEventDestroy(dummyEvent));

		for (int i = 0; i < nStreams; ++i)
		{
			checkCuda(hipStreamDestroy(stream[i]));
		}

		hipFree(d_a);
		hipHostFree(h_a);

		return cudaStatus;
	}
}